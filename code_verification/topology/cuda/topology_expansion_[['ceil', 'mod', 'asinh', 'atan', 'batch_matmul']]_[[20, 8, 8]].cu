
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_6) {
  float T_batch_matmul_NN_local[2];
  __shared__ float ph_6_shared[50];
  for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
    T_batch_matmul_NN_local[b_c_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_outer_outer * 4) + (((int)threadIdx.x) / 10)) < 5) {
      ph_6_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 40) + ((int)threadIdx.x))] = ph_6[(((((int)blockIdx.x) * 50) + (ax0_ax1_fused_ax2_fused_outer_outer * 40)) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 5; ++k_inner) {
    for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
      T_batch_matmul_NN_local[b_c_inner] = (T_batch_matmul_NN_local[b_c_inner] + (ph_0[((((((int)blockIdx.x) * 80) + (b_c_inner * 40)) + ((((int)threadIdx.x) / 5) * 5)) + k_inner)] * ph_6_shared[(((b_c_inner * 25) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[(((((int)blockIdx.x) * 80) + (b_inner * 40)) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[b_inner];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

