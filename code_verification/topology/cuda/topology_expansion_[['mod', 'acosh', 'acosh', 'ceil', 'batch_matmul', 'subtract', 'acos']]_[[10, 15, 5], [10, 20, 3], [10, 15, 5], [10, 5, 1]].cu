
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void default_function_kernel_3(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((int)blockIdx.x)] = (fmodf(ph_0[((int)blockIdx.x)], ph_3[((int)blockIdx.x)]) - ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(18) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0) {
  float T_batch_matmul_NN_local[32];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_batch_matmul_NN_local[(((i_c_outer_inner_init * 8) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
        for (int k_inner = 0; k_inner < 4; ++k_inner) {
          for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
            T_batch_matmul_NN_local[(((i_c_outer_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_batch_matmul_NN_local[(((i_c_outer_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner)] + (acoshf(ph_0[((((((int)threadIdx.x) * 32) + (i_c_outer_inner * 8)) + (k_outer_outer * 4)) + k_inner)]) * ph_0[((((((((int)threadIdx.x) >> 1) * 64) + (k_outer_outer * 32)) + (k_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_batch_matmul_NN[(((((int)threadIdx.x) * 32) + (i_inner * 8)) + j_inner)] = T_batch_matmul_NN_local[((i_inner * 8) + j_inner)];
    }
  }
}

