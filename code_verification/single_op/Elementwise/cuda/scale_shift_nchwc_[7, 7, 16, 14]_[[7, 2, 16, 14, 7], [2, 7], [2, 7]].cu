
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) % 49) * 2) + (((int)threadIdx.x) >> 5)) / 49) * 7) + ((((int)blockIdx.x) + ((int)threadIdx.x)) % 7))]) + Shift[((((((((int)blockIdx.x) % 49) * 2) + (((int)threadIdx.x) >> 5)) / 49) * 7) + ((((int)blockIdx.x) + ((int)threadIdx.x)) % 7))]);
}

