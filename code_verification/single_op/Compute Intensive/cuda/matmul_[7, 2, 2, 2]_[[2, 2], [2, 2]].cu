
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[2];
  __shared__ float left_matrix_shared[4];
  __shared__ float right_matrix_shared[4];
  T_matmul_local[0] = 0.000000e+00f;
  T_matmul_local[1] = 0.000000e+00f;
  for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
    left_matrix_shared[((ax0_ax1_fused_outer_outer * 2) + ((int)threadIdx.x))] = left_matrix[((ax0_ax1_fused_outer_outer * 2) + ((int)threadIdx.x))];
  }
  for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 2; ++ax0_ax1_fused_outer_outer_1) {
    right_matrix_shared[((ax0_ax1_fused_outer_outer_1 * 2) + ((int)threadIdx.x))] = right_matrix[((ax0_ax1_fused_outer_outer_1 * 2) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 2; ++k_inner) {
    T_matmul_local[0] = (T_matmul_local[0] + (left_matrix_shared[((((int)threadIdx.x) * 2) + k_inner)] * right_matrix_shared[(k_inner * 2)]));
    T_matmul_local[1] = (T_matmul_local[1] + (left_matrix_shared[((((int)threadIdx.x) * 2) + k_inner)] * right_matrix_shared[((k_inner * 2) + 1)]));
  }
  T_matmul[(((int)threadIdx.x) * 2)] = T_matmul_local[0];
  T_matmul[((((int)threadIdx.x) * 2) + 1)] = T_matmul_local[1];
}

