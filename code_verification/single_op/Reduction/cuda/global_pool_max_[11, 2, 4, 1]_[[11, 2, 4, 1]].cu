
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(22) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((int)threadIdx.x)] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 4; ++rv0) {
    adaptive_pool_max[((int)threadIdx.x)] = max(adaptive_pool_max[((int)threadIdx.x)], data[((((int)threadIdx.x) * 4) + rv0)]);
  }
}

