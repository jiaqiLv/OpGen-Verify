
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __log2f(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

