
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 6074775) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(21) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))]));
}
