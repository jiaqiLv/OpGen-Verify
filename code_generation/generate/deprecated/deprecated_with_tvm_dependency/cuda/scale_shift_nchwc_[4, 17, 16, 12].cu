
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 136) / 68) * 17) + (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 17))]) + Shift[((((((int)blockIdx.x) % 136) / 68) * 17) + (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 17))]);
}

