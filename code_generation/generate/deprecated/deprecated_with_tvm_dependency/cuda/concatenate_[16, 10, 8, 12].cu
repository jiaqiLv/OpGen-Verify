
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_concat, float* __restrict__ data_a, float* __restrict__ data_b);
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_concat, float* __restrict__ data_a, float* __restrict__ data_b) {
  T_concat[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = ((80 <= ((int)blockIdx.x)) ? data_b[(((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) - 1280)] : data_a[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

