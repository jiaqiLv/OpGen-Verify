
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight);
extern "C" __global__ void __launch_bounds__(15) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 375) {
    T_cast[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 150) / 15)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 75) * 15) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 15))] * 1.000000e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(15) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 10; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))] + (data[(((((int)blockIdx.x) * 150) + (k1 * 15)) + ((int)threadIdx.x))] * data[(((((int)blockIdx.x) * 150) + (k1 * 15)) + ((int)threadIdx.x))]));
  }
}

