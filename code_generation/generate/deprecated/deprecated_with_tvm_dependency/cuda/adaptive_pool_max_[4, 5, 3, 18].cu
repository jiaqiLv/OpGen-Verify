
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((int)threadIdx.x) >> 3) * 3) + 3) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 3) + 3) >> 3) : (((((((int)threadIdx.x) >> 3) * 3) + 3) >> 3) + 1)) - (((((int)threadIdx.x) >> 3) * 3) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 2) + 2) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 9) + 9) >> 2) : (((((((int)threadIdx.x) & 7) * 9) + 9) >> 2) + 1)) - (((((int)threadIdx.x) & 7) * 18) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) * 54) + ((((((int)threadIdx.x) >> 3) * 3) >> 3) * 18)) + (rv0 * 18)) + (((((int)threadIdx.x) & 7) * 18) >> 3)) + rv1)]);
    }
  }
}

