
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ data) {
  T_strided_slice[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = data[((((((((((int)blockIdx.x) & 3) >> 1) * 3456) + (((((int)threadIdx.x) & 15) >> 3) * 1728)) + ((((int)blockIdx.x) >> 2) * 48)) + ((((int)blockIdx.x) & 1) * 24)) + ((((int)threadIdx.x) >> 4) * 8)) + (((int)threadIdx.x) & 7))];
}
