
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(28) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum);
extern "C" __global__ void __launch_bounds__(28) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(28) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] * 1.666667e-02f);
}

extern "C" __global__ void __launch_bounds__(28) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)threadIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 20; ++rv1) {
      adaptive_pool_sum[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] + data[(((((int)threadIdx.x) * 60) + (rv0 * 20)) + rv1)]);
    }
  }
}

