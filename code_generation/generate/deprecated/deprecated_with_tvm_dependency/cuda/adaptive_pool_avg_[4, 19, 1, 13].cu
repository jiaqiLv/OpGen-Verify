
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum);
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / (((float)(((((((((int)blockIdx.x) & 1) * 4) + (((int)threadIdx.x) >> 3)) + 1) % 8) == 0) ? ((((((int)threadIdx.x) + 8) >> 5) + (((int)blockIdx.x) & 1)) >> 1) : (((((((int)threadIdx.x) + 8) >> 5) + (((int)blockIdx.x) & 1)) >> 1) + 1))) * ((float)(((((((((int)threadIdx.x) & 7) * 5) + 5) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 13) + 13) >> 3) : (((((((int)threadIdx.x) & 7) * 13) + 13) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 13) >> 3)))));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < (((((((int)blockIdx.x) & 7) + 1) % 8) == 0) ? (((((int)blockIdx.x) & 7) + 1) >> 3) : ((((((int)blockIdx.x) & 7) + 1) >> 3) + 1)); ++rv0) {
    for (int rv1 = 0; rv1 < ((((((((int)threadIdx.x) * 5) + 5) % 8) == 0) ? (((((int)threadIdx.x) * 13) + 13) >> 3) : ((((((int)threadIdx.x) * 13) + 13) >> 3) + 1)) - ((((int)threadIdx.x) * 13) >> 3)); ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + data[(((((((int)blockIdx.x) >> 3) * 13) + (rv0 * 13)) + ((((int)threadIdx.x) * 13) >> 3)) + rv1)]);
    }
  }
}

