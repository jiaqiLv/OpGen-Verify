
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ PadInput, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ PadInput, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 95) {
    PadInput[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (((((5 <= ((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2))) && (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 85)) && (1 <= (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) % 10))) && ((((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 20) < 19)) ? data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) / 5) * 17) + (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 20)) - 19)] : 0.000000e+00f);
  }
}

