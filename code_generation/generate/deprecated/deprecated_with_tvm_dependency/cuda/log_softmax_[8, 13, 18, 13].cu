
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 1521) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 13)]) - __logf(compute_1[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 13)]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 117) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 13; ++k) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 117) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 416) + (((int)threadIdx.x) * 13)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 13; ++k) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 208) + (((int)threadIdx.x) * 13)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

