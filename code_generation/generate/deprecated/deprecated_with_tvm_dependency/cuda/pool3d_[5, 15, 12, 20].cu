
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max);
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 30) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      for (int rv2 = 0; rv2 < 3; ++rv2) {
        pool_max[((((int)blockIdx.x) * 30) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 30) + ((int)threadIdx.x))], ((((1 <= ((((((((int)blockIdx.x) & 15) * 3) + (((int)threadIdx.x) / 10)) >> 3) * 2) + rv0)) && (1 <= ((((((((int)blockIdx.x) * 15) + (((int)threadIdx.x) >> 1)) % 40) >> 2) * 2) + rv1))) && (1 <= (((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) & 7) * 2) + rv2))) ? data[((((((((((((int)blockIdx.x) * 3) + (((int)threadIdx.x) / 10)) >> 3) * 640) + (rv0 * 320)) + (((((((int)blockIdx.x) * 15) + (((int)threadIdx.x) >> 1)) % 40) >> 2) * 32)) + (rv1 * 16)) + ((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) & 7) * 2)) + rv2) - 337)] : -3.402823e+38f));
      }
    }
  }
}

