
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ compute, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = tanhf(data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))]);
}

