
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(56) default_function_kernel(float* __restrict__ compute, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(56) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 14) + (((int)threadIdx.x) >> 2)) < 1485) {
    compute[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - data[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))]))));
  }
}

