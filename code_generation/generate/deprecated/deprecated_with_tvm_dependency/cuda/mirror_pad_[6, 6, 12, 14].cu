
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(63) default_function_kernel(float* __restrict__ MirrorPadInput, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(63) default_function_kernel(float* __restrict__ MirrorPadInput, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 21) + (((int)threadIdx.x) / 3)) < 85) {
    MirrorPadInput[((((int)blockIdx.x) * 63) + ((int)threadIdx.x))] = data[((((221 <= ((((int)blockIdx.x) * 63) + ((int)threadIdx.x))) ? (24 - (((((int)blockIdx.x) * 63) + ((int)threadIdx.x)) / 17)) : ((((((int)blockIdx.x) * 63) + ((int)threadIdx.x)) < 17) ? 0 : ((((((int)blockIdx.x) * 63) + ((int)threadIdx.x)) / 17) - 1))) * 14) + (((((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 17) == 16) ? (29 - (((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 17)) : (((((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 17) < 2) ? (1 - (((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 17)) : ((((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 17) - 2))))];
  }
}

