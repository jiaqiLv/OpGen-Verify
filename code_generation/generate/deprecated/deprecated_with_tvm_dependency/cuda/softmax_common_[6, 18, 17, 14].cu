
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 459) {
    T_softmax_expsum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 14; ++k) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 459) {
        int v_ = ((int)(floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
      T_softmax_expsum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[(((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 459) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 14; ++k) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 459) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 448) + (((int)threadIdx.x) * 14)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 3213) {
      int v_ = ((int)(floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_softmax_norm[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)])) / T_softmax_expsum[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 7)]);
  }
}

