
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1);
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1);
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]));
}

