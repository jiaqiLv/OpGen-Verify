
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var);
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] - moving_mean[((((int)blockIdx.x) % 60) >> 2)]) / sqrtf((moving_var[((((int)blockIdx.x) % 60) >> 2)] + 1.000000e-05f)));
}

