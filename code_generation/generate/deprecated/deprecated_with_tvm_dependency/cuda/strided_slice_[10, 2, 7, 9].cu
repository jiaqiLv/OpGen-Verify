
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ a);
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ a) {
  T_strided_slice[((int)threadIdx.x)] = a[((((((int)threadIdx.x) / 6) * 9) + (((int)threadIdx.x) % 6)) + 84)];
}

