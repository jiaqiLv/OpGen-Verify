
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space);
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space) {
  depth_to_space[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = data[(((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) / 85) * 1615) + ((((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) % 170) / 17) % 2) * 680)) + (((((((int)blockIdx.x) * 30) + ((int)threadIdx.x)) % 34) % 2) * 340)) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) % 340) / 85) * 85)) + ((((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) % 170) / 17) / 2) * 17)) + ((((((int)blockIdx.x) * 30) + ((int)threadIdx.x)) % 34) / 2))];
}

