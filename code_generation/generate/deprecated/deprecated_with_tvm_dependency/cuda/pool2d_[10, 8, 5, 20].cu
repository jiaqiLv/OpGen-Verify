
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(18) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max);
extern "C" __global__ void __launch_bounds__(18) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      pool_max[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))], ((((1 <= ((((((((int)blockIdx.x) % 5) * 3) + (((int)threadIdx.x) / 6)) / 5) * 2) + rv0)) && ((((((((int)blockIdx.x) % 5) * 3) + (((int)threadIdx.x) / 6)) / 5) + (rv0 >> 1)) < 3)) && (1 <= (((((((int)blockIdx.x) * 18) + ((int)threadIdx.x)) % 30) * 2) + rv1))) ? data[(((((((((int)blockIdx.x) / 5) * 300) + (((((((int)blockIdx.x) % 5) * 3) + (((int)threadIdx.x) / 6)) / 5) * 120)) + (rv0 * 60)) + ((((((int)blockIdx.x) * 18) + ((int)threadIdx.x)) % 30) * 2)) + rv1) - 61)] : -3.402823e+38f));
    }
  }
}

