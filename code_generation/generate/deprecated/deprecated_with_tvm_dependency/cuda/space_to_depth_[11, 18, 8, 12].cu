
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(54) default_function_kernel(float* __restrict__ data, float* __restrict__ space_to_depth);
extern "C" __global__ void __launch_bounds__(54) default_function_kernel(float* __restrict__ data, float* __restrict__ space_to_depth) {
  space_to_depth[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))] = data[(((((((((int)blockIdx.x) >> 7) * 6912) + ((((((((int)blockIdx.x) & 127) * 9) + (((int)threadIdx.x) / 6)) >> 4) % 18) * 384)) + (((((((int)blockIdx.x) * 9) + (((int)threadIdx.x) / 6)) & 15) >> 1) * 48)) + ((((((((int)blockIdx.x) & 127) * 9) + (((int)threadIdx.x) / 6)) >> 4) / 36) * 24)) + ((((((int)blockIdx.x) * 6) + ((int)threadIdx.x)) % 12) * 2)) + ((((((((int)blockIdx.x) & 127) * 9) + (((int)threadIdx.x) / 6)) >> 4) % 36) / 18))];
}

