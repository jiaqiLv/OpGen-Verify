
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(51) default_function_kernel(float* __restrict__ T_fast_erf, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(51) default_function_kernel(float* __restrict__ T_fast_erf, float* __restrict__ data) {
  T_fast_erf[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))] = ((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * -2.726142e-10f) + 2.770681e-08f)) + -2.101024e-06f)) + -5.692506e-05f)) + -7.349906e-04f)) + -2.954600e-03f)) + -1.609603e-02f)) / (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * (((max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f) * max(min(data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))], 4.000000e+00f), -4.000000e+00f)) * -1.456607e-05f) + -2.133740e-04f)) + -1.682827e-03f)) + -7.373329e-03f)) + -1.426474e-02f));
}

