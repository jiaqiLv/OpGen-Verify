
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 15; ++rv0) {
    for (int rv1 = 0; rv1 < 17; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 2040) + (((int)threadIdx.x) * 255)) + (rv0 * 17)) + rv1)]);
    }
  }
}

