
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int k = 0; k < 18; ++k) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 576) + (((int)threadIdx.x) * 18)) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  T_softmax_norm[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (__expf((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) / 9)])) / T_softmax_expsum[(((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) / 9)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_expsum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 18; ++k) {
    T_softmax_expsum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 576) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

