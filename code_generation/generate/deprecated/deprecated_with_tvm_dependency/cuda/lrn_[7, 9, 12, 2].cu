
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor);
extern "C" __global__ void __launch_bounds__(21) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor);
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 189) {
    T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / powf((2.000000e+00f + (1.000000e-04f * tensor[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])), 7.500000e-01f));
  }
}

extern "C" __global__ void __launch_bounds__(21) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor) {
  tensor[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] = 0.000000e+00f;
  tensor[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] = (tensor[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] + (data[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] * data[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))]));
}

