
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif


#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ data_red);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ data_red) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  normal_reduce_temp0[0] = 1.000000e+00f;
  for (int k0_k1_fused_k2_fused_k3_fused_outer = 0; k0_k1_fused_k2_fused_k3_fused_outer < 1617; ++k0_k1_fused_k2_fused_k3_fused_outer) {
    if (((k0_k1_fused_k2_fused_k3_fused_outer * 16) + (((int)threadIdx.x) >> 1)) < 25857) {
      normal_reduce_temp0[0] = (normal_reduce_temp0[0] * data[((k0_k1_fused_k2_fused_k3_fused_outer * 32) + ((int)threadIdx.x))]);
    }
  }
  uint mask[1];
  float t0[1];
  red_buf0[0] = normal_reduce_temp0[0];
  mask[0] = __activemask();
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
  red_buf0[0] = (red_buf0[0] * t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
  red_buf0[0] = (red_buf0[0] * t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  red_buf0[0] = (red_buf0[0] * t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = (red_buf0[0] * t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = (red_buf0[0] * t0[0]);
  red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], 0, 32);
  if (((int)threadIdx.x) == 0) {
    data_red[0] = red_buf0[0];
  }
}

