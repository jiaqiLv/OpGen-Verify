
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ conv1d_ncw, float* __restrict__ data, float* __restrict__ kernel);
extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ conv1d_ncw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv1d_ncw_local[1];
  __shared__ float pad_temp_shared[10];
  __shared__ float kernel_shared[15];
  conv1d_ncw_local[0] = 0.000000e+00f;
  for (int rc_outer_outer = 0; rc_outer_outer < 3; ++rc_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 10) {
      pad_temp_shared[((int)threadIdx.x)] = data[(((((((int)threadIdx.x) / 5) * 30) + (rc_outer_outer * 10)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) % 5))];
    }
    if (((int)threadIdx.x) < 15) {
      kernel_shared[((int)threadIdx.x)] = kernel[((((((int)threadIdx.x) / 3) * 9) + (rc_outer_outer * 3)) + (((int)threadIdx.x) % 3))];
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      conv1d_ncw_local[0] = (conv1d_ncw_local[0] + (pad_temp_shared[((((((int)threadIdx.x) / 10) * 5) + ((((int)threadIdx.x) & 1) * 2)) + ry_inner)] * kernel_shared[((((((int)threadIdx.x) % 10) >> 1) * 3) + ry_inner)]));
    }
  }
  conv1d_ncw[((((((int)threadIdx.x) >> 1) * 4) + (((int)blockIdx.x) * 2)) + (((int)threadIdx.x) & 1))] = conv1d_ncw_local[0];
}

