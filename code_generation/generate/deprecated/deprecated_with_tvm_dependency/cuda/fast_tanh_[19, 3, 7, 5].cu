
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(57) default_function_kernel(float* __restrict__ T_fast_tanh, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(57) default_function_kernel(float* __restrict__ T_fast_tanh, float* __restrict__ data) {
  T_fast_tanh[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))] = ((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * -2.760768e-16f) + 2.000188e-13f)) + -8.604672e-11f)) + 5.122297e-08f)) + 1.485722e-05f)) + 6.372619e-04f)) + 4.893525e-03f)) / (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * (((max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))])) * max(-9.000000e+00f, min(9.000000e+00f, data[((((int)blockIdx.x) * 57) + ((int)threadIdx.x))]))) * 1.198258e-06f) + 1.185347e-04f)) + 2.268435e-03f)) + 4.893525e-03f));
}

