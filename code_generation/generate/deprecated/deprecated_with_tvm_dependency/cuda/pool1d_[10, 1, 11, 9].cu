
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max);
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) < 25) {
    pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) < 25) {
      pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], (((1 <= ((((((int)blockIdx.x) + ((int)threadIdx.x)) % 5) * 2) + rv0)) && (((rv0 >> 1) + ((((int)blockIdx.x) + ((int)threadIdx.x)) % 5)) < 5)) ? data[(((((((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) / 5) * 9) + (((((int)blockIdx.x) + ((int)threadIdx.x)) % 5) * 2)) + rv0) - 1)] : -3.402823e+38f));
    }
  }
}

