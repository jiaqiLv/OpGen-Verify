
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_fast_exp, float* __restrict__ data);
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_fast_exp, float* __restrict__ data) {
    int v_ = ((int)(floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
  T_fast_exp[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

