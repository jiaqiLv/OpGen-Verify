
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(49) default_function_kernel(float* __restrict__ output_unpack, float* __restrict__ packed_out);
extern "C" __global__ void __launch_bounds__(49) default_function_kernel(float* __restrict__ output_unpack, float* __restrict__ packed_out) {
  output_unpack[((((int)blockIdx.x) * 49) + ((int)threadIdx.x))] = packed_out[(((((((int)blockIdx.x) >> 2) * 196) + ((((int)blockIdx.x) & 1) * 98)) + (((int)threadIdx.x) * 2)) + ((((int)blockIdx.x) & 3) >> 1))];
}

