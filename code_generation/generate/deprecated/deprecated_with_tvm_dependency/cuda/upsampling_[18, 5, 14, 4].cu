
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(36) default_function_kernel(float* __restrict__ data, float* __restrict__ resize);
extern "C" __global__ void __launch_bounds__(36) default_function_kernel(float* __restrict__ data, float* __restrict__ resize) {
  resize[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) * 9) + (((int)threadIdx.x) >> 2)) / 224) * 224) + ((((((((int)blockIdx.x) * 9) + (((int)threadIdx.x) >> 2)) % 224) >> 2) / 2) * 8)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) & 15) / 2))];
}

