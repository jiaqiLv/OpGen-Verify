
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(5) default_function_kernel(float* __restrict__ conv1d_ncw, float* __restrict__ data, float* __restrict__ kernel);
extern "C" __global__ void __launch_bounds__(5) default_function_kernel(float* __restrict__ conv1d_ncw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv1d_ncw_local[8];
  __shared__ float pad_temp_shared[54];
  __shared__ float kernel_shared[45];
  conv1d_ncw_local[0] = 0.000000e+00f;
  conv1d_ncw_local[4] = 0.000000e+00f;
  conv1d_ncw_local[1] = 0.000000e+00f;
  conv1d_ncw_local[5] = 0.000000e+00f;
  conv1d_ncw_local[2] = 0.000000e+00f;
  conv1d_ncw_local[6] = 0.000000e+00f;
  conv1d_ncw_local[3] = 0.000000e+00f;
  conv1d_ncw_local[7] = 0.000000e+00f;
  pad_temp_shared[((int)threadIdx.x)] = data[((int)threadIdx.x)];
  pad_temp_shared[(((int)threadIdx.x) + 5)] = data[((((((int)threadIdx.x) + 5) / 9) * 10) + ((((int)threadIdx.x) + 5) % 9))];
  pad_temp_shared[(((int)threadIdx.x) + 10)] = data[(((((((int)threadIdx.x) + 10) / 9) * 10) + ((int)threadIdx.x)) + 1)];
  pad_temp_shared[(((int)threadIdx.x) + 15)] = data[((((((int)threadIdx.x) + 15) / 9) * 10) + ((((int)threadIdx.x) + 6) % 9))];
  pad_temp_shared[(((int)threadIdx.x) + 20)] = data[(((((((int)threadIdx.x) + 20) / 9) * 10) + ((int)threadIdx.x)) + 2)];
  pad_temp_shared[(((int)threadIdx.x) + 25)] = data[((((((int)threadIdx.x) + 25) / 9) * 10) + ((((int)threadIdx.x) + 7) % 9))];
  pad_temp_shared[(((int)threadIdx.x) + 30)] = data[(((((((int)threadIdx.x) + 30) / 9) * 10) + ((int)threadIdx.x)) + 3)];
  pad_temp_shared[(((int)threadIdx.x) + 35)] = data[((((((int)threadIdx.x) + 35) / 9) * 10) + ((((int)threadIdx.x) + 8) % 9))];
  pad_temp_shared[(((int)threadIdx.x) + 40)] = data[(((((((int)threadIdx.x) + 40) / 9) * 10) + ((int)threadIdx.x)) + 4)];
  pad_temp_shared[(((int)threadIdx.x) + 45)] = data[(((int)threadIdx.x) + 50)];
  if (((int)threadIdx.x) < 4) {
    pad_temp_shared[(((int)threadIdx.x) + 50)] = data[(((((((int)threadIdx.x) + 50) / 9) * 10) + ((int)threadIdx.x)) + 5)];
  }
  kernel_shared[((int)threadIdx.x)] = kernel[((int)threadIdx.x)];
  kernel_shared[(((int)threadIdx.x) + 5)] = kernel[(((int)threadIdx.x) + 5)];
  kernel_shared[(((int)threadIdx.x) + 10)] = kernel[(((int)threadIdx.x) + 10)];
  kernel_shared[(((int)threadIdx.x) + 15)] = kernel[(((int)threadIdx.x) + 15)];
  kernel_shared[(((int)threadIdx.x) + 20)] = kernel[(((int)threadIdx.x) + 20)];
  kernel_shared[(((int)threadIdx.x) + 25)] = kernel[(((int)threadIdx.x) + 25)];
  kernel_shared[(((int)threadIdx.x) + 30)] = kernel[(((int)threadIdx.x) + 30)];
  kernel_shared[(((int)threadIdx.x) + 35)] = kernel[(((int)threadIdx.x) + 35)];
  kernel_shared[(((int)threadIdx.x) + 40)] = kernel[(((int)threadIdx.x) + 40)];
  __syncthreads();
  for (int rc_outer_inner = 0; rc_outer_inner < 3; ++rc_outer_inner) {
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      conv1d_ncw_local[0] = (conv1d_ncw_local[0] + (pad_temp_shared[((rc_outer_inner * 9) + ry_inner)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[4] = (conv1d_ncw_local[4] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 4)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[1] = (conv1d_ncw_local[1] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 2)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[5] = (conv1d_ncw_local[5] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 6)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[2] = (conv1d_ncw_local[2] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 27)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[6] = (conv1d_ncw_local[6] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 31)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[3] = (conv1d_ncw_local[3] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 29)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
      conv1d_ncw_local[7] = (conv1d_ncw_local[7] + (pad_temp_shared[(((rc_outer_inner * 9) + ry_inner) + 33)] * kernel_shared[(((((int)threadIdx.x) * 9) + (rc_outer_inner * 3)) + ry_inner)]));
    }
  }
  for (int nn_inner = 0; nn_inner < 2; ++nn_inner) {
    for (int yy_inner = 0; yy_inner < 2; ++yy_inner) {
      conv1d_ncw[(((nn_inner * 20) + (((int)threadIdx.x) * 4)) + yy_inner)] = conv1d_ncw_local[((nn_inner * 2) + yy_inner)];
      conv1d_ncw[((((nn_inner * 20) + (((int)threadIdx.x) * 4)) + yy_inner) + 2)] = conv1d_ncw_local[(((nn_inner * 2) + yy_inner) + 4)];
    }
  }
}

