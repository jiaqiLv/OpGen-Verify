
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(int* __restrict__ T_shape);
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(int* __restrict__ T_shape) {
  T_shape[((int)threadIdx.x)] = ((((int)threadIdx.x) == 3) ? 1 : ((((int)threadIdx.x) == 2) ? 2 : ((((int)threadIdx.x) == 1) ? 15 : 16)));
}

