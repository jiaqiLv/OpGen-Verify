
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 11115) {
    ScaleShift[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) % 2223) / 171)]) + Shift[((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) % 2223) / 171)]);
  }
}

