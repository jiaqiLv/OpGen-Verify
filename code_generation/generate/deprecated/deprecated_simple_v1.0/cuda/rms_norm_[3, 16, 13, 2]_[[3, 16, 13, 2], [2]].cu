
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) % 26) * 8) + (((int)threadIdx.x) >> 1)) / 13)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((int)blockIdx.x) / 26) * 26) + ((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) % 13) * 2)) + (((int)threadIdx.x) & 1))] * 6.250000e-02f) + 1.000000e-05f))));
}

extern "C" __global__ void default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((int)blockIdx.x)] = 0.000000e+00f;
  for (int k1 = 0; k1 < 16; ++k1) {
    T_multiply_red[((int)blockIdx.x)] = (T_multiply_red[((int)blockIdx.x)] + (data[((((((int)blockIdx.x) / 26) * 416) + (k1 * 26)) + (((int)blockIdx.x) % 26))] * data[((((((int)blockIdx.x) / 26) * 416) + (k1 * 26)) + (((int)blockIdx.x) % 26))]));
  }
}

