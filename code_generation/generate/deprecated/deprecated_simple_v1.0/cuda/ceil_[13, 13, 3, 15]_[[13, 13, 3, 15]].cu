
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(50) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 10) + (((int)threadIdx.x) / 5)) < 1521) {
    compute[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] = ceilf(data[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))]);
  }
}

