
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) & 15) >> 3) * 16) + (((int)threadIdx.x) & 15))]) + Shift[((((((int)blockIdx.x) & 15) >> 3) * 16) + (((int)threadIdx.x) & 15))]);
}

