
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * Scale[0]) + Shift[0]);
}

