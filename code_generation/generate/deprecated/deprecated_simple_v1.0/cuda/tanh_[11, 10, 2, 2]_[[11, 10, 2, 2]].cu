
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 1)) < 220) {
    compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = tanhf(data[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]);
  }
}

