
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 143) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __log2f(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

