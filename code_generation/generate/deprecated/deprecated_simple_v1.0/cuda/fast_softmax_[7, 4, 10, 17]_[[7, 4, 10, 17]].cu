
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
    int v_ = ((int)(floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
  T_softmax_norm[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)])) / T_softmax_expsum[(((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 17)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 35) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 17; ++k) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 35) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 544) + (((int)threadIdx.x) * 17)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_expsum[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 17; ++k) {
      int v_ = ((int)(floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_softmax_expsum[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] + max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[(((((int)blockIdx.x) * 170) + (((int)threadIdx.x) * 17)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))])));
  }
}

