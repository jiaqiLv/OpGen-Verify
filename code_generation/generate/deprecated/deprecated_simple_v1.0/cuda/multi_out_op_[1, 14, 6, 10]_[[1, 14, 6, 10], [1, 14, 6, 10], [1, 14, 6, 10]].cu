
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 105) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(21) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))]));
}

