
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 8; ++rv0) {
    for (int rv1 = 0; rv1 < 8; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) * 64)) + (rv0 * 8)) + rv1)]);
    }
  }
}

