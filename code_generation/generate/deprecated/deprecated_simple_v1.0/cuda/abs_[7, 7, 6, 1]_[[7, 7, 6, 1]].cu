
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) < 147) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = fabsf(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

