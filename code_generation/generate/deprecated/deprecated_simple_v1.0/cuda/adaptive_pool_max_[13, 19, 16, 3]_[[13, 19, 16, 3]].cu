
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 2; ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 3) + 3) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 3) + 3) >> 3) : (((((((int)threadIdx.x) & 7) * 3) + 3) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 3) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) * 48) + ((((int)threadIdx.x) >> 3) * 6)) + (rv0 * 3)) + (((((int)threadIdx.x) & 7) * 3) >> 3)) + rv1)]);
    }
  }
}

