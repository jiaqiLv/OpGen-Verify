
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 2475) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acosf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}
