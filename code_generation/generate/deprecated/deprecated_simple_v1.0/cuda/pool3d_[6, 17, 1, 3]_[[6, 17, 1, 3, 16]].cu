
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      for (int rv2 = 0; rv2 < 3; ++rv2) {
        pool_max[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))], ((((((1 <= rv0) && (rv0 < 2)) && (1 <= (((((((int)threadIdx.x) >> 3) + ((int)blockIdx.x)) & 1) * 2) + rv1))) && (((rv1 >> 1) + (((((int)threadIdx.x) >> 3) + ((int)blockIdx.x)) & 1)) < 2)) && (1 <= (((((int)threadIdx.x) & 7) * 2) + rv2))) ? data[((((((((((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 3)) >> 1) * 48) + (rv0 * 48)) + ((((((int)threadIdx.x) >> 3) + ((int)blockIdx.x)) & 1) * 32)) + (rv1 * 16)) + ((((int)threadIdx.x) & 7) * 2)) + rv2) - 65)] : -3.402823e+38f));
      }
    }
  }
}

