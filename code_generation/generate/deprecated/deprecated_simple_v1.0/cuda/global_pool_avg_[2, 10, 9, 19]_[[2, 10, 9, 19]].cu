
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(20) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] * 5.847953e-03f);
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)threadIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 9; ++rv0) {
    for (int rv1 = 0; rv1 < 19; ++rv1) {
      adaptive_pool_sum[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] + data[(((((int)threadIdx.x) * 171) + (rv0 * 19)) + rv1)]);
    }
  }
}
