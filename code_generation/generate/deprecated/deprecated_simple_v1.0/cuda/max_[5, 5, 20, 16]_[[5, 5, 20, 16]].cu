
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ data, float* __restrict__ data_red) {
  data_red[0] = -3.402823e+38f;
  for (int k0 = 0; k0 < 5; ++k0) {
    for (int k1 = 0; k1 < 5; ++k1) {
      for (int k2 = 0; k2 < 20; ++k2) {
        for (int k3 = 0; k3 < 16; ++k3) {
          data_red[0] = max(data_red[0], data[((((k0 * 1600) + (k1 * 320)) + (k2 * 16)) + k3)]);
        }
      }
    }
  }
}

