
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 6669) {
    T_cast[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 702) / 39)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 351) * 39) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 39))] * 5.555556e-02f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 20) + ((int)threadIdx.x)) < 741) {
    T_multiply_red[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 18; ++k1) {
    if (((((int)blockIdx.x) * 20) + ((int)threadIdx.x)) < 741) {
      T_multiply_red[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 20) + ((int)threadIdx.x)) / 39) * 702) + (k1 * 39)) + (((((int)blockIdx.x) * 20) + ((int)threadIdx.x)) % 39))] * data[((((((((int)blockIdx.x) * 20) + ((int)threadIdx.x)) / 39) * 702) + (k1 * 39)) + (((((int)blockIdx.x) * 20) + ((int)threadIdx.x)) % 39))]));
    }
  }
}

