
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[16];
  __shared__ float left_matrix_shared[72];
  __shared__ float right_matrix_shared[36];
  for (int ax0_c_outer_inner_init = 0; ax0_c_outer_inner_init < 2; ++ax0_c_outer_inner_init) {
    for (int ax1_c_outer_inner_init = 0; ax1_c_outer_inner_init < 2; ++ax1_c_outer_inner_init) {
      for (int ax1_c_inner_init = 0; ax1_c_inner_init < 2; ++ax1_c_inner_init) {
        T_matmul_local[(((ax0_c_outer_inner_init * 4) + (ax1_c_outer_inner_init * 2)) + ax1_c_inner_init)] = 0.000000e+00f;
        T_matmul_local[((((ax0_c_outer_inner_init * 4) + (ax1_c_outer_inner_init * 2)) + ax1_c_inner_init) + 8)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 12; ++ax0_ax1_fused_outer_outer) {
    *(float3*)(left_matrix_shared + ((ax0_ax1_fused_outer_outer * 6) + (((int)threadIdx.x) * 3))) = *(float3*)(left_matrix + ((ax0_ax1_fused_outer_outer * 6) + (((int)threadIdx.x) * 3)));
  }
  for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 18; ++ax0_ax1_fused_outer_outer_1) {
    right_matrix_shared[((ax0_ax1_fused_outer_outer_1 * 2) + ((int)threadIdx.x))] = right_matrix[(((((ax0_ax1_fused_outer_outer_1 >> 1) * 8) + (((int)blockIdx.x) * 4)) + ((ax0_ax1_fused_outer_outer_1 & 1) * 2)) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 9; ++k_outer_inner) {
    for (int ax0_c_outer_inner = 0; ax0_c_outer_inner < 2; ++ax0_c_outer_inner) {
      for (int ax1_c_outer_inner = 0; ax1_c_outer_inner < 2; ++ax1_c_outer_inner) {
        for (int ax1_c_inner = 0; ax1_c_inner < 2; ++ax1_c_inner) {
          T_matmul_local[(((ax0_c_outer_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner)] = (T_matmul_local[(((ax0_c_outer_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner)] + (left_matrix_shared[(((((int)threadIdx.x) * 18) + (ax0_c_outer_inner * 9)) + k_outer_inner)] * right_matrix_shared[(((k_outer_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner)]));
          T_matmul_local[((((ax0_c_outer_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner) + 8)] = (T_matmul_local[((((ax0_c_outer_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner) + 8)] + (left_matrix_shared[((((((int)threadIdx.x) * 18) + (ax0_c_outer_inner * 9)) + k_outer_inner) + 36)] * right_matrix_shared[(((k_outer_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner)]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      T_matmul[((((((int)threadIdx.x) * 16) + (ax0_inner * 8)) + (((int)blockIdx.x) * 4)) + ax1_inner)] = T_matmul_local[((ax0_inner * 4) + ax1_inner)];
      T_matmul[(((((((int)threadIdx.x) * 16) + (ax0_inner * 8)) + (((int)blockIdx.x) * 4)) + ax1_inner) + 32)] = T_matmul_local[(((ax0_inner * 4) + ax1_inner) + 8)];
    }
  }
}

