
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(5) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] * 1.960784e-02f);
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 17; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 102) + (((int)threadIdx.x) * 51)) + (rv0 * 3)) + rv1)]);
    }
  }
}

