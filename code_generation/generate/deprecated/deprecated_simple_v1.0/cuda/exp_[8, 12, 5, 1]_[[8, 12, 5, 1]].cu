
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

