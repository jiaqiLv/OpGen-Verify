
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(45) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 45) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 45) + ((int)threadIdx.x))] - moving_mean[((((((int)blockIdx.x) % 88) * 9) + (((int)threadIdx.x) / 5)) / 44)]) / sqrtf((moving_var[((((((int)blockIdx.x) % 88) * 9) + (((int)threadIdx.x) / 5)) / 44)] + 1.000000e-05f)));
}

