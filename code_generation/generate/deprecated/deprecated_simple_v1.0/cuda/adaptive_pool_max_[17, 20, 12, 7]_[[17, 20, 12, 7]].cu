
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < ((((((((((int)threadIdx.x) >> 3) * 4) + 4) % 8) == 0) ? (((((int)blockIdx.x) & 1) * 6) + ((((((int)threadIdx.x) >> 3) * 3) + 3) >> 1)) : ((((((int)blockIdx.x) & 1) * 6) + ((((((int)threadIdx.x) >> 3) * 3) + 3) >> 1)) + 1)) - (((((int)threadIdx.x) >> 3) * 12) >> 3)) - ((((int)blockIdx.x) & 1) * 6)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 7) + 7) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 7) + 7) >> 3) : (((((((int)threadIdx.x) & 7) * 7) + 7) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 7) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) * 42) + ((((((int)threadIdx.x) >> 3) * 12) >> 3) * 7)) + (rv0 * 7)) + (((((int)threadIdx.x) & 7) * 7) >> 3)) + rv1)]);
    }
  }
}

