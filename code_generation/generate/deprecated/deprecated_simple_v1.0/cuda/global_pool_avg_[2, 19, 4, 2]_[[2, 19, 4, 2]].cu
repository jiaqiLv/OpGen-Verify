
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  if (((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 1)) < 19) {
    adaptive_pool_avg[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] * 1.250000e-01f);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 19) {
    adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rv0 = 0; rv0 < 4; ++rv0) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 19) {
        adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) * 8)) + (rv0 * 2)) + rv1)]);
      }
    }
  }
}

