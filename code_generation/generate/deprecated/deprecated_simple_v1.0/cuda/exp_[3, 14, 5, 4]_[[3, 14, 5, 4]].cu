
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 105) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = __expf(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

