
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(35) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 35) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 35) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) % 48) * 5) + (((int)threadIdx.x) / 7)) / 24)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((int)blockIdx.x) / 48) * 168) + (((((int)blockIdx.x) * 35) + ((int)threadIdx.x)) % 168))] * 1.000000e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 10; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) / 21) * 1680) + (k1 * 168)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 168))] * data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) / 21) * 1680) + (k1 * 168)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 168))]));
  }
}

