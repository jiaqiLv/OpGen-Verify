
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ output_unpack, float* __restrict__ packed_out) {
  output_unpack[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = packed_out[((((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) / 20) * 20)) + ((((int)threadIdx.x) % 10) * 2)) + ((((int)threadIdx.x) % 20) / 10))];
}

