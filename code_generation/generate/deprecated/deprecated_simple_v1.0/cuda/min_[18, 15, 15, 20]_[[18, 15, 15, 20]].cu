
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ data, float* __restrict__ data_red) {
  data_red[0] = 3.402823e+38f;
  for (int k0 = 0; k0 < 18; ++k0) {
    for (int k1 = 0; k1 < 15; ++k1) {
      for (int k2 = 0; k2 < 15; ++k2) {
        for (int k3 = 0; k3 < 20; ++k3) {
          data_red[0] = min(data_red[0], data[((((k0 * 4500) + (k1 * 300)) + (k2 * 20)) + k3)]);
        }
      }
    }
  }
}

