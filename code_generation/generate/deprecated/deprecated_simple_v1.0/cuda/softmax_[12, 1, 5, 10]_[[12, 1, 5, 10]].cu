
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_expsum[((int)blockIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 10; ++k) {
    T_softmax_expsum[((int)blockIdx.x)] = (T_softmax_expsum[((int)blockIdx.x)] + __expf((data[((((int)blockIdx.x) * 10) + k)] - T_softmax_maxelem[((int)blockIdx.x)])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 75) {
    T_softmax_norm[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 5)])) / T_softmax_expsum[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 5)]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 15) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 10; ++k) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 15) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 320) + (((int)threadIdx.x) * 10)) + k)]);
    }
  }
}

