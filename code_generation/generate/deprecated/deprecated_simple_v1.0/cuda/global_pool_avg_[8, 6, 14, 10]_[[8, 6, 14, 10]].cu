
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] * 7.142857e-03f);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 14; ++rv0) {
    for (int rv1 = 0; rv1 < 10; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 1680) + (((int)threadIdx.x) * 140)) + (rv0 * 10)) + rv1)]);
    }
  }
}

