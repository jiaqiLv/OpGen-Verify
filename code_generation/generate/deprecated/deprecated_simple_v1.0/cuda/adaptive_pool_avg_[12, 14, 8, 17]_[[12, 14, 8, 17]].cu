
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / ((float)((((((((int)threadIdx.x) & 7) + 1) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 17) + 17) >> 3) : (((((((int)threadIdx.x) & 7) * 17) + 17) >> 3) + 1)) - ((((int)threadIdx.x) & 7) * 2))));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv1 = 0; rv1 < ((((((((int)threadIdx.x) & 7) + 1) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 17) + 17) >> 3) : (((((((int)threadIdx.x) & 7) * 17) + 17) >> 3) + 1)) - ((((int)threadIdx.x) & 7) * 2)); ++rv1) {
    adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 136) + ((((int)threadIdx.x) >> 3) * 17)) + ((((int)threadIdx.x) & 7) * 2)) + rv1)]);
  }
}

