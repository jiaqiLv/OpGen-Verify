
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(21) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))] = asinf(data[((((int)blockIdx.x) * 21) + ((int)threadIdx.x))]);
}

