
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = __log10f(data[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))]);
}
