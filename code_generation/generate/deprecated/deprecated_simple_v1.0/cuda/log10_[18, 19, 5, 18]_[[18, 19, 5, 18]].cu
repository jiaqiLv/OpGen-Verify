
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 12)) < 2565) {
    compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = __log10f(data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
  }
}

