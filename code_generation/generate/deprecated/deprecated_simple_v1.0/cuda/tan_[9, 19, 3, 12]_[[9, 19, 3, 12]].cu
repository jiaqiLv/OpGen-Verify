
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(19) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = tanf(data[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))]);
}
