
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 9) {
    T_cast[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) % 18) / 9)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) / 9) * 36) + (((((((int)blockIdx.x) * 28) + ((int)threadIdx.x)) % 36) / 9) * 9)) + ((((int)blockIdx.x) + ((int)threadIdx.x)) % 9))] * 5.000000e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 2; ++k1) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
      T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) / 9) * 72) + (k1 * 36)) + (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 36))] * data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) / 9) * 72) + (k1 * 36)) + (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 36))]));
    }
  }
}

