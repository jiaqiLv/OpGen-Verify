
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 55) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 55) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
  }
}

