
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) < 2079) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = sqrtf(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

