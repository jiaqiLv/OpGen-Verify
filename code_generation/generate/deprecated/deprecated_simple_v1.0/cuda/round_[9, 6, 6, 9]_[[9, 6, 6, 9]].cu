
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 729) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = roundf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
