
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) / 6)) < 9945) {
    compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 29835) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

