
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 25) {
    T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / powf((2.000000e+00f + (1.000000e-04f * tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])), 7.500000e-01f));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 25) {
    tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 25) {
    tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (tensor[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

