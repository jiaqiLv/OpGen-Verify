
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ data, float* __restrict__ new_buffer) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 12)) < 405) {
    new_buffer[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))];
  }
}

