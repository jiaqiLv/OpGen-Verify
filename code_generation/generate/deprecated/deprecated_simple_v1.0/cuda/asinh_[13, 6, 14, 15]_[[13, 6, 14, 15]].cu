
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) / 12)) < 1365) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = asinhf(data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
  }
}
