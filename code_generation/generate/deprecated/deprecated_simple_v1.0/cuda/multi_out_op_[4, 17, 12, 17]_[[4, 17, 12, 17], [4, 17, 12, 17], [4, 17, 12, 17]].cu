
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]));
}

