
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] * weight[0]) * (1.000000e+00f / sqrtf((T_multiply_red[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 117) {
    T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 117) {
    T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + (data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
  }
}

