
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ data, float* __restrict__ data_red) {
  data_red[0] = 3.402823e+38f;
  for (int k1 = 0; k1 < 8; ++k1) {
    for (int k2 = 0; k2 < 6; ++k2) {
      for (int k3 = 0; k3 < 15; ++k3) {
        data_red[0] = min(data_red[0], data[(((k1 * 90) + (k2 * 15)) + k3)]);
      }
    }
  }
}

