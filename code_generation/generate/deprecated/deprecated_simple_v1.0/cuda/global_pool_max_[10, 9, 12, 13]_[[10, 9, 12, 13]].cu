
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 12; ++rv0) {
    for (int rv1 = 0; rv1 < 13; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 936) + (((int)threadIdx.x) * 156)) + (rv0 * 13)) + rv1)]);
    }
  }
}

