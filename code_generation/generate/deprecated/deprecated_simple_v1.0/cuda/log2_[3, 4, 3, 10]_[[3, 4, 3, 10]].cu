
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = __log2f(data[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]);
}

