
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 5355) {
    T_cast[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 765) / 255)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 765) * 255) + (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 255))] * 3.333333e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) < 1785) {
    T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 3; ++k1) {
    if (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) < 1785) {
      T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 255) * 765) + (k1 * 255)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 255))] * data[((((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 255) * 765) + (k1 * 255)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 255))]));
    }
  }
}

