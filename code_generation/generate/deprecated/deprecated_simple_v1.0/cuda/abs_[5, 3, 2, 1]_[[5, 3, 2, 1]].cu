
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((int)threadIdx.x)] = fabsf(data[((int)threadIdx.x)]);
}

