
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ resize) {
  resize[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) & 31) / 2))];
}

