
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ new_buffer) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 405) {
    new_buffer[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))];
  }
}

