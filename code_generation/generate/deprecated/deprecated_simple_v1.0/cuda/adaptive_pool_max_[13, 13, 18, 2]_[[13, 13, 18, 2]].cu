
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(26) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < ((((((((((((int)blockIdx.x) * 13) + (((int)threadIdx.x) >> 1)) & 31) >> 2) * 2) + 2) % 8) == 0) ? (((((((((int)blockIdx.x) * 13) + (((int)threadIdx.x) >> 1)) & 31) >> 2) * 9) + 9) >> 2) : ((((((((((int)blockIdx.x) * 13) + (((int)threadIdx.x) >> 1)) & 31) >> 2) * 9) + 9) >> 2) + 1)) - ((((((((int)blockIdx.x) * 13) + (((int)threadIdx.x) >> 1)) & 31) >> 2) * 18) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) & 7) * 2) + 2) % 8) == 0) ? (((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) & 7) + 1) >> 2) : ((((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) & 7) + 1) >> 2) + 1)) - ((((((int)threadIdx.x) >> 1) + ((int)blockIdx.x)) & 3) >> 1)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 13) + (((int)threadIdx.x) >> 1)) >> 5) * 36) + (((((((((int)blockIdx.x) * 13) + (((int)threadIdx.x) >> 1)) & 31) >> 2) * 18) >> 3) * 2)) + (rv0 * 2)) + ((((((int)threadIdx.x) >> 1) + ((int)blockIdx.x)) & 3) >> 1)) + rv1)]);
    }
  }
}

