
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = asinhf(data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

