
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      for (int rv2 = 0; rv2 < 3; ++rv2) {
        pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], (((((1 <= ((((((int)blockIdx.x) & 3) * 4) + ((((int)threadIdx.x) >> 3) * 2)) + rv0)) && (1 <= rv1)) && (rv1 < 2)) && (1 <= (((((int)threadIdx.x) & 7) * 2) + rv2))) ? data[(((((((((int)blockIdx.x) * 64) + ((((int)threadIdx.x) >> 3) * 32)) + (rv0 * 16)) + (rv1 * 16)) + ((((int)threadIdx.x) & 7) * 2)) + rv2) - 33)] : -3.402823e+38f));
      }
    }
  }
}

