
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) % 63) * 2) + (((int)threadIdx.x) / 30)) / 63) * 15) + (((int)threadIdx.x) % 15))]) + Shift[((((((((int)blockIdx.x) % 63) * 2) + (((int)threadIdx.x) / 30)) / 63) * 15) + (((int)threadIdx.x) % 15))]);
}

