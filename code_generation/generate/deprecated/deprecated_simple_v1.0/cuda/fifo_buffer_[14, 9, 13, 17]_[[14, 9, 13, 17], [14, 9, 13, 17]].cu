
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ data, float* __restrict__ new_buffer) {
  if (((((int)blockIdx.x) * 10) + (((int)threadIdx.x) / 6)) < 4641) {
    new_buffer[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))];
  }
}

