
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 10)) % 126) / 7)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) / 20)) / 63) * 70) + (((((int)blockIdx.x) * 40) + ((int)threadIdx.x)) % 70))] * 5.555556e-02f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 245) {
    T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 18; ++k1) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 245) {
      T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 35) * 1260) + (k1 * 70)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 70))] * data[((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 35) * 1260) + (k1 * 70)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 70))]));
    }
  }
}

