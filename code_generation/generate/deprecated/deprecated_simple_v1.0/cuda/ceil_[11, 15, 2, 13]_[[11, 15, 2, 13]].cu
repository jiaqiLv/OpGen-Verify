
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 2145) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

