
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(28) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 14) + (((int)threadIdx.x) >> 1)) < 675) {
    T_cast[((((int)blockIdx.x) * 28) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 28) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 14) + (((int)threadIdx.x) >> 1)) % 45) / 9)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 14) + (((int)threadIdx.x) >> 1)) / 45) * 18) + (((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 18))] * 2.000000e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 135) {
    T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 5; ++k1) {
    if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 135) {
      T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9) * 90) + (k1 * 18)) + (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 18))] * data[((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9) * 90) + (k1 * 18)) + (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 18))]));
    }
  }
}

