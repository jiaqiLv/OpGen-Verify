
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) / 6)) < 969) {
    compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = coshf(data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
  }
}

