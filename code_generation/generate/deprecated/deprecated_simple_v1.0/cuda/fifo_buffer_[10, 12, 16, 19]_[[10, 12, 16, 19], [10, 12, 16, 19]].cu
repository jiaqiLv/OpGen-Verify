
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ new_buffer) {
  new_buffer[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))];
}
