
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) < 1235) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __expf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}
