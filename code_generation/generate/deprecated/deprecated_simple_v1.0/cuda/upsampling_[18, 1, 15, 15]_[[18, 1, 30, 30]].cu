
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ data, float* __restrict__ resize) {
  resize[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = data[((((((int)blockIdx.x) / 60) * 900) + (((((int)blockIdx.x) % 60) / 2) * 30)) + (((int)threadIdx.x) / 2))];
}

