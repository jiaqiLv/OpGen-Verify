
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(20) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] * 4.444444e-03f);
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 15; ++rv0) {
    for (int rv1 = 0; rv1 < 15; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 4500) + (((int)threadIdx.x) * 225)) + (rv0 * 15)) + rv1)]);
    }
  }
}

