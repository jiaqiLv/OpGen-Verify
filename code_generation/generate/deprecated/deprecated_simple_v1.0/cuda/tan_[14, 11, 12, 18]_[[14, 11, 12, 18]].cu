
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(56) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = tanf(data[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))]);
}

