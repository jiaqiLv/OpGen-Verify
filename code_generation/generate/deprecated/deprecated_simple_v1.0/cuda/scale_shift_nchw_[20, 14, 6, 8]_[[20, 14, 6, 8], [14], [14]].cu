
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) % 42) / 3)]) + Shift[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) % 42) / 3)]);
}

