
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(3) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))] = sqrtf(data[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))]);
}

