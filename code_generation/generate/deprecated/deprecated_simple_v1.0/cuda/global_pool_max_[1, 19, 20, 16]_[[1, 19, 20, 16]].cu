
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(19) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((int)threadIdx.x)] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 20; ++rv0) {
    for (int rv1 = 0; rv1 < 16; ++rv1) {
      adaptive_pool_max[((int)threadIdx.x)] = max(adaptive_pool_max[((int)threadIdx.x)], data[(((((int)threadIdx.x) * 320) + (rv0 * 16)) + rv1)]);
    }
  }
}

