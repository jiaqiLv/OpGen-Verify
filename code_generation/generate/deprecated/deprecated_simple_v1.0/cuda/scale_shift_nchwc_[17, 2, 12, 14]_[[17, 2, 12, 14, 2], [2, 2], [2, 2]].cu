
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(42) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 42) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 42) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) & 15) >> 3) * 2) + (((int)threadIdx.x) & 1))]) + Shift[((((((int)blockIdx.x) & 15) >> 3) * 2) + (((int)threadIdx.x) & 1))]);
}

