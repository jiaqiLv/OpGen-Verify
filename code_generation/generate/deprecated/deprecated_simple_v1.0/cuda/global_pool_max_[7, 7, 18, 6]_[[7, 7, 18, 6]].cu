
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 49) {
    adaptive_pool_max[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 18; ++rv0) {
    for (int rv1 = 0; rv1 < 6; ++rv1) {
      if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 49) {
        adaptive_pool_max[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 432) + (((int)threadIdx.x) * 108)) + (rv0 * 6)) + rv1)]);
      }
    }
  }
}

