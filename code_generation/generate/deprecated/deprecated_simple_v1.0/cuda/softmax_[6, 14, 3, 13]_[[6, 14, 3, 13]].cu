
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 63) {
    T_softmax_expsum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 13; ++k) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 63) {
      T_softmax_expsum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 208) + (((int)threadIdx.x) * 13)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 63) {
    T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 13; ++k) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 63) {
      T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 208) + (((int)threadIdx.x) * 13)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) < 819) {
    T_softmax_norm[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (__expf((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 13)])) / T_softmax_expsum[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 13)]);
  }
}
