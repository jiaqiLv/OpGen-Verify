
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ DilatedInput, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 117) {
    DilatedInput[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))];
  }
}

