
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 21) {
    adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 12; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 21) {
        adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 1152) + (((int)threadIdx.x) * 36)) + (rv0 * 3)) + rv1)]);
      }
    }
  }
}

