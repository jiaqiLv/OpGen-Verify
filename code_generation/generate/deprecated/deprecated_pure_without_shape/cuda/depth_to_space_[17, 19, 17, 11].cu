
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 3179) {
    depth_to_space[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[(((((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) / 187) * 3553) + ((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) % 374) / 11) % 2) * 1496)) + (((((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22) % 2) * 748)) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) % 748) / 187) * 187)) + ((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) % 374) / 11) / 2) * 11)) + ((((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22) / 2))];
  }
}

