
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((int)threadIdx.x) >> 3) * 2) + 2) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 5) + 5) >> 2) : (((((((int)threadIdx.x) >> 3) * 5) + 5) >> 2) + 1)) - (((((int)threadIdx.x) >> 3) * 10) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 5) + 5) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) : (((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 5) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) * 50) + ((((((int)threadIdx.x) >> 3) * 10) >> 3) * 5)) + (rv0 * 5)) + (((((int)threadIdx.x) & 7) * 5) >> 3)) + rv1)]);
    }
  }
}
