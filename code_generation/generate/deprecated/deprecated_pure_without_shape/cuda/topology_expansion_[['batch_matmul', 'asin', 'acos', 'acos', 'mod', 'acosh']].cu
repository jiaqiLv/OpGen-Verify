
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(180) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[2];
  __shared__ float ph_3_shared[180];
  for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
    T_batch_matmul_NN[i_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    ph_3_shared[((int)threadIdx.x)] = ph_3[((((((int)threadIdx.x) / 20) * 40) + (k_outer_outer * 20)) + (((int)threadIdx.x) % 20))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int i_inner = 0; i_inner < 2; ++i_inner) {
          T_batch_matmul_NN[i_inner] = (T_batch_matmul_NN[i_inner] + (ph_0[((((((((int)threadIdx.x) / 5) * 16) + (i_inner * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) / 20) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((int)threadIdx.x) / 5) * 10) + (i1_inner * 5)) + (((int)threadIdx.x) % 5))] = acoshf(T_batch_matmul_NN[i1_inner]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

