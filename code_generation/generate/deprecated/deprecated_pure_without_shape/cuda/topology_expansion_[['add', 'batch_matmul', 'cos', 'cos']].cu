
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  float T_batch_matmul_NN_local[9];
  __shared__ float ph_5_shared[1];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 3; ++i_c_outer_inner_init) {
    T_batch_matmul_NN_local[i_c_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(i_c_outer_inner_init + 3)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(i_c_outer_inner_init + 6)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    ph_5_shared[0] = ph_5[((k_outer_outer * 2) + ((int)blockIdx.x))];
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 3; ++i_c_outer_inner) {
      T_batch_matmul_NN_local[i_c_outer_inner] = (T_batch_matmul_NN_local[i_c_outer_inner] + (ph_0[((i_c_outer_inner * 8) + k_outer_outer)] * ph_5_shared[0]));
      T_batch_matmul_NN_local[(i_c_outer_inner + 3)] = (T_batch_matmul_NN_local[(i_c_outer_inner + 3)] + (ph_0[(((i_c_outer_inner * 8) + k_outer_outer) + 24)] * ph_5_shared[0]));
      T_batch_matmul_NN_local[(i_c_outer_inner + 6)] = (T_batch_matmul_NN_local[(i_c_outer_inner + 6)] + (ph_0[(((i_c_outer_inner * 8) + k_outer_outer) + 48)] * ph_5_shared[0]));
    }
  }
  for (int i_inner = 0; i_inner < 3; ++i_inner) {
    T_batch_matmul_NN[((i_inner * 2) + ((int)blockIdx.x))] = T_batch_matmul_NN_local[i_inner];
    T_batch_matmul_NN[(((i_inner * 2) + ((int)blockIdx.x)) + 6)] = T_batch_matmul_NN_local[(i_inner + 3)];
    T_batch_matmul_NN[(((i_inner * 2) + ((int)blockIdx.x)) + 12)] = T_batch_matmul_NN_local[(i_inner + 6)];
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

