
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ data, float* __restrict__ data_red) {
  data_red[0] = -3.402823e+38f;
  for (int k0 = 0; k0 < 7; ++k0) {
    for (int k1 = 0; k1 < 20; ++k1) {
      for (int k2 = 0; k2 < 9; ++k2) {
        data_red[0] = max(data_red[0], data[(((k0 * 180) + (k1 * 9)) + k2)]);
      }
    }
  }
}

