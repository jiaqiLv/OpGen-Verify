
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = atanhf(atanf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}
