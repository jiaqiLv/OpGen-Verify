
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 30) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 30) + ((int)threadIdx.x))] - moving_mean[((((((int)blockIdx.x) & 7) * 3) + (((int)threadIdx.x) / 10)) >> 1)]) / sqrtf((moving_var[((((((int)blockIdx.x) & 7) * 3) + (((int)threadIdx.x) / 10)) >> 1)] + 1.000000e-05f)));
}

