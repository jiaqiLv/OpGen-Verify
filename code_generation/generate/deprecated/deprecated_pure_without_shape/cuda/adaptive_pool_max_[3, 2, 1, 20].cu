
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((int)blockIdx.x) & 1) * 4) + (((int)threadIdx.x) >> 3)) + 1) % 8) == 0) ? ((((((int)threadIdx.x) + 8) >> 5) + (((int)blockIdx.x) & 1)) >> 1) : (((((((int)threadIdx.x) + 8) >> 5) + (((int)blockIdx.x) & 1)) >> 1) + 1)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 4) + 4) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 5) + 5) >> 1) : (((((((int)threadIdx.x) & 7) * 5) + 5) >> 1) + 1)) - (((((int)threadIdx.x) & 7) * 20) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) >> 1) * 20) + (rv0 * 20)) + (((((int)threadIdx.x) & 7) * 20) >> 3)) + rv1)]);
    }
  }
}

