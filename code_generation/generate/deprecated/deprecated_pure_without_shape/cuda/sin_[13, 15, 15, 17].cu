
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(62) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 62) + ((int)threadIdx.x)) < 49725) {
    compute[((((int)blockIdx.x) * 62) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 62) + ((int)threadIdx.x))]);
  }
}

