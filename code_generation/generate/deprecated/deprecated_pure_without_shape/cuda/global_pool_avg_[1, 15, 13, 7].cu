
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(15) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] * 1.098901e-02f);
}

extern "C" __global__ void __launch_bounds__(15) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)threadIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 13; ++rv0) {
    for (int rv1 = 0; rv1 < 7; ++rv1) {
      adaptive_pool_sum[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] + data[(((((int)threadIdx.x) * 91) + (rv0 * 7)) + rv1)]);
    }
  }
}

