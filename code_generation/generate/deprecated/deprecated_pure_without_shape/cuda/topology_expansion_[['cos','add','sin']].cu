
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(3) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))] = __sinf((ph_0[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))] + __cosf(ph_0[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))])));
}