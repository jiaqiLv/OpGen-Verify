
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(50) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] - moving_mean[((((int)blockIdx.x) % 12) >> 2)]) / sqrtf((moving_var[((((int)blockIdx.x) % 12) >> 2)] + 1.000000e-05f)));
}

