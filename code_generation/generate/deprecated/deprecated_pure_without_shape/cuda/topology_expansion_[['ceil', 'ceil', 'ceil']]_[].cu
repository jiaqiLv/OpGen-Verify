
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(36) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = ceilf(ceilf(ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

