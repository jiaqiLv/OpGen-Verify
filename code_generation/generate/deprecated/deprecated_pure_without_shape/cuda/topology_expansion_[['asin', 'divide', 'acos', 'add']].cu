
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + acosf((ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / asinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]))));
}
