
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = __cosf(data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
}

