
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(5) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] = __log2f(data[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))]);
}

