
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

