
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 6) + 6) % 8) == 0) ? ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 7) + 7) >> 2) : (((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 7) + 7) >> 2) + 1)) - (((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 14) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) >> 3) * 224) + ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 14) >> 3) * 16)) + (rv0 * 16)) + ((((int)threadIdx.x) & 7) * 2)) + rv1)]);
    }
  }
}

