
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 9) * 4) + (((int)threadIdx.x) / 15)) / 9)]) + Shift[((((((int)blockIdx.x) % 9) * 4) + (((int)threadIdx.x) / 15)) / 9)]);
}

