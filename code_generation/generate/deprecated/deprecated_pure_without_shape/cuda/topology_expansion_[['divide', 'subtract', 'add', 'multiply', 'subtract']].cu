
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(36) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] + (ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

