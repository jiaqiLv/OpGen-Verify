
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(13) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 13) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 9; ++rv0) {
    for (int rv1 = 0; rv1 < 7; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 13) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 13) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 819) + (((int)threadIdx.x) * 63)) + (rv0 * 7)) + rv1)]);
    }
  }
}

