
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(17) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] * 8.547009e-03f);
}

extern "C" __global__ void __launch_bounds__(17) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)threadIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 13; ++rv0) {
    for (int rv1 = 0; rv1 < 9; ++rv1) {
      adaptive_pool_sum[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] + data[(((((int)threadIdx.x) * 117) + (rv0 * 9)) + rv1)]);
    }
  }
}

