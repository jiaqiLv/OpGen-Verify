
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(25) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 25) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 25) + ((int)threadIdx.x))] * weight[((((int)blockIdx.x) & 3) >> 1)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((int)blockIdx.x) >> 2) * 50) + ((((int)blockIdx.x) & 1) * 25)) + ((int)threadIdx.x))] * 5.000000e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) < 425) {
    T_multiply_red[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 2; ++k1) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) < 425) {
      T_multiply_red[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) / 25) * 100) + (k1 * 50)) + (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 50))] * data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) / 25) * 100) + (k1 * 50)) + (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 50))]));
    }
  }
}

