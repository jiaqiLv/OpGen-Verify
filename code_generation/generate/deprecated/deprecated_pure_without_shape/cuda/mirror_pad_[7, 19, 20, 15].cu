
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ MirrorPadInput, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 207) {
    MirrorPadInput[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((189 <= ((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1))) ? (40 - (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)) : ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 9) ? 0 : ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9) - 1))) * 15) + (((((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 18) == 17) ? (31 - (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 18)) : (((((((int)blockIdx.x) * 7) + (((int)threadIdx.x) >> 1)) % 9) < 1) ? (1 - (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 18)) : ((((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 18) - 2))))];
  }
}

