
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 18) / 9) * 15) + (((int)threadIdx.x) % 15))]) + Shift[((((((int)blockIdx.x) % 18) / 9) * 15) + (((int)threadIdx.x) % 15))]);
}

