
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(9) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = tanf(data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))]);
}

