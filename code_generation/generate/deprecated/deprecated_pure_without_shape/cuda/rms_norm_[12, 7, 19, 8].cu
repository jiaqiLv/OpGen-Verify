
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) % 133) / 19)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) / 133) * 152) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) % 19) * 8)) + (((int)threadIdx.x) & 7))] * 1.428571e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(38) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 38) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 7; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 38) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 38) + ((int)threadIdx.x))] + (data[(((((((int)blockIdx.x) >> 2) * 1064) + (k1 * 152)) + ((((int)blockIdx.x) & 3) * 38)) + ((int)threadIdx.x))] * data[(((((((int)blockIdx.x) >> 2) * 1064) + (k1 * 152)) + ((((int)blockIdx.x) & 3) * 38)) + ((int)threadIdx.x))]));
  }
}

