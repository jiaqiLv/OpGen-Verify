
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 143) {
    depth_to_space[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[(((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) / 11) * 66) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) % 22) / 11) * 22)) + (((((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22) % 2) * 11)) + ((((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22) / 2))];
  }
}

