
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fmodf(__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __expf(__sinf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

