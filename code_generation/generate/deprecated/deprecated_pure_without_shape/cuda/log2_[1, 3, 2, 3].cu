
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((int)threadIdx.x)] = __log2f(data[((int)threadIdx.x)]);
}

