
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 8379) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

