
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(19) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 13; ++rv0) {
    for (int rv1 = 0; rv1 < 9; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 2223) + (((int)threadIdx.x) * 117)) + (rv0 * 9)) + rv1)]);
    }
  }
}

