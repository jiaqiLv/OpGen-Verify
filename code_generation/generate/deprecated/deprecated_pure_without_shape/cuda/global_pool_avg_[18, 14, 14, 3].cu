
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(28) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 28) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 28) + ((int)threadIdx.x))] * 2.380952e-02f);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 63) {
    adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rv0 = 0; rv0 < 14; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 63) {
        adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 672) + (((int)threadIdx.x) * 42)) + (rv0 * 3)) + rv1)]);
      }
    }
  }
}

