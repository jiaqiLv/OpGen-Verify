
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(10) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] * 1.315789e-02f);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 4; ++rv0) {
    for (int rv1 = 0; rv1 < 19; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 608) + (((int)threadIdx.x) * 76)) + (rv0 * 19)) + rv1)]);
    }
  }
}

