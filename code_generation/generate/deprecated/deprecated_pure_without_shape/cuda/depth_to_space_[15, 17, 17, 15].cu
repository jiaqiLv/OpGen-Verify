
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 3825) {
    depth_to_space[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = data[(((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) / 255) * 4335) + ((((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) % 510) / 15) % 2) * 2040)) + (((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 30) % 2) * 1020)) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) % 1020) / 255) * 255)) + ((((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) % 510) / 15) / 2) * 15)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 30) / 2))];
  }
}

