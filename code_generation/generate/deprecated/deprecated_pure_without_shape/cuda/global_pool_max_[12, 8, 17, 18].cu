
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 17; ++rv0) {
    for (int rv1 = 0; rv1 < 18; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 612) + (((int)threadIdx.x) * 306)) + (rv0 * 18)) + rv1)]);
    }
  }
}

