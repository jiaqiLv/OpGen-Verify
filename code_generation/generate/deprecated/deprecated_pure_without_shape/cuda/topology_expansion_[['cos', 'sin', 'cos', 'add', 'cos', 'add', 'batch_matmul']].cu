
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __cosf(__sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[1];
  __shared__ float ph_8_shared[40];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  if (((int)threadIdx.x) < 40) {
    ph_8_shared[((int)threadIdx.x)] = ph_8[((int)threadIdx.x)];
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 5; ++k_inner) {
    T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (__cosf(ph_0[((((int)threadIdx.x) * 5) + k_inner)]) * ph_8_shared[(((((int)threadIdx.x) >> 3) * 5) + k_inner)]));
  }
  T_batch_matmul_NN[((int)threadIdx.x)] = T_batch_matmul_NN_local[0];
}

