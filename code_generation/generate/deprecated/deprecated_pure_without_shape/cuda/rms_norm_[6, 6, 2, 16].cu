
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 9) + (((int)threadIdx.x) >> 1)) % 96) >> 4)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 3) + (((int)threadIdx.x) / 6)) >> 5) * 32) + (((((int)blockIdx.x) * 18) + ((int)threadIdx.x)) & 31))] * 1.666667e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 6; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + (data[(((((((int)blockIdx.x) >> 1) * 192) + (k1 * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((int)threadIdx.x))] * data[(((((((int)blockIdx.x) >> 1) * 192) + (k1 * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((int)threadIdx.x))]));
  }
}

