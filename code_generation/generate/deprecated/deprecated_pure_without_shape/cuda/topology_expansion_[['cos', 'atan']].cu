
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]);
}
