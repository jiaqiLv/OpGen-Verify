
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] * 7.936508e-03f);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 7; ++rv0) {
    for (int rv1 = 0; rv1 < 18; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 2016) + (((int)threadIdx.x) * 126)) + (rv0 * 18)) + rv1)]);
    }
  }
}

