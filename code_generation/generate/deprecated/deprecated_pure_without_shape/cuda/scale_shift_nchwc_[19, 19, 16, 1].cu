
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) % 19) * 2) + (((int)threadIdx.x) >> 4)) / 19) * 19) + (((((int)blockIdx.x) * 13) + ((int)threadIdx.x)) % 19))]) + Shift[((((((((int)blockIdx.x) % 19) * 2) + (((int)threadIdx.x) >> 4)) / 19) * 19) + (((((int)blockIdx.x) * 13) + ((int)threadIdx.x)) % 19))]);
}

