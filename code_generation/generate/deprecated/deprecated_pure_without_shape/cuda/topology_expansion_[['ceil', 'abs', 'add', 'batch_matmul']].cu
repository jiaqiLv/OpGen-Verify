
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (fabsf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(50) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  float T_batch_matmul_NN_local[4];
  __shared__ float ph_5_shared[80];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_outer_inner_init * 2) + b_c_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_outer_outer * 5) + (((int)threadIdx.x) / 10)) < 8) {
        ph_5_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 50) + ((int)threadIdx.x))] = ph_5[((((ax0_ax1_fused_ax2_fused_outer_outer * 100) + ((((int)threadIdx.x) / 10) * 20)) + (k_outer_outer * 10)) + (((int)threadIdx.x) % 10))];
      }
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
          T_batch_matmul_NN_local[((b_c_outer_inner * 2) + b_c_inner)] = (T_batch_matmul_NN_local[((b_c_outer_inner * 2) + b_c_inner)] + (ph_0[(((((((((int)threadIdx.x) / 25) * 80) + (b_c_outer_inner * 40)) + (b_c_inner * 20)) + (((((int)threadIdx.x) % 25) / 5) * 4)) + (k_outer_outer * 2)) + k_inner)] * ph_5_shared[((((((((int)threadIdx.x) / 25) * 40) + (b_c_outer_inner * 20)) + (b_c_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    T_batch_matmul_NN[((((((int)threadIdx.x) / 25) * 100) + (b_inner * 25)) + (((int)threadIdx.x) % 25))] = T_batch_matmul_NN_local[b_inner];
  }
}

