
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(56) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 28) + (((int)threadIdx.x) >> 1)) < 4845) {
    compute[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = asinhf(data[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))]);
  }
}

