
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(3) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 16; ++rv0) {
    for (int rv1 = 0; rv1 < 18; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 864) + (((int)threadIdx.x) * 288)) + (rv0 * 18)) + rv1)]);
    }
  }
}
