
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(38) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 38) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 38) + ((int)threadIdx.x))] - moving_mean[0]) / sqrtf((moving_var[0] + 1.000000e-05f)));
}

