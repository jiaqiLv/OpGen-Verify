
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 6)) < 4725) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))];
  }
}

