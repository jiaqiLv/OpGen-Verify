
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ conv1d_ncw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv1d_ncw_local[2];
  __shared__ float pad_temp_shared[6];
  __shared__ float kernel_shared[3];
  conv1d_ncw_local[0] = 0.000000e+00f;
  conv1d_ncw_local[1] = 0.000000e+00f;
  for (int ry_outer_outer = 0; ry_outer_outer < 3; ++ry_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 6; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      pad_temp_shared[ax0_ax1_fused_ax2_fused_outer_outer] = data[(((ax0_ax1_fused_ax2_fused_outer_outer * 10) + ((((int)blockIdx.x) & 3) * 2)) + ry_outer_outer)];
    }
    int3 v_ = make_int3(((((((int)blockIdx.x) >> 2) * 9) + ry_outer_outer))+(3*0), ((((((int)blockIdx.x) >> 2) * 9) + ry_outer_outer))+(3*1), ((((((int)blockIdx.x) >> 2) * 9) + ry_outer_outer))+(3*2));
    *(float3*)(kernel_shared + 0) = make_float3(kernel[v_.x],kernel[v_.y],kernel[v_.z]);
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
      conv1d_ncw_local[0] = (conv1d_ncw_local[0] + (pad_temp_shared[rc_inner] * kernel_shared[rc_inner]));
      conv1d_ncw_local[1] = (conv1d_ncw_local[1] + (pad_temp_shared[(rc_inner + 3)] * kernel_shared[rc_inner]));
    }
  }
  conv1d_ncw[((int)blockIdx.x)] = conv1d_ncw_local[0];
  conv1d_ncw[(((int)blockIdx.x) + 20)] = conv1d_ncw_local[1];
}

