
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 1615) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __logf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

