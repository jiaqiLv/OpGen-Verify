
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] * weight[(((((int)blockIdx.x) % 10) * 2) + (((int)threadIdx.x) / 20))]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((int)blockIdx.x) / 10) * 20) + (((int)threadIdx.x) % 20))] * 5.000000e-02f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(19) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 20; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 19) + ((int)threadIdx.x)) / 20) * 400) + (k1 * 20)) + (((((int)blockIdx.x) * 19) + ((int)threadIdx.x)) % 20))] * data[((((((((int)blockIdx.x) * 19) + ((int)threadIdx.x)) / 20) * 400) + (k1 * 20)) + (((((int)blockIdx.x) * 19) + ((int)threadIdx.x)) % 20))]));
  }
}

