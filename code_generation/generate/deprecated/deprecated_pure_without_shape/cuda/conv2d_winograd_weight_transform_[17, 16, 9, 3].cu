
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(238) default_function_kernel(float* __restrict__ data, float* __restrict__ transform_weight) {
  float transform_weight_local[28];
  __shared__ float data_shared[4896];
  __shared__ float G_shared[42];
  for (int eps_c_inner_init = 0; eps_c_inner_init < 7; ++eps_c_inner_init) {
    for (int ci_c_inner_init = 0; ci_c_inner_init < 4; ++ci_c_inner_init) {
      transform_weight_local[((eps_c_inner_init * 4) + ci_c_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer < 21; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 7) + (((int)threadIdx.x) / 34)) < 144) {
      data_shared[((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 238) + ((int)threadIdx.x))] = data[((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 119) + (((int)threadIdx.x) >> 1)) / 144) * 576) + (((int)blockIdx.x) * 288)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer * 238) + ((int)threadIdx.x)) % 288))];
    }
  }
  if (((int)threadIdx.x) < 42) {
    G_shared[((int)threadIdx.x)] = ((((int)threadIdx.x) == 41) ? 1.000000e+00f : ((((int)threadIdx.x) == 40) ? 0.000000e+00f : ((((int)threadIdx.x) == 39) ? 0.000000e+00f : ((((int)threadIdx.x) == 38) ? 0.000000e+00f : ((((int)threadIdx.x) == 37) ? 0.000000e+00f : ((((int)threadIdx.x) == 36) ? 0.000000e+00f : ((((int)threadIdx.x) == 35) ? 5.555556e-02f : ((((int)threadIdx.x) == 34) ? -1.111111e-01f : ((((int)threadIdx.x) == 33) ? 2.222222e-01f : ((((int)threadIdx.x) == 32) ? -4.444444e-01f : ((((int)threadIdx.x) == 31) ? 8.888889e-01f : ((((int)threadIdx.x) == 30) ? -1.777778e+00f : ((((int)threadIdx.x) == 29) ? 1.422222e+00f : ((((int)threadIdx.x) == 28) ? -7.111111e-01f : ((((int)threadIdx.x) == 27) ? 3.555556e-01f : ((((int)threadIdx.x) == 26) ? -1.777778e-01f : ((((int)threadIdx.x) == 25) ? 8.888889e-02f : ((((int)threadIdx.x) == 24) ? -4.444445e-02f : ((((int)threadIdx.x) == 23) ? -3.333334e-02f : ((((int)threadIdx.x) == 22) ? -6.666667e-02f : ((((int)threadIdx.x) == 21) ? -1.333333e-01f : ((((int)threadIdx.x) == 20) ? -2.666667e-01f : ((((int)threadIdx.x) == 19) ? -5.333334e-01f : ((((int)threadIdx.x) == 18) ? -1.066667e+00f : ((((int)threadIdx.x) == 17) ? 2.222222e-01f : ((((int)threadIdx.x) == 16) ? 2.222222e-01f : ((((int)threadIdx.x) == 15) ? 2.222222e-01f : ((((int)threadIdx.x) == 14) ? 2.222222e-01f : ((((int)threadIdx.x) == 13) ? 2.222222e-01f : ((((int)threadIdx.x) == 12) ? 2.222222e-01f : ((((int)threadIdx.x) == 11) ? -6.666667e-01f : ((((int)threadIdx.x) == 10) ? 6.666667e-01f : ((((int)threadIdx.x) == 9) ? -6.666667e-01f : ((((int)threadIdx.x) == 8) ? 6.666667e-01f : ((((int)threadIdx.x) == 7) ? -6.666667e-01f : ((((int)threadIdx.x) == 6) ? 6.666667e-01f : ((((int)threadIdx.x) == 5) ? 0.000000e+00f : ((((int)threadIdx.x) == 4) ? 0.000000e+00f : ((((int)threadIdx.x) == 3) ? 0.000000e+00f : ((((int)threadIdx.x) == 2) ? 0.000000e+00f : ((((int)threadIdx.x) == 1) ? 0.000000e+00f : 2.000000e+00f)))))))))))))))))))))))))))))))))))))))));
  }
  __syncthreads();
  for (int r_kh_outer_inner = 0; r_kh_outer_inner < 2; ++r_kh_outer_inner) {
    for (int r_kh_inner = 0; r_kh_inner < 3; ++r_kh_inner) {
      for (int r_kw_inner = 0; r_kw_inner < 6; ++r_kw_inner) {
        for (int eps_c_inner = 0; eps_c_inner < 7; ++eps_c_inner) {
          for (int ci_c_inner = 0; ci_c_inner < 4; ++ci_c_inner) {
            transform_weight_local[((eps_c_inner * 4) + ci_c_inner)] = (transform_weight_local[((eps_c_inner * 4) + ci_c_inner)] + ((data_shared[((((((((int)threadIdx.x) % 34) * 144) + (ci_c_inner * 36)) + (r_kh_outer_inner * 18)) + (r_kh_inner * 6)) + r_kw_inner)] * G_shared[(((eps_c_inner * 6) + (r_kh_outer_inner * 3)) + r_kh_inner)]) * G_shared[(((((int)threadIdx.x) / 34) * 6) + r_kw_inner)]));
          }
        }
      }
    }
  }
  for (int eps_inner = 0; eps_inner < 7; ++eps_inner) {
    for (int ci_inner = 0; ci_inner < 4; ++ci_inner) {
      transform_weight[(((((eps_inner * 1904) + ((((int)threadIdx.x) >> 1) * 16)) + (((int)blockIdx.x) * 8)) + ((((int)threadIdx.x) & 1) * 4)) + ci_inner)] = transform_weight_local[((eps_inner * 4) + ci_inner)];
    }
  }
}

