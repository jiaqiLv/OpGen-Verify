
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ new_buffer) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9025) {
    new_buffer[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))];
  }
}

