
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}
