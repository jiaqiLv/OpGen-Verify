
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_add, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 247) {
    T_add[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (sqrtf(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]) + __cosf(data_1[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
  }
}

