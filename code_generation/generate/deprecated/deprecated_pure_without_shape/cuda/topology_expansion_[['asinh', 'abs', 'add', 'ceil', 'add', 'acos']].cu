
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = (fabsf(ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

