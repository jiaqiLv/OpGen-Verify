
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_5(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_6(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acoshf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) + fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fabsf(asinhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

