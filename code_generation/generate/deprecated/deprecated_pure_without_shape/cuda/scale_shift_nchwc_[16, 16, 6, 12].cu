
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 48) / 24) * 16) + (((int)threadIdx.x) & 15))]) + Shift[((((((int)blockIdx.x) % 48) / 24) * 16) + (((int)threadIdx.x) & 15))]);
}

