
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 50) / 25)]) + Shift[((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 50) / 25)]);
}

