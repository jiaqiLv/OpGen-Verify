
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(27) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 27) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < ((((((((((((int)blockIdx.x) * 27) + ((int)threadIdx.x)) & 63) >> 3) * 5) + 5) % 8) == 0) ? (((((((((int)blockIdx.x) * 27) + ((int)threadIdx.x)) & 63) >> 3) * 13) + 13) >> 3) : ((((((((((int)blockIdx.x) * 27) + ((int)threadIdx.x)) & 63) >> 3) * 13) + 13) >> 3) + 1)) - ((((((((int)blockIdx.x) * 27) + ((int)threadIdx.x)) & 63) >> 3) * 13) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) & 7) * 5) + 5) % 8) == 0) ? ((((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) & 7) * 13) + 13) >> 3) : (((((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) & 7) * 13) + 13) >> 3) + 1)) - (((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) & 7) * 13) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 27) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 27) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 27) + ((int)threadIdx.x)) >> 6) * 169) + (((((((((int)blockIdx.x) * 27) + ((int)threadIdx.x)) & 63) >> 3) * 13) >> 3) * 13)) + (rv0 * 13)) + (((((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) & 7) * 13) >> 3)) + rv1)]);
    }
  }
}

