
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  if (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) < 99) {
    adaptive_pool_avg[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * 7.575758e-03f);
  }
}

extern "C" __global__ void __launch_bounds__(9) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 11; ++rv0) {
    for (int rv1 = 0; rv1 < 12; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 1188) + (((int)threadIdx.x) * 132)) + (rv0 * 12)) + rv1)]);
    }
  }
}

