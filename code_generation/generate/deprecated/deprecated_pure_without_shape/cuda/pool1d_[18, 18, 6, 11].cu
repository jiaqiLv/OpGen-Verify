
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 243) {
    pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 243) {
      pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], (((1 <= (((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 6) * 2) + rv0)) && (((rv0 >> 1) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 6)) < 6)) ? data[(((((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) / 3) * 11) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 6) * 2)) + rv0) - 1)] : -3.402823e+38f));
    }
  }
}

