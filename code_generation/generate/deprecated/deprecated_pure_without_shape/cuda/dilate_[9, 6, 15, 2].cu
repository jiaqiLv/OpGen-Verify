
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ DilatedInput, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 405) {
    DilatedInput[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))];
  }
}

