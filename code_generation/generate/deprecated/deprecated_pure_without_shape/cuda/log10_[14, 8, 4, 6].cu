
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(56) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = __log10f(data[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))]);
}
