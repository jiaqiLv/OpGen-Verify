
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 3315) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))];
  }
}

