
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = (asinhf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
}
