
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(50) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[4];
  __shared__ float ph_3_shared[80];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    T_batch_matmul_NN_local[b_c_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_outer_inner_init + 2)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_outer_outer * 5) + (((int)threadIdx.x) / 10)) < 8) {
        ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 50) + ((int)threadIdx.x))] = ph_3[((((ax0_ax1_fused_ax2_fused_outer_outer * 100) + ((((int)threadIdx.x) / 10) * 20)) + (k_outer_outer * 10)) + (((int)threadIdx.x) % 10))];
      }
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_batch_matmul_NN_local[b_c_outer_inner] = (T_batch_matmul_NN_local[b_c_outer_inner] + (ph_0[((((((((int)threadIdx.x) / 25) * 40) + (b_c_outer_inner * 20)) + (((((int)threadIdx.x) % 25) / 5) * 4)) + (k_outer_outer * 2)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) / 25) * 20) + (b_c_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN_local[(b_c_outer_inner + 2)] = (T_batch_matmul_NN_local[(b_c_outer_inner + 2)] + (ph_0[(((((((((int)threadIdx.x) / 25) * 40) + (b_c_outer_inner * 20)) + (((((int)threadIdx.x) % 25) / 5) * 4)) + (k_outer_outer * 2)) + k_inner) + 80)] * ph_3_shared[((((((((int)threadIdx.x) / 25) * 20) + (b_c_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5)) + 40)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[((((((int)threadIdx.x) / 25) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 25))] = T_batch_matmul_NN_local[b_inner];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 25) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 25)) + 100)] = T_batch_matmul_NN_local[(b_inner + 2)];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

