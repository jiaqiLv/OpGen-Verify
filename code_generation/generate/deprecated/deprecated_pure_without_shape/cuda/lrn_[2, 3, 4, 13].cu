
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor) {
  T_divide[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] / powf((2.000000e+00f + (1.000000e-04f * tensor[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])), 7.500000e-01f));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor) {
  tensor[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = 0.000000e+00f;
  tensor[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (tensor[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + (data[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] * data[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

