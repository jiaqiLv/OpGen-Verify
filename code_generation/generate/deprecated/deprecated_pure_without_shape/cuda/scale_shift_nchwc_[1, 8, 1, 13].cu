
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) / 26) * 8) + ((((int)blockIdx.x) & 1) * 4)) + ((int)threadIdx.x))]) + Shift[((((((int)blockIdx.x) / 26) * 8) + ((((int)blockIdx.x) & 1) * 4)) + ((int)threadIdx.x))]);
}

