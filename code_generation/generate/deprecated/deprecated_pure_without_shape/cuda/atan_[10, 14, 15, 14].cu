
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 3675) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

