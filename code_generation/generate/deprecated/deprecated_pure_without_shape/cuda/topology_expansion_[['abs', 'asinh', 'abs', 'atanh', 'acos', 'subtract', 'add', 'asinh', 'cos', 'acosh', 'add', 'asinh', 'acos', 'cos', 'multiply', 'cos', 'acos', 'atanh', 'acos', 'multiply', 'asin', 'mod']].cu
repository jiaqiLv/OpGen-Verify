
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_13(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_7(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(acoshf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_11(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __cosf(fabsf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_8(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(acoshf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_10(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_14(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf((fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_6(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(acoshf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_5(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acoshf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) + fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_15(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf((fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_9(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acoshf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) * fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_12(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(fabsf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

