
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 15; ++k) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 120) + (((int)threadIdx.x) * 15)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 675) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 15)]) - __logf(compute_1[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 15)]));
  }
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int k = 0; k < 15; ++k) {
    T_softmax_maxelem[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 360) + (((int)threadIdx.x) * 15)) + k)]);
  }
}

