
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor) {
  T_divide[((int)threadIdx.x)] = (data[((int)threadIdx.x)] / powf((2.000000e+00f + (1.000000e-04f * tensor[((int)threadIdx.x)])), 7.500000e-01f));
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor) {
  tensor[((int)threadIdx.x)] = 0.000000e+00f;
  tensor[((int)threadIdx.x)] = (tensor[((int)threadIdx.x)] + (data[((int)threadIdx.x)] * data[((int)threadIdx.x)]));
}

