
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) / 6)) < 9) {
    compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = floorf(data[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
  }
}

