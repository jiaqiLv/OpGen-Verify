
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((int)threadIdx.x) >> 3) * 7) + 7) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 7) + 7) >> 3) : (((((((int)threadIdx.x) >> 3) * 7) + 7) >> 3) + 1)) - (((((int)threadIdx.x) >> 3) * 7) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 3) + 3) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 11) + 11) >> 3) : (((((((int)threadIdx.x) & 7) * 11) + 11) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 11) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) * 77) + ((((((int)threadIdx.x) >> 3) * 7) >> 3) * 11)) + (rv0 * 11)) + (((((int)threadIdx.x) & 7) * 11) >> 3)) + rv1)]);
    }
  }
}

