
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(signed char* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 27) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = ((signed char)(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] != data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
  }
}

