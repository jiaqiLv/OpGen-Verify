
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(9) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 19; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 855) + (((int)threadIdx.x) * 95)) + (rv0 * 19)) + rv1)]);
    }
  }
}

