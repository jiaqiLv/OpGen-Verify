
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((int)blockIdx.x)] = acosf(data[((int)blockIdx.x)]);
}

