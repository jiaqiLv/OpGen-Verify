
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) < 3179) {
    compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = fabsf(data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
  }
}

