
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (0.000000e+00f - __expf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))]);
}

