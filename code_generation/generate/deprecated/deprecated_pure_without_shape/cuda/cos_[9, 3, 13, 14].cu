
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = __cosf(data[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]);
}

