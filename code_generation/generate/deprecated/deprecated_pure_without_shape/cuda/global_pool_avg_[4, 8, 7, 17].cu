
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] * 8.403361e-03f);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)threadIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 7; ++rv0) {
    for (int rv1 = 0; rv1 < 17; ++rv1) {
      adaptive_pool_sum[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] + data[(((((int)threadIdx.x) * 119) + (rv0 * 17)) + rv1)]);
    }
  }
}
