
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 2565) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

