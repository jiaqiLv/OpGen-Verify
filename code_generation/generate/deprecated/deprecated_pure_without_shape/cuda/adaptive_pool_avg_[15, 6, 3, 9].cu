
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / (((float)(((((((((int)threadIdx.x) >> 3) * 3) + 3) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 3) + 3) >> 3) : (((((((int)threadIdx.x) >> 3) * 3) + 3) >> 3) + 1)) - (((((int)threadIdx.x) >> 3) * 3) >> 3))) * ((float)((((((((int)threadIdx.x) & 7) + 1) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 9) + 9) >> 3) : (((((((int)threadIdx.x) & 7) * 9) + 9) >> 3) + 1)) - (((int)threadIdx.x) & 7)))));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < (((((((((int)threadIdx.x) >> 3) * 3) + 3) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 3) + 3) >> 3) : (((((((int)threadIdx.x) >> 3) * 3) + 3) >> 3) + 1)) - (((((int)threadIdx.x) >> 3) * 3) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < ((((((((int)threadIdx.x) & 7) + 1) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 9) + 9) >> 3) : (((((((int)threadIdx.x) & 7) * 9) + 9) >> 3) + 1)) - (((int)threadIdx.x) & 7)); ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + data[(((((((int)blockIdx.x) * 27) + ((((((int)threadIdx.x) >> 3) * 3) >> 3) * 9)) + (rv0 * 9)) + rv1) + (((int)threadIdx.x) & 7))]);
    }
  }
}
