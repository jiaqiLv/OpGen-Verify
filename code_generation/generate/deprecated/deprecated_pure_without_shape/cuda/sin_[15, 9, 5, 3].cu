
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) / 3)) < 675) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
  }
}

