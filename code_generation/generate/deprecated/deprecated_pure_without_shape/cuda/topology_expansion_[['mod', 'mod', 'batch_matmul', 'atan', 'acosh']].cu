
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ compute) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(T_batch_matmul_NN[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[24];
  __shared__ float T_mod_shared[192];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 3; ++b_c_outer_inner_init) {
    for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 8; ++j_c_outer_inner_init) {
      T_batch_matmul_NN_local[((b_c_outer_inner_init * 8) + j_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 96; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    T_mod_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x))] = fmodf(ph_0[((((((int)blockIdx.x) >> 2) * 192) + (ax0_ax1_fused_ax2_fused_outer_outer * 2)) + ((int)threadIdx.x))], ph_3[((((((int)blockIdx.x) >> 2) * 192) + (ax0_ax1_fused_ax2_fused_outer_outer * 2)) + ((int)threadIdx.x))]);
  }
  __syncthreads();
  for (int b_c_outer_inner = 0; b_c_outer_inner < 3; ++b_c_outer_inner) {
    for (int j_c_outer_inner = 0; j_c_outer_inner < 8; ++j_c_outer_inner) {
      for (int k_inner = 0; k_inner < 8; ++k_inner) {
        T_batch_matmul_NN_local[((b_c_outer_inner * 8) + j_c_outer_inner)] = (T_batch_matmul_NN_local[((b_c_outer_inner * 8) + j_c_outer_inner)] + (ph_0[((((((((int)blockIdx.x) >> 2) * 192) + (b_c_outer_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) * 8)) + k_inner)] * T_mod_shared[(((b_c_outer_inner * 64) + (k_inner * 8)) + j_c_outer_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 3; ++b_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_batch_matmul_NN[((((((((int)blockIdx.x) >> 2) * 192) + (b_inner * 64)) + ((((int)blockIdx.x) & 3) * 16)) + (((int)threadIdx.x) * 8)) + j_inner)] = T_batch_matmul_NN_local[((b_inner * 8) + j_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ compute) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(T_batch_matmul_NN[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

