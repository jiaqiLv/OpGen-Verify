
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 4725) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(15) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))]));
}
