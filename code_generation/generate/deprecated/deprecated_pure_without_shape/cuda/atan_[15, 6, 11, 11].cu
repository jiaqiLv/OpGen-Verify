
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 5445) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

