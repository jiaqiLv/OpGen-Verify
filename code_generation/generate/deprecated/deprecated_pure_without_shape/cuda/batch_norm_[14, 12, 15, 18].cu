
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] - moving_mean[((((((int)blockIdx.x) % 54) * 2) + (((int)threadIdx.x) / 30)) / 9)]) / sqrtf((moving_var[((((((int)blockIdx.x) % 54) * 2) + (((int)threadIdx.x) / 30)) / 9)] + 1.000000e-05f)));
}

