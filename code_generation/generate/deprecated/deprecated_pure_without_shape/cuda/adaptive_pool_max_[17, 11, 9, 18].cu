
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((((int)blockIdx.x) & 1) * 4) + (((int)threadIdx.x) >> 3)) + 1) % 8) == 0) ? ((((((int)blockIdx.x) & 1) * 9) + ((((((int)threadIdx.x) >> 3) * 9) + 9) >> 2)) >> 1) : (((((((int)blockIdx.x) & 1) * 9) + ((((((int)threadIdx.x) >> 3) * 9) + 9) >> 2)) >> 1) + 1)) - (((int)threadIdx.x) >> 3)) - ((((int)blockIdx.x) & 1) * 4)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 2) + 2) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 9) + 9) >> 2) : (((((((int)threadIdx.x) & 7) * 9) + 9) >> 2) + 1)) - (((((int)threadIdx.x) & 7) * 18) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((((((int)blockIdx.x) >> 1) * 162) + ((((int)blockIdx.x) & 1) * 72)) + ((((int)threadIdx.x) >> 3) * 18)) + (rv0 * 18)) + (((((int)threadIdx.x) & 7) * 18) >> 3)) + rv1)]);
    }
  }
}

