
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __cosf(acosf(ph_0[((int)blockIdx.x)]));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_6) {
  float T_batch_matmul_NN_local[20];
  __shared__ float ph_6_shared[160];
  for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
    T_batch_matmul_NN_local[b_c_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 2)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 4)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 6)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 8)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 10)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 12)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 14)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 16)] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 18)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_6_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 10) + ((int)threadIdx.x))] = ph_6[((ax0_ax1_fused_ax2_fused_outer_outer * 10) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
    for (int k_inner = 0; k_inner < 2; ++k_inner) {
      for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
        T_batch_matmul_NN_local[b_c_inner] = (T_batch_matmul_NN_local[b_c_inner] + (ph_0[(((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner)] * ph_6_shared[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN_local[(b_c_inner + 2)] = (T_batch_matmul_NN_local[(b_c_inner + 2)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 4)] * ph_6_shared[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN_local[(b_c_inner + 4)] = (T_batch_matmul_NN_local[(b_c_inner + 4)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 8)] * ph_6_shared[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN_local[(b_c_inner + 6)] = (T_batch_matmul_NN_local[(b_c_inner + 6)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 12)] * ph_6_shared[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN_local[(b_c_inner + 8)] = (T_batch_matmul_NN_local[(b_c_inner + 8)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 16)] * ph_6_shared[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN_local[(b_c_inner + 10)] = (T_batch_matmul_NN_local[(b_c_inner + 10)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 80)] * ph_6_shared[(((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5)) + 80)]));
        T_batch_matmul_NN_local[(b_c_inner + 12)] = (T_batch_matmul_NN_local[(b_c_inner + 12)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 84)] * ph_6_shared[(((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5)) + 80)]));
        T_batch_matmul_NN_local[(b_c_inner + 14)] = (T_batch_matmul_NN_local[(b_c_inner + 14)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 88)] * ph_6_shared[(((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5)) + 80)]));
        T_batch_matmul_NN_local[(b_c_inner + 16)] = (T_batch_matmul_NN_local[(b_c_inner + 16)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 92)] * ph_6_shared[(((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5)) + 80)]));
        T_batch_matmul_NN_local[(b_c_inner + 18)] = (T_batch_matmul_NN_local[(b_c_inner + 18)] + (ph_0[((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 2)) + k_inner) + 96)] * ph_6_shared[(((((((((int)threadIdx.x) / 5) * 40) + (b_c_inner * 20)) + (k_outer_inner * 10)) + (k_inner * 5)) + (((int)threadIdx.x) % 5)) + 80)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5))] = T_batch_matmul_NN_local[b_inner];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 5)] = T_batch_matmul_NN_local[(b_inner + 2)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 10)] = T_batch_matmul_NN_local[(b_inner + 4)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 15)] = T_batch_matmul_NN_local[(b_inner + 6)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 20)] = T_batch_matmul_NN_local[(b_inner + 8)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 100)] = T_batch_matmul_NN_local[(b_inner + 10)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 105)] = T_batch_matmul_NN_local[(b_inner + 12)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 110)] = T_batch_matmul_NN_local[(b_inner + 14)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 115)] = T_batch_matmul_NN_local[(b_inner + 16)];
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 5) * 50) + (b_inner * 25)) + (((int)threadIdx.x) % 5)) + 120)] = T_batch_matmul_NN_local[(b_inner + 18)];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

