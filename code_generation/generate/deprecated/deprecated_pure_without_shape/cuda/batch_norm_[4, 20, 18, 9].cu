
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] - moving_mean[((((((int)blockIdx.x) % 81) * 20) + (((int)threadIdx.x) >> 1)) / 81)]) / sqrtf((moving_var[((((((int)blockIdx.x) % 81) * 20) + (((int)threadIdx.x) >> 1)) / 81)] + 1.000000e-05f)));
}

