
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] * 2.380952e-02f);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)threadIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 14; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      adaptive_pool_sum[((int)threadIdx.x)] = (adaptive_pool_sum[((int)threadIdx.x)] + data[(((((int)threadIdx.x) * 42) + (rv0 * 3)) + rv1)]);
    }
  }
}
