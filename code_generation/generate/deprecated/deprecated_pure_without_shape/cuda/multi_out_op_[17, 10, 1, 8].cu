
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(34) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 34) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 34) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 34) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 85) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}
