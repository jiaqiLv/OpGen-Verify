
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 225) {
    T_cast[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 30) / 5)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) / 15) * 5) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 5))] * 1.666667e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) < 75) {
    T_multiply_red[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 6; ++k1) {
    if (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) < 75) {
      T_multiply_red[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 5) * 30) + (k1 * 5)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 5))] * data[((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 5) * 30) + (k1 * 5)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 5))]));
    }
  }
}

