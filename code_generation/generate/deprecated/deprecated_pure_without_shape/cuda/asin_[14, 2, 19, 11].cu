
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) < 1463) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinf(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

