
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4))]) - __logf(compute_1[((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 5) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 16; ++k) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 5) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 512) + (((int)threadIdx.x) * 16)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 5) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 16; ++k) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 5) {
      compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 1024) + (((int)threadIdx.x) * 16)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])));
    }
  }
}

