
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 432) / 27)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) / 27) * 27) + (((((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 27) / 3) * 3)) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 3))] * 6.250000e-02f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(54) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 16; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 54) + ((int)threadIdx.x))] + (data[((((((int)blockIdx.x) * 864) + ((((int)threadIdx.x) / 27) * 432)) + (k1 * 27)) + (((int)threadIdx.x) % 27))] * data[((((((int)blockIdx.x) * 864) + ((((int)threadIdx.x) / 27) * 432)) + (k1 * 27)) + (((int)threadIdx.x) % 27))]));
  }
}

