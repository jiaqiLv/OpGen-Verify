
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_7(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) / acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_10(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_6(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_9(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - __sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_13(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_11(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_8(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_12(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_5(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = acosf(__cosf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}
