
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < ((((((((((int)blockIdx.x) & 3) * 4) + ((((int)threadIdx.x) >> 3) * 2)) + 2) % 8) == 0) ? ((((((int)threadIdx.x) + 8) >> 4) + (((int)blockIdx.x) & 3)) >> 1) : (((((((int)threadIdx.x) + 8) >> 4) + (((int)blockIdx.x) & 3)) >> 1) + 1)) - ((((int)blockIdx.x) & 3) >> 1)); ++rv0) {
    adaptive_pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) >> 1) * 8) + (rv0 * 8)) + (((int)threadIdx.x) & 7))]);
  }
}

