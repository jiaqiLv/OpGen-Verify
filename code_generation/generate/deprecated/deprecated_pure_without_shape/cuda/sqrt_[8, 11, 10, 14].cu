
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = sqrtf(data[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))]);
}

