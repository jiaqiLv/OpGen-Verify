
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ new_buffer) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 1105) {
    new_buffer[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))];
  }
}

