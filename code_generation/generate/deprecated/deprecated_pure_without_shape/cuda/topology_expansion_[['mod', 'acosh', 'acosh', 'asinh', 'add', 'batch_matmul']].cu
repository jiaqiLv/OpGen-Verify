
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void default_function_kernel_2(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((int)blockIdx.x)] = (acoshf(ph_0[((int)blockIdx.x)]) + ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(25) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3, float* __restrict__ ph_9) {
  float T_batch_matmul_NN_local[8];
  __shared__ float T_mod_shared[40];
  __shared__ float ph_9_shared[40];
  for (int b_c_inner_init = 0; b_c_inner_init < 8; ++b_c_inner_init) {
    T_batch_matmul_NN_local[b_c_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      if (((ax0_ax1_fused_ax2_fused_outer_outer * 5) + (((int)threadIdx.x) / 5)) < 8) {
        T_mod_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 25) + ((int)threadIdx.x))] = fmodf(ph_0[(((ax0_ax1_fused_ax2_fused_outer_outer * 100) + (((int)threadIdx.x) * 4)) + k_outer_outer)], ph_3[(((ax0_ax1_fused_ax2_fused_outer_outer * 100) + (((int)threadIdx.x) * 4)) + k_outer_outer)]);
      }
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 2; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
      if (((ax0_ax1_fused_ax2_fused_outer_outer_1 * 5) + (((int)threadIdx.x) / 5)) < 8) {
        ph_9_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 25) + ((int)threadIdx.x))] = ph_9[((((ax0_ax1_fused_ax2_fused_outer_outer_1 * 100) + ((((int)threadIdx.x) / 5) * 20)) + (k_outer_outer * 5)) + (((int)threadIdx.x) % 5))];
      }
    }
    __syncthreads();
    for (int b_c_inner = 0; b_c_inner < 8; ++b_c_inner) {
      T_batch_matmul_NN_local[b_c_inner] = (T_batch_matmul_NN_local[b_c_inner] + (T_mod_shared[((b_c_inner * 5) + (((int)threadIdx.x) / 5))] * ph_9_shared[((b_c_inner * 5) + (((int)threadIdx.x) % 5))]));
    }
  }
  for (int b_inner = 0; b_inner < 8; ++b_inner) {
    T_batch_matmul_NN[((b_inner * 25) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[b_inner];
  }
}

