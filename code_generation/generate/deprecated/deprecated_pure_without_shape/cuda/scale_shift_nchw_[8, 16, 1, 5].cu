
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 5) * 16) + ((int)threadIdx.x)) / 5)]) + Shift[((((((int)blockIdx.x) % 5) * 16) + ((int)threadIdx.x)) / 5)]);
}
