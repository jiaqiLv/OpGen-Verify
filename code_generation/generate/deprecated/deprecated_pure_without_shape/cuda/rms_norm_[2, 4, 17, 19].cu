
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 323) {
    T_cast[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 1292) / 323)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) / 323) * 323) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 323))] * 2.500000e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(19) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 4; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] + (data[(((((((int)blockIdx.x) / 17) * 1292) + (k1 * 323)) + ((((int)blockIdx.x) % 17) * 19)) + ((int)threadIdx.x))] * data[(((((((int)blockIdx.x) / 17) * 1292) + (k1 * 323)) + ((((int)blockIdx.x) % 17) * 19)) + ((int)threadIdx.x))]));
  }
}

