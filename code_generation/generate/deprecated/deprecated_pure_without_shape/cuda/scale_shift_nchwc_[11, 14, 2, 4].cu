
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(11) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 11) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 11) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) * 11) + ((int)threadIdx.x)) % 224) / 112) * 14) + (((((int)blockIdx.x) * 11) + ((int)threadIdx.x)) % 14))]) + Shift[((((((((int)blockIdx.x) * 11) + ((int)threadIdx.x)) % 224) / 112) * 14) + (((((int)blockIdx.x) * 11) + ((int)threadIdx.x)) % 14))]);
}

