
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(36) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = (0.000000e+00f - acoshf(ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]));
}

extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __expf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

