
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 243) {
    pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 243) {
      pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], (((1 <= (((((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 9) * 2) + rv0)) && (((rv0 >> 1) + (((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 9)) < 9)) ? data[(((((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 9) * 17) + ((((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 9) * 2)) + rv0) - 1)] : -3.402823e+38f));
    }
  }
}

