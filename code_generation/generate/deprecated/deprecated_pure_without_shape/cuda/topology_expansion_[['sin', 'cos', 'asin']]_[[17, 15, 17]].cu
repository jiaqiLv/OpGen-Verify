
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinf(__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
}

