
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = asinhf(data[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

