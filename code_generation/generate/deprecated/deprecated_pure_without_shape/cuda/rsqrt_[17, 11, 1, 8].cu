
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 3)) < 187) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = (1.000000e+00f / sqrtf(data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]));
  }
}

