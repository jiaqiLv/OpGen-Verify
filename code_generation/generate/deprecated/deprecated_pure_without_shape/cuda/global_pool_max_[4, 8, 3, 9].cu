
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((int)threadIdx.x)] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 9; ++rv1) {
      adaptive_pool_max[((int)threadIdx.x)] = max(adaptive_pool_max[((int)threadIdx.x)], data[(((((int)threadIdx.x) * 27) + (rv0 * 9)) + rv1)]);
    }
  }
}

