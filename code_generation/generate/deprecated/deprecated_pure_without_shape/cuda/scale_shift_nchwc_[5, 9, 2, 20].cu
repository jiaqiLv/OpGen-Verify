
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(50) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) / 10)) % 72) / 36) * 9) + (((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 9))]) + Shift[((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) / 10)) % 72) / 36) * 9) + (((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) % 9))]);
}

