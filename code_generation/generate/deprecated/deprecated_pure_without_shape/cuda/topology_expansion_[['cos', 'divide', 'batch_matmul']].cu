
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[4];
  __shared__ float T_divide_shared[320];
  __shared__ float ph_4_shared[40];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    T_batch_matmul_NN_local[i_c_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(i_c_outer_inner_init + 2)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 20; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    T_divide_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 16) + ((int)threadIdx.x))] = (ph_0[((ax0_ax1_fused_ax2_fused_outer_outer * 16) + ((int)threadIdx.x))] / __cosf(ph_0[((ax0_ax1_fused_ax2_fused_outer_outer * 16) + ((int)threadIdx.x))]));
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 2; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    if (((ax0_ax1_fused_ax2_fused_outer_outer_1 * 4) + (((int)threadIdx.x) >> 2)) < 5) {
      *(float2*)(ph_4_shared + ((ax0_ax1_fused_ax2_fused_outer_outer_1 * 32) + (((int)threadIdx.x) * 2))) = *(float2*)(ph_4 + ((ax0_ax1_fused_ax2_fused_outer_outer_1 * 32) + (((int)threadIdx.x) * 2)));
    }
  }
  __syncthreads();
  for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
    for (int k_inner = 0; k_inner < 5; ++k_inner) {
      T_batch_matmul_NN_local[i_c_outer_inner] = (T_batch_matmul_NN_local[i_c_outer_inner] + (T_divide_shared[(((((((int)threadIdx.x) >> 1) * 40) + ((((int)threadIdx.x) & 1) * 10)) + (i_c_outer_inner * 5)) + k_inner)] * ph_4_shared[(((((int)threadIdx.x) >> 1) * 5) + k_inner)]));
      T_batch_matmul_NN_local[(i_c_outer_inner + 2)] = (T_batch_matmul_NN_local[(i_c_outer_inner + 2)] + (T_divide_shared[((((((((int)threadIdx.x) >> 1) * 40) + ((((int)threadIdx.x) & 1) * 10)) + (i_c_outer_inner * 5)) + k_inner) + 20)] * ph_4_shared[(((((int)threadIdx.x) >> 1) * 5) + k_inner)]));
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    T_batch_matmul_NN[((((((int)threadIdx.x) >> 1) * 8) + ((((int)threadIdx.x) & 1) * 2)) + i_inner)] = T_batch_matmul_NN_local[i_inner];
    T_batch_matmul_NN[(((((((int)threadIdx.x) >> 1) * 8) + ((((int)threadIdx.x) & 1) * 2)) + i_inner) + 4)] = T_batch_matmul_NN_local[(i_inner + 2)];
  }
}

