
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(56) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 7) + (((int)threadIdx.x) >> 3)) % 306) / 34)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 7) + (((int)threadIdx.x) >> 3)) / 306) * 272) + (((((int)blockIdx.x) * 56) + ((int)threadIdx.x)) % 272))] * 1.111111e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 9; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + (data[(((((((int)blockIdx.x) / 17) * 2448) + (k1 * 272)) + ((((int)blockIdx.x) % 17) * 16)) + ((int)threadIdx.x))] * data[(((((((int)blockIdx.x) / 17) * 2448) + (k1 * 272)) + ((((int)blockIdx.x) % 17) * 16)) + ((int)threadIdx.x))]));
  }
}

