
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 243) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = sinhf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
