
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[2];
  __shared__ float compute_shared[32];
  __shared__ float ph_4_shared[8];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    T_batch_matmul_NN_local[b_c_outer_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_inner_s < 10; ++ax0_ax1_fused_ax2_fused_inner_s) {
      if (((((int)threadIdx.x) * 5) + (ax0_ax1_fused_ax2_fused_inner_s >> 1)) < 16) {
        if (((int)threadIdx.x) < 4) {
          compute_shared[((((int)threadIdx.x) * 10) + ax0_ax1_fused_ax2_fused_inner_s)] = acosf(ph_0[(((((((((int)threadIdx.x) * 5) + (ax0_ax1_fused_ax2_fused_inner_s >> 1)) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_inner_s) & 3) * 5)) + k_outer_outer)]);
        }
      }
    }
    if (((int)threadIdx.x) < 8) {
      ph_4_shared[((int)threadIdx.x)] = ph_4[((((int)threadIdx.x) * 5) + k_outer_outer)];
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      T_batch_matmul_NN_local[b_c_outer_inner] = (T_batch_matmul_NN_local[b_c_outer_inner] + (compute_shared[((((((int)threadIdx.x) >> 2) * 8) + (b_c_outer_inner * 4)) + (((int)threadIdx.x) & 3))] * ph_4_shared[(((((int)threadIdx.x) >> 2) * 2) + b_c_outer_inner)]));
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[(((((((int)threadIdx.x) >> 2) * 16) + (b_inner * 8)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) & 3))] = T_batch_matmul_NN_local[b_inner];
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

