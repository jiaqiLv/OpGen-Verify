
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((int)threadIdx.x)] = __logf(data[((int)threadIdx.x)]);
}
