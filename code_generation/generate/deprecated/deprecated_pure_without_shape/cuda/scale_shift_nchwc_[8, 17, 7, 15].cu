
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 1785) {
    ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 3570) / 1785) * 17) + (((((int)blockIdx.x) * 13) + ((int)threadIdx.x)) % 17))]) + Shift[((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 3570) / 1785) * 17) + (((((int)blockIdx.x) * 13) + ((int)threadIdx.x)) % 17))]);
  }
}

