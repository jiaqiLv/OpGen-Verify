
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(45) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 45) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 45) + ((int)threadIdx.x))] - moving_mean[((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) / 9)) % 28) >> 2)]) / sqrtf((moving_var[((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) / 9)) % 28) >> 2)] + 1.000000e-05f)));
}

