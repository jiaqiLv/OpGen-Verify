
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ MirrorPadInput, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 253) {
    MirrorPadInput[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((231 <= ((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1))) ? (40 - (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 11)) : ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 11) ? 0 : ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 11) - 1))) * 19) + (((((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22) == 21) ? (39 - (((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22)) : (((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 1)) % 11) < 1) ? (1 - (((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22)) : ((((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 22) - 2))))];
  }
}

