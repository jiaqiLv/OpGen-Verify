
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_expsum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 18; ++k) {
      int v_ = ((int)(floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_softmax_expsum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] + max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[(((((int)blockIdx.x) * 216) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 255) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 18; ++k) {
    if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 255) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 576) + (((int)threadIdx.x) * 18)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 2295) {
      int v_ = ((int)(floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_softmax_norm[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)])) / T_softmax_expsum[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]);
  }
}

