
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = 0.000000e+00f;
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + __expf((data[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]) - __logf(compute_1[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]));
}

