
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv1 = 0; rv1 < ((((((((int)threadIdx.x) & 7) + 1) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 9) + 9) >> 3) : (((((((int)threadIdx.x) & 7) * 9) + 9) >> 3) + 1)) - (((int)threadIdx.x) & 7)); ++rv1) {
    adaptive_pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 18) + ((((int)threadIdx.x) >> 3) * 9)) + rv1) + (((int)threadIdx.x) & 7))]);
  }
}

