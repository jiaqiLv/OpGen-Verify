
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(19) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 19) + ((int)threadIdx.x))] * 6.250000e-03f);
}

extern "C" __global__ void default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((int)blockIdx.x)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 8; ++rv0) {
    for (int rv1 = 0; rv1 < 20; ++rv1) {
      adaptive_pool_sum[((int)blockIdx.x)] = (adaptive_pool_sum[((int)blockIdx.x)] + data[(((((int)blockIdx.x) * 160) + (rv0 * 20)) + rv1)]);
    }
  }
}

