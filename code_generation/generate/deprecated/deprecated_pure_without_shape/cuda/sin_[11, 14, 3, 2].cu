
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 231) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
  }
}
