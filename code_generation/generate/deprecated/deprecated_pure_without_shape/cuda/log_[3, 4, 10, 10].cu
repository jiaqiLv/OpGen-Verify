
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = __logf(data[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]);
}

