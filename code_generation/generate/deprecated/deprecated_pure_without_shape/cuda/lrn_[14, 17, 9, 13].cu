
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 13923) {
    T_divide[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / powf((2.000000e+00f + (1.000000e-04f * tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])), 7.500000e-01f));
  }
}

extern "C" __global__ void __launch_bounds__(51) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor) {
  tensor[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))] = 0.000000e+00f;
  tensor[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))] = (tensor[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))] + (data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))] * data[((((int)blockIdx.x) * 51) + ((int)threadIdx.x))]));
}

