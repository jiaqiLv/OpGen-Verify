
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(28) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 28) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 28) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) % 44) * 7) + (((int)threadIdx.x) >> 2)) / 22)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((int)blockIdx.x) / 44) * 88) + (((((int)blockIdx.x) * 28) + ((int)threadIdx.x)) % 88))] * 7.142857e-02f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(56) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 14; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 7) + (((int)threadIdx.x) >> 3)) / 11) * 1232) + (k1 * 88)) + (((((int)blockIdx.x) * 56) + ((int)threadIdx.x)) % 88))] * data[((((((((int)blockIdx.x) * 7) + (((int)threadIdx.x) >> 3)) / 11) * 1232) + (k1 * 88)) + (((((int)blockIdx.x) * 56) + ((int)threadIdx.x)) % 88))]));
  }
}

