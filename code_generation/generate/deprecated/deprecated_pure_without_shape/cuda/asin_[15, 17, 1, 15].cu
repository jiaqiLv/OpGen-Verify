
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 3825) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = asinf(data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
  }
}

