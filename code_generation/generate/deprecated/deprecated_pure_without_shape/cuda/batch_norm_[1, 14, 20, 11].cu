
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(55) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))] - moving_mean[(((int)blockIdx.x) >> 2)]) / sqrtf((moving_var[(((int)blockIdx.x) >> 2)] + 1.000000e-05f)));
}
