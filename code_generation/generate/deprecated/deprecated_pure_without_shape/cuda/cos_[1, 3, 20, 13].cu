
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) / 12)) < 65) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = __cosf(data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
  }
}

