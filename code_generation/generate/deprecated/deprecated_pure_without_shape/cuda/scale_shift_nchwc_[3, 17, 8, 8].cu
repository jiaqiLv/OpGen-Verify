
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 4)) % 136) / 68) * 17) + (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 17))]) + Shift[((((((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 4)) % 136) / 68) * 17) + (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) % 17))]);
}

