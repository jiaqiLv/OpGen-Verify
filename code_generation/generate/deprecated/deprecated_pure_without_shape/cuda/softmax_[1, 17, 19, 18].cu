
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) < 323) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 18; ++k) {
    if (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) < 323) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 576) + (((int)threadIdx.x) * 18)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 323) {
    T_softmax_expsum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 18; ++k) {
    if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 323) {
      T_softmax_expsum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 72) + (((int)threadIdx.x) * 18)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 2907) {
    T_softmax_norm[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)])) / T_softmax_expsum[(((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) / 9)]);
  }
}
