
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

