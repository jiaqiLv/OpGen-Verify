
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[5];
  __shared__ float left_matrix_shared[4];
  __shared__ float right_matrix_shared[20];
  for (int ax1_c_outer_inner_init = 0; ax1_c_outer_inner_init < 5; ++ax1_c_outer_inner_init) {
    T_matmul_local[ax1_c_outer_inner_init] = 0.000000e+00f;
  }
  *(float4*)(left_matrix_shared + 0) = *(float4*)(left_matrix + (((int)blockIdx.x) * 4));
  for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 20; ++ax0_ax1_fused_outer_outer) {
    right_matrix_shared[ax0_ax1_fused_outer_outer] = right_matrix[ax0_ax1_fused_outer_outer];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
    for (int ax1_c_outer_inner = 0; ax1_c_outer_inner < 5; ++ax1_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_matmul_local[ax1_c_outer_inner] = (T_matmul_local[ax1_c_outer_inner] + (left_matrix_shared[((k_outer_inner * 2) + k_inner)] * right_matrix_shared[(((k_outer_inner * 10) + (k_inner * 5)) + ax1_c_outer_inner)]));
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 5; ++ax1_inner) {
    T_matmul[((((int)blockIdx.x) * 5) + ax1_inner)] = T_matmul_local[ax1_inner];
  }
}

