
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) < 57) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = __log2f(data[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}
