
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((int)blockIdx.x)] = fmodf(ph_0[((int)blockIdx.x)], ph_3[((int)blockIdx.x)]);
}
