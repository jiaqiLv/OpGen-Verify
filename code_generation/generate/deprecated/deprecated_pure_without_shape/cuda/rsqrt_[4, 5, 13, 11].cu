
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) < 715) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = (1.000000e+00f / sqrtf(data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]));
  }
}

