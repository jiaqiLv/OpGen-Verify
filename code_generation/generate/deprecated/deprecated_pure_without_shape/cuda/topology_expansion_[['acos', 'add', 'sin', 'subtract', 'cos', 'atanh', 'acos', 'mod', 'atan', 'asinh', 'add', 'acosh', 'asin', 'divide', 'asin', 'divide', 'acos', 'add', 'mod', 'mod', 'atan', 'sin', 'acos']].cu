
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_12(float* __restrict__ T_add, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_16(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(fmodf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_11(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_7(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) / acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ T_add, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_15(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(fmodf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_10(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = atanf(acosf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_6(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(asinhf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_9(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - __sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_13(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_8(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinf(asinhf(acosf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_5(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(asinhf(acosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_14(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(fmodf(__sinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]))), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

