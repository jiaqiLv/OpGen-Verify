
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = fmodf(asinhf(ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

