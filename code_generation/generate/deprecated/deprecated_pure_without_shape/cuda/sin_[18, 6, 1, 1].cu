
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(36) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]);
}

