
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 117) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanf(data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
  }
}

