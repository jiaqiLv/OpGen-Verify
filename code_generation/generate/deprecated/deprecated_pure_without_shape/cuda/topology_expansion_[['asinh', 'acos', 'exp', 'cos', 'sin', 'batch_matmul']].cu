
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_7) {
  float T_batch_matmul_NN_local[16];
  __shared__ float compute_shared[64];
  __shared__ float ph_7_shared[8];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 4; ++b_c_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        T_batch_matmul_NN_local[(((b_c_inner_init * 4) + (i_c_outer_inner_init * 2)) + i_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 16; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))] = __sinf(ph_0[(((ax0_ax1_fused_ax2_fused_outer_outer * 20) + (((int)threadIdx.x) * 5)) + k_outer_outer)]);
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 2; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
      ph_7_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 4) + ((int)threadIdx.x))] = ph_7[(((ax0_ax1_fused_ax2_fused_outer_outer_1 * 20) + (((int)threadIdx.x) * 5)) + k_outer_outer)];
    }
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int b_c_inner = 0; b_c_inner < 4; ++b_c_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          T_batch_matmul_NN_local[(((b_c_inner * 4) + (i_c_outer_inner * 2)) + i_c_inner)] = (T_batch_matmul_NN_local[(((b_c_inner * 4) + (i_c_outer_inner * 2)) + i_c_inner)] + (compute_shared[((((((((int)threadIdx.x) >> 1) * 32) + (b_c_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + (i_c_outer_inner * 2)) + i_c_inner)] * ph_7_shared[(((((int)threadIdx.x) >> 1) * 4) + b_c_inner)]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 4; ++i_inner) {
      T_batch_matmul_NN[(((((((int)threadIdx.x) >> 1) * 32) + (b_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 4) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

