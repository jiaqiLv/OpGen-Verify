
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = acosf(__sinf(ph_0[((int)blockIdx.x)]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (__sinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

