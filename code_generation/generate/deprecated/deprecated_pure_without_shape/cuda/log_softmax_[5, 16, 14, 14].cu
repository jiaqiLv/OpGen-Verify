
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int k = 0; k < 14; ++k) {
    T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 112) + (((int)threadIdx.x) * 14)) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 14; ++k) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 448) + (((int)threadIdx.x) * 14)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(56) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 56) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 14))]) - __logf(compute_1[((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 14))]));
}

