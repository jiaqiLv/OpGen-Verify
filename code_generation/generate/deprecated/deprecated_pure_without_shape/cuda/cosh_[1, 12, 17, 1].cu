
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 51) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = coshf(data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
  }
}

