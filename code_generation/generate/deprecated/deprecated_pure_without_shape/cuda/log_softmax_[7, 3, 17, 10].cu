
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 357) {
    compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 10; ++k) {
    if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 357) {
      compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 40) + (((int)threadIdx.x) * 10)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(17) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 17) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 17) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 17) + ((int)threadIdx.x)) / 10)]) - __logf(compute_1[(((((int)blockIdx.x) * 17) + ((int)threadIdx.x)) / 10)]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) < 357) {
    T_softmax_maxelem[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 10; ++k) {
    if (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) < 357) {
      T_softmax_maxelem[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 20) + (((int)threadIdx.x) * 10)) + k)]);
    }
  }
}
