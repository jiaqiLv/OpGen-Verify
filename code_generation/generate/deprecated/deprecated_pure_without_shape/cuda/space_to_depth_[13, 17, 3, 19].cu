
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ space_to_depth) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) < 12597) {
    space_to_depth[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = data[(((((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) / 969) * 3876) + ((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 3876) / 57) % 17) * 228)) + (((((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 57) / 19) * 76)) + ((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 3876) / 57) / 34) * 38)) + ((((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 19) * 2)) + ((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 3876) / 57) % 34) / 17))];
  }
}

