
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < ((((((((((int)blockIdx.x) & 1) * 20) + ((((int)threadIdx.x) >> 3) * 5)) + 5) % 8) == 0) ? ((((((int)blockIdx.x) & 1) * 5) + ((((((int)threadIdx.x) >> 3) * 5) + 5) >> 2)) >> 1) : (((((((int)blockIdx.x) & 1) * 5) + ((((((int)threadIdx.x) >> 3) * 5) + 5) >> 2)) >> 1) + 1)) - ((((((int)blockIdx.x) & 1) * 5) + (((int)threadIdx.x) >> 3)) >> 1)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 5) + 5) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) : (((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 5) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[((((((((int)blockIdx.x) >> 1) * 25) + (((((((int)blockIdx.x) & 1) * 5) + (((int)threadIdx.x) >> 3)) >> 1) * 5)) + (rv0 * 5)) + (((((int)threadIdx.x) & 7) * 5) >> 3)) + rv1)]);
    }
  }
}

