
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 9) {
    adaptive_pool_avg[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * 8.333333e-02f);
  }
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv1 = 0; rv1 < 12; ++rv1) {
    adaptive_pool_sum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] + data[(((((int)blockIdx.x) * 144) + (((int)threadIdx.x) * 12)) + rv1)]);
  }
}

