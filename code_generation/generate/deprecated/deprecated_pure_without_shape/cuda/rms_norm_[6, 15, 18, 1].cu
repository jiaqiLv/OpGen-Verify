
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) < 405) {
    T_cast[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) % 135) / 9)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) / 135) * 18) + (((((int)blockIdx.x) * 10) + ((int)threadIdx.x)) % 18))] * 6.666667e-02f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 27) {
    T_multiply_red[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 15; ++k1) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 27) {
      T_multiply_red[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) / 9) * 270) + (k1 * 18)) + (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 18))] * data[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 1)) / 9) * 270) + (k1 * 18)) + (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) % 18))]));
    }
  }
}

