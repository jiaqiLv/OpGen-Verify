
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 221) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = roundf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

