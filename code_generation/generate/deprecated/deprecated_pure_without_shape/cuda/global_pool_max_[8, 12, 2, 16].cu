
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 2; ++rv0) {
    for (int rv1 = 0; rv1 < 16; ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) * 32)) + (rv0 * 16)) + rv1)]);
    }
  }
}

