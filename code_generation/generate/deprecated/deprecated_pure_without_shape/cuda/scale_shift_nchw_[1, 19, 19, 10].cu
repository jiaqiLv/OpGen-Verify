
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 6) + (((int)threadIdx.x) / 10)) < 361) {
    ScaleShift[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] * Scale[(((((int)blockIdx.x) * 6) + (((int)threadIdx.x) / 10)) / 19)]) + Shift[(((((int)blockIdx.x) * 6) + (((int)threadIdx.x) / 10)) / 19)]);
  }
}

