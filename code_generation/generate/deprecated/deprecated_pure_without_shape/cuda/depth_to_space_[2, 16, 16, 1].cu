
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space) {
  depth_to_space[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[((((((((int)blockIdx.x) >> 3) * 256) + (((((int)threadIdx.x) >> 1) % 2) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + ((((int)blockIdx.x) & 7) * 8)) + ((((int)threadIdx.x) >> 1) / 2))];
}

