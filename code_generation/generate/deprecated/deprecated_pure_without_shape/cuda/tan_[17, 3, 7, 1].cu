
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) < 357) {
    compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = tanf(data[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
  }
}

