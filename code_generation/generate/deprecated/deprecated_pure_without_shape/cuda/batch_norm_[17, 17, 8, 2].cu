
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 289) {
    T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - moving_mean[(((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) % 17)]) / sqrtf((moving_var[(((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) % 17)] + 1.000000e-05f)));
  }
}

