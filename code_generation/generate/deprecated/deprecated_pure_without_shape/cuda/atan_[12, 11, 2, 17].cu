
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(33) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 33) + ((int)threadIdx.x))] = atanf(data[((((int)blockIdx.x) * 33) + ((int)threadIdx.x))]);
}

