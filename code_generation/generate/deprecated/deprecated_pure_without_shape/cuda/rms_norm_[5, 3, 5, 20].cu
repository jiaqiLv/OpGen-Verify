
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 12)) < 125) {
    T_cast[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] * weight[((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 2)) % 75) / 25)]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 12)) / 25) * 100) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 100))] * 3.333333e-01f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 125) {
    T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k1 = 0; k1 < 3; ++k1) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 125) {
      T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) / 25) * 300) + (k1 * 100)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 100))] * data[((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) / 25) * 300) + (k1 * 100)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 100))]));
    }
  }
}

