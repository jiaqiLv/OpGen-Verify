
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(5) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) < 42) {
    adaptive_pool_max[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 14; ++rv0) {
    for (int rv1 = 0; rv1 < 17; ++rv1) {
      if (((((int)blockIdx.x) * 5) + ((int)threadIdx.x)) < 42) {
        adaptive_pool_max[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 1190) + (((int)threadIdx.x) * 238)) + (rv0 * 17)) + rv1)]);
      }
    }
  }
}

