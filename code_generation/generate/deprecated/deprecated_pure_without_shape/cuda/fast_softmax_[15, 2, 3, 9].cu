
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(9) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
    int v_ = ((int)(floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
  T_softmax_norm[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = (max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)])) / T_softmax_expsum[((int)blockIdx.x)]);
}

extern "C" __global__ void default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((int)blockIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 9; ++k) {
    T_softmax_maxelem[((int)blockIdx.x)] = max(T_softmax_maxelem[((int)blockIdx.x)], data[((((int)blockIdx.x) * 9) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 45) {
    T_softmax_expsum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 9; ++k) {
    if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 45) {
        int v_ = ((int)(floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
      T_softmax_expsum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), (data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
    }
  }
}

