
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((int)threadIdx.x) >> 3) * 2) + 2) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 9) + 9) >> 2) : (((((((int)threadIdx.x) >> 3) * 9) + 9) >> 2) + 1)) - (((((int)threadIdx.x) >> 3) * 18) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < ((((((((int)threadIdx.x) & 7) + 1) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 17) + 17) >> 3) : (((((((int)threadIdx.x) & 7) * 17) + 17) >> 3) + 1)) - ((((int)threadIdx.x) & 7) * 2)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], data[(((((((int)blockIdx.x) * 306) + ((((((int)threadIdx.x) >> 3) * 18) >> 3) * 17)) + (rv0 * 17)) + ((((int)threadIdx.x) & 7) * 2)) + rv1)]);
    }
  }
}

