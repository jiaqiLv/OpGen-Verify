
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) < 12375) {
    ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 2475) / 165)]) + Shift[((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 2475) / 165)]);
  }
}

