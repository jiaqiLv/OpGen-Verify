
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ data_red) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  normal_reduce_temp0[0] = 0.000000e+00f;
  for (int k0_k1_fused_k2_fused_k3_fused_outer = 0; k0_k1_fused_k2_fused_k3_fused_outer < 413; ++k0_k1_fused_k2_fused_k3_fused_outer) {
    if (((k0_k1_fused_k2_fused_k3_fused_outer * 2) + (((int)threadIdx.x) >> 4)) < 825) {
      normal_reduce_temp0[0] = (normal_reduce_temp0[0] + data[((k0_k1_fused_k2_fused_k3_fused_outer * 32) + ((int)threadIdx.x))]);
    }
  }
  uint mask[1];
  float t0[1];
  red_buf0[0] = normal_reduce_temp0[0];
  mask[0] = __activemask();
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 16, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 8, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 4, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 2, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  t0[0] = __shfl_down_sync(mask[0], red_buf0[0], 1, 32);
  red_buf0[0] = (red_buf0[0] + t0[0]);
  red_buf0[0] = __shfl_sync(mask[0], red_buf0[0], 0, 32);
  if (((int)threadIdx.x) == 0) {
    data_red[0] = red_buf0[0];
  }
}

