
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 459) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

