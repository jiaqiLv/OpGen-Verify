
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(9) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = fabsf(asinhf(ph_0[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

