
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / asinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = asinhf(asinhf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]));
}

