
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (__sinf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

