
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = atanhf(__expf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fabsf(atanhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

