
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = atanf(fmodf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))], (ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = ceilf(fmodf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

