
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel_3(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = (asinhf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_4(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (asinhf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

