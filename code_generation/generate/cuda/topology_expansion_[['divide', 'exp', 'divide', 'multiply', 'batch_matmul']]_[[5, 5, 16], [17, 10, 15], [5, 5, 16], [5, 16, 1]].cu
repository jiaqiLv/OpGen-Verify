
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[16];
  __shared__ float ph_8_shared[40];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 8; ++i_c_outer_inner_init) {
    T_batch_matmul_NN_local[i_c_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(i_c_outer_inner_init + 8)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 10; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_8_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))] = ph_8[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int i_c_outer_inner = 0; i_c_outer_inner < 8; ++i_c_outer_inner) {
      T_batch_matmul_NN_local[i_c_outer_inner] = (T_batch_matmul_NN_local[i_c_outer_inner] + ((ph_0[(((((int)threadIdx.x) * 40) + (i_c_outer_inner * 5)) + k_outer_inner)] / __expf(ph_0[(((((int)threadIdx.x) * 40) + (i_c_outer_inner * 5)) + k_outer_inner)])) * ph_8_shared[((((int)threadIdx.x) * 5) + k_outer_inner)]));
      T_batch_matmul_NN_local[(i_c_outer_inner + 8)] = (T_batch_matmul_NN_local[(i_c_outer_inner + 8)] + ((ph_0[((((((int)threadIdx.x) * 40) + (i_c_outer_inner * 5)) + k_outer_inner) + 160)] / __expf(ph_0[((((((int)threadIdx.x) * 40) + (i_c_outer_inner * 5)) + k_outer_inner) + 160)])) * ph_8_shared[(((((int)threadIdx.x) * 5) + k_outer_inner) + 20)]));
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    T_batch_matmul_NN[((((int)threadIdx.x) * 8) + i_inner)] = T_batch_matmul_NN_local[i_inner];
    T_batch_matmul_NN[(((((int)threadIdx.x) * 8) + i_inner) + 32)] = T_batch_matmul_NN_local[(i_inner + 8)];
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

