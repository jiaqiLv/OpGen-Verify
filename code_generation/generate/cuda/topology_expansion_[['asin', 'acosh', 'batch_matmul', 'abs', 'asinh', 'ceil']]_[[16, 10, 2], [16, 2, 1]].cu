
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[3];
  __shared__ float compute_shared[12];
  __shared__ float ph_4_shared[4];
  for (int i_c_inner_init = 0; i_c_inner_init < 3; ++i_c_inner_init) {
    T_batch_matmul_NN_local[i_c_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 12; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      compute_shared[ax0_ax1_fused_ax2_fused_outer_outer] = acoshf(ph_0[((((((int)blockIdx.x) * 24) + ((ax0_ax1_fused_ax2_fused_outer_outer >> 2) * 8)) + (k_outer_outer * 4)) + (ax0_ax1_fused_ax2_fused_outer_outer & 3))]);
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 4; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
      ph_4_shared[ax0_ax1_fused_ax2_fused_outer_outer_1] = ph_4[((k_outer_outer * 4) + ax0_ax1_fused_ax2_fused_outer_outer_1)];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 4; ++k_inner) {
      for (int i_c_inner = 0; i_c_inner < 3; ++i_c_inner) {
        T_batch_matmul_NN_local[i_c_inner] = (T_batch_matmul_NN_local[i_c_inner] + (compute_shared[((i_c_inner * 4) + k_inner)] * ph_4_shared[k_inner]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 3; ++i_inner) {
    T_batch_matmul_NN[((((int)blockIdx.x) * 3) + i_inner)] = T_batch_matmul_NN_local[i_inner];
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(fabsf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

