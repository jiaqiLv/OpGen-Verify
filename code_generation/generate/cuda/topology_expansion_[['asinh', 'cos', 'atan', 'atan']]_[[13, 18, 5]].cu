
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanf(__cosf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

