
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[8];
  __shared__ float compute_shared[160];
  __shared__ float ph_4_shared[20];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 4; ++b_c_outer_inner_init) {
    for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
      T_batch_matmul_NN_local[((b_c_outer_inner_init * 2) + i_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 40; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))] = __sinf(ph_0[(((((int)blockIdx.x) * 160) + (ax0_ax1_fused_ax2_fused_outer_outer * 4)) + ((int)threadIdx.x))]);
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 5; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_4_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 4) + ((int)threadIdx.x))] = ph_4[(((((int)blockIdx.x) * 20) + (ax0_ax1_fused_ax2_fused_outer_outer_1 * 4)) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
        T_batch_matmul_NN_local[((b_c_outer_inner * 2) + i_c_outer_inner)] = (T_batch_matmul_NN_local[((b_c_outer_inner * 2) + i_c_outer_inner)] + (compute_shared[((((b_c_outer_inner * 40) + (((int)threadIdx.x) * 10)) + (i_c_outer_inner * 5)) + k_outer_inner)] * ph_4_shared[((b_c_outer_inner * 5) + k_outer_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      T_batch_matmul_NN[((((((int)blockIdx.x) * 32) + (b_inner * 8)) + (((int)threadIdx.x) * 2)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 2) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

