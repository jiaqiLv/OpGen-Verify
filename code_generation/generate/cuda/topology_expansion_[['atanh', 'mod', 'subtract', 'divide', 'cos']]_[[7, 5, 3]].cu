
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] / (fmodf(ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))], atanhf(ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))])) - ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]));
}
