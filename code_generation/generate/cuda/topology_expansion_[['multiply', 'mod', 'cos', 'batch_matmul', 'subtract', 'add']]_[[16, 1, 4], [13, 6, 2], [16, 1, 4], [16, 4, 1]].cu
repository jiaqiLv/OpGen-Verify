
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_7) {
  float T_batch_matmul_NN_local[32];
  __shared__ float compute_shared[64];
  __shared__ float ph_7_shared[8];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 4; ++b_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_outer_inner_init * 2) + b_c_inner_init)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_outer_inner_init * 2) + b_c_inner_init) + 8)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_outer_inner_init * 2) + b_c_inner_init) + 16)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_outer_inner_init * 2) + b_c_inner_init) + 24)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x))] = __cosf(ph_0[(((ax0_ax1_fused_ax2_fused_outer_outer * 10) + (((int)threadIdx.x) * 5)) + k_outer_outer)]);
    }
    for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 4; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
      ph_7_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 2) + ((int)threadIdx.x))] = ph_7[(((ax0_ax1_fused_ax2_fused_outer_outer_1 * 10) + (((int)threadIdx.x) * 5)) + k_outer_outer)];
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
      for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
        T_batch_matmul_NN_local[((b_c_outer_inner * 2) + b_c_inner)] = (T_batch_matmul_NN_local[((b_c_outer_inner * 2) + b_c_inner)] + (compute_shared[(((b_c_outer_inner * 16) + (b_c_inner * 8)) + ((int)threadIdx.x))] * ph_7_shared[((b_c_outer_inner * 2) + b_c_inner)]));
        T_batch_matmul_NN_local[(((b_c_outer_inner * 2) + b_c_inner) + 8)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 2) + b_c_inner) + 8)] + (compute_shared[((((b_c_outer_inner * 16) + (b_c_inner * 8)) + ((int)threadIdx.x)) + 2)] * ph_7_shared[((b_c_outer_inner * 2) + b_c_inner)]));
        T_batch_matmul_NN_local[(((b_c_outer_inner * 2) + b_c_inner) + 16)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 2) + b_c_inner) + 16)] + (compute_shared[((((b_c_outer_inner * 16) + (b_c_inner * 8)) + ((int)threadIdx.x)) + 4)] * ph_7_shared[((b_c_outer_inner * 2) + b_c_inner)]));
        T_batch_matmul_NN_local[(((b_c_outer_inner * 2) + b_c_inner) + 24)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 2) + b_c_inner) + 24)] + (compute_shared[((((b_c_outer_inner * 16) + (b_c_inner * 8)) + ((int)threadIdx.x)) + 6)] * ph_7_shared[((b_c_outer_inner * 2) + b_c_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 8; ++b_inner) {
    T_batch_matmul_NN[((b_inner * 8) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[b_inner];
    T_batch_matmul_NN[(((b_inner * 8) + ((int)threadIdx.x)) + 2)] = T_batch_matmul_NN_local[(b_inner + 8)];
    T_batch_matmul_NN[(((b_inner * 8) + ((int)threadIdx.x)) + 4)] = T_batch_matmul_NN_local[(b_inner + 16)];
    T_batch_matmul_NN[(((b_inner * 8) + ((int)threadIdx.x)) + 6)] = T_batch_matmul_NN_local[(b_inner + 24)];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

