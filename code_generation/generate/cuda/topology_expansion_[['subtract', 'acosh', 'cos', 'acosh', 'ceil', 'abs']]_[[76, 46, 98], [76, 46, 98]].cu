
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute_6, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 114985) {
    compute_6[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute_10, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 114985) {
    compute_10[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute_14, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 114985) {
    compute_14[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fabsf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute_12, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 114985) {
    compute_12[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

