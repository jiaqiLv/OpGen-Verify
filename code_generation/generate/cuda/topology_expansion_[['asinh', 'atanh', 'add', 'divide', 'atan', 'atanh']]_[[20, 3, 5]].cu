
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] / (atanhf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

