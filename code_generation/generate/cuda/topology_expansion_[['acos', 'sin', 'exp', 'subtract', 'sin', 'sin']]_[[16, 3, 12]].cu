
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = acosf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] - __expf(__sinf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(__sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

