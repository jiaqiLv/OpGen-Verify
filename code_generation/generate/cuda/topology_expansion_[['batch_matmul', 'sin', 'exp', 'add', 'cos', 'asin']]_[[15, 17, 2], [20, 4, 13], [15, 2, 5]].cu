
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(45) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[8];
  __shared__ float ph_3_shared[180];
  for (int i_inner_init = 0; i_inner_init < 8; ++i_inner_init) {
    T_batch_matmul_NN[i_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 45) + ((int)threadIdx.x))] = ph_3[(((((((ax0_ax1_fused_ax2_fused_outer_outer * 9) + (((int)threadIdx.x) / 5)) >> 2) * 40) + (k_outer_outer * 20)) + ((((((int)threadIdx.x) / 5) + ax0_ax1_fused_ax2_fused_outer_outer) & 3) * 5)) + (((int)threadIdx.x) % 5))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 4; ++k_inner) {
      for (int i_inner = 0; i_inner < 8; ++i_inner) {
        T_batch_matmul_NN[i_inner] = (T_batch_matmul_NN[i_inner] + (ph_0[(((((((int)threadIdx.x) / 5) * 64) + (i_inner * 8)) + (k_outer_outer * 4)) + k_inner)] * ph_3_shared[((((((int)threadIdx.x) / 5) * 20) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 8; ++i1_inner) {
    compute[((((((int)threadIdx.x) / 5) * 40) + (i1_inner * 5)) + (((int)threadIdx.x) % 5))] = asinf(T_batch_matmul_NN[i1_inner]);
  }
}

