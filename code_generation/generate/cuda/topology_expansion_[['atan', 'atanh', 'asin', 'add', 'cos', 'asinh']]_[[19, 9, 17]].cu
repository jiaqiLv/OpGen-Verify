
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] + asinf(atanhf(ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}
