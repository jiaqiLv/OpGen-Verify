
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_3(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[8];
  __shared__ float ph_3_shared[40];
  for (int i_inner_init = 0; i_inner_init < 4; ++i_inner_init) {
    T_batch_matmul_NN[i_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN[(i_inner_init + 4)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 8) + ((int)threadIdx.x))] = ph_3[(((ax0_ax1_fused_ax2_fused_outer_outer * 40) + (((int)threadIdx.x) * 5)) + ((int)blockIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int i_inner = 0; i_inner < 4; ++i_inner) {
      T_batch_matmul_NN[i_inner] = (T_batch_matmul_NN[i_inner] + (ph_0[(((((int)threadIdx.x) * 40) + (i_inner * 5)) + k_outer_inner)] * ph_3_shared[((((int)threadIdx.x) * 5) + k_outer_inner)]));
      T_batch_matmul_NN[(i_inner + 4)] = (T_batch_matmul_NN[(i_inner + 4)] + (ph_0[((((((int)threadIdx.x) * 40) + (i_inner * 5)) + k_outer_inner) + 20)] * ph_3_shared[((((int)threadIdx.x) * 5) + k_outer_inner)]));
    }
  }
  for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
    T_subtract[(((((int)threadIdx.x) * 40) + (ax1_inner * 5)) + ((int)blockIdx.x))] = (T_batch_matmul_NN[ax1_inner] - ph_0[(((((int)threadIdx.x) * 40) + (ax1_inner * 5)) + ((int)blockIdx.x))]);
    T_subtract[((((((int)threadIdx.x) * 40) + (ax1_inner * 5)) + ((int)blockIdx.x)) + 20)] = (T_batch_matmul_NN[(ax1_inner + 4)] - ph_0[((((((int)threadIdx.x) * 40) + (ax1_inner * 5)) + ((int)blockIdx.x)) + 20)]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanhf(atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

