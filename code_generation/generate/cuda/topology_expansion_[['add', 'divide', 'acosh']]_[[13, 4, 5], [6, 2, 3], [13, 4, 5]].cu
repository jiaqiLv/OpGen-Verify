
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

