
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ceilf(__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]);
}

