
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __cosf(__sinf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + (atanf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

