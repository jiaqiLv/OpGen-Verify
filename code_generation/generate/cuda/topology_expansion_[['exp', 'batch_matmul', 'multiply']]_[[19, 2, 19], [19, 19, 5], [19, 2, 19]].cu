
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * ph_5[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(80) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[4];
  __shared__ float ph_3_shared[40];
  for (int b_c_inner_init = 0; b_c_inner_init < 4; ++b_c_inner_init) {
    T_batch_matmul_NN_local[b_c_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 40) {
      ph_3_shared[((int)threadIdx.x)] = ph_3[((((((int)threadIdx.x) / 5) * 25) + (k_outer_outer * 5)) + (((int)threadIdx.x) % 5))];
    }
    __syncthreads();
    for (int b_c_inner = 0; b_c_inner < 4; ++b_c_inner) {
      T_batch_matmul_NN_local[b_c_inner] = (T_batch_matmul_NN_local[b_c_inner] + (ph_0[(((((((int)threadIdx.x) / 40) * 160) + (b_c_inner * 40)) + (((((int)threadIdx.x) % 40) / 5) * 5)) + k_outer_outer)] * ph_3_shared[((((((int)threadIdx.x) / 40) * 20) + (b_c_inner * 5)) + (((int)threadIdx.x) % 5))]));
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    T_batch_matmul_NN[((((((int)threadIdx.x) / 40) * 160) + (b_inner * 40)) + (((int)threadIdx.x) % 40))] = T_batch_matmul_NN_local[b_inner];
  }
}

