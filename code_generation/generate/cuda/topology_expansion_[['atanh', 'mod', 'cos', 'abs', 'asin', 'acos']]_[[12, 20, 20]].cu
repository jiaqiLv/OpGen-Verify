
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(18) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = acosf(asinf(ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}
