
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = acosf((ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] / (ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

