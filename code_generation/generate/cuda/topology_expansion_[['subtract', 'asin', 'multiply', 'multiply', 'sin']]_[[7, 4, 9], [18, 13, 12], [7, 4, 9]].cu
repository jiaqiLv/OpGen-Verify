
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = __sinf((ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * asinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((int)blockIdx.x)] = ((ph_0[((int)blockIdx.x)] * asinf(ph_0[((int)blockIdx.x)])) * ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
