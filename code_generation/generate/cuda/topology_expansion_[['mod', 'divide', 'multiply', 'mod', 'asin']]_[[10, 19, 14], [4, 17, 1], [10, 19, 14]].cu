
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = fmodf((ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] * (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])), ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinf((ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] * (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

