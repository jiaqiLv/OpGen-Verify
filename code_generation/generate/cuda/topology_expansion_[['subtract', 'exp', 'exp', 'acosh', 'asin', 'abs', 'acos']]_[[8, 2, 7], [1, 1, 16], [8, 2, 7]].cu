
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = acoshf(__expf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinf(__expf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

