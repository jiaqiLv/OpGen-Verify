
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(20) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[8];
  __shared__ float ph_3_shared[40];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 4; ++b_c_outer_inner_init) {
    T_batch_matmul_NN_local[b_c_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_outer_inner_init + 4)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 20) + ((int)threadIdx.x))] = ph_3[((((ax0_ax1_fused_ax2_fused_outer_outer * 100) + ((((int)threadIdx.x) / 5) * 25)) + (k_outer_outer * 5)) + (((int)threadIdx.x) % 5))];
    }
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
      T_batch_matmul_NN_local[b_c_outer_inner] = (T_batch_matmul_NN_local[b_c_outer_inner] + (ph_0[((((b_c_outer_inner * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) / 5) * 5)) + k_outer_outer)] * ph_3_shared[((b_c_outer_inner * 5) + (((int)threadIdx.x) % 5))]));
      T_batch_matmul_NN_local[(b_c_outer_inner + 4)] = (T_batch_matmul_NN_local[(b_c_outer_inner + 4)] + (ph_0[(((((b_c_outer_inner * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) / 5) * 5)) + k_outer_outer) + 160)] * ph_3_shared[(((b_c_outer_inner * 5) + (((int)threadIdx.x) % 5)) + 20)]));
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    T_batch_matmul_NN[(((b_inner * 40) + (((int)blockIdx.x) * 20)) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[b_inner];
    T_batch_matmul_NN[((((b_inner * 40) + (((int)blockIdx.x) * 20)) + ((int)threadIdx.x)) + 160)] = T_batch_matmul_NN_local[(b_inner + 4)];
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_5[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

