
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = acoshf(acoshf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = acosf((ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

