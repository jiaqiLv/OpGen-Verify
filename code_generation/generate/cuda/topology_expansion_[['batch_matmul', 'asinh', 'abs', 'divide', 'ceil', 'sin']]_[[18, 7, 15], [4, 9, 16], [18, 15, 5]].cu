
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(18) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[10];
  __shared__ float ph_3_shared[180];
  T_batch_matmul_NN[0] = 0.000000e+00f;
  T_batch_matmul_NN[1] = 0.000000e+00f;
  T_batch_matmul_NN[2] = 0.000000e+00f;
  T_batch_matmul_NN[3] = 0.000000e+00f;
  T_batch_matmul_NN[4] = 0.000000e+00f;
  T_batch_matmul_NN[5] = 0.000000e+00f;
  T_batch_matmul_NN[6] = 0.000000e+00f;
  T_batch_matmul_NN[7] = 0.000000e+00f;
  T_batch_matmul_NN[8] = 0.000000e+00f;
  T_batch_matmul_NN[9] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      *(float2*)(ph_3_shared + ((ax0_ax1_fused_ax2_fused_outer_outer * 36) + (((int)threadIdx.x) * 2))) = *(float2*)(ph_3 + ((((((ax0_ax1_fused_ax2_fused_outer_outer * 36) + (((int)threadIdx.x) * 2)) / 20) * 40) + (k_outer_outer * 20)) + (((ax0_ax1_fused_ax2_fused_outer_outer * 16) + (((int)threadIdx.x) * 2)) % 20)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_batch_matmul_NN[0] = (T_batch_matmul_NN[0] + (ph_0[(((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5))]));
        T_batch_matmul_NN[1] = (T_batch_matmul_NN[1] + (ph_0[(((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 1)]));
        T_batch_matmul_NN[2] = (T_batch_matmul_NN[2] + (ph_0[(((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 2)]));
        T_batch_matmul_NN[3] = (T_batch_matmul_NN[3] + (ph_0[(((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 3)]));
        T_batch_matmul_NN[4] = (T_batch_matmul_NN[4] + (ph_0[(((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 4)]));
        T_batch_matmul_NN[5] = (T_batch_matmul_NN[5] + (ph_0[((((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner) + 16)] * ph_3_shared[((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5))]));
        T_batch_matmul_NN[6] = (T_batch_matmul_NN[6] + (ph_0[((((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner) + 16)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 1)]));
        T_batch_matmul_NN[7] = (T_batch_matmul_NN[7] + (ph_0[((((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner) + 16)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 2)]));
        T_batch_matmul_NN[8] = (T_batch_matmul_NN[8] + (ph_0[((((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner) + 16)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 3)]));
        T_batch_matmul_NN[9] = (T_batch_matmul_NN[9] + (ph_0[((((((((((int)threadIdx.x) >> 1) * 64) + (((int)blockIdx.x) * 32)) + ((((int)threadIdx.x) & 1) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner) + 16)] * ph_3_shared[(((((((int)threadIdx.x) >> 1) * 20) + (k_outer_inner * 10)) + (k_inner * 5)) + 4)]));
      }
    }
  }
  compute[((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5))] = __sinf(T_batch_matmul_NN[0]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 1)] = __sinf(T_batch_matmul_NN[1]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 2)] = __sinf(T_batch_matmul_NN[2]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 3)] = __sinf(T_batch_matmul_NN[3]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 4)] = __sinf(T_batch_matmul_NN[4]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 10)] = __sinf(T_batch_matmul_NN[5]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 11)] = __sinf(T_batch_matmul_NN[6]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 12)] = __sinf(T_batch_matmul_NN[7]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 13)] = __sinf(T_batch_matmul_NN[8]);
  compute[(((((((int)threadIdx.x) >> 1) * 40) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 5)) + 14)] = __sinf(T_batch_matmul_NN[9]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

