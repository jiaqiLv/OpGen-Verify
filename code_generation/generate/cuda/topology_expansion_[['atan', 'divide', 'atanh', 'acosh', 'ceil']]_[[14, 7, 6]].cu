
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = atanhf((ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] / atanf(ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))])));
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = acoshf(ph_0[((int)blockIdx.x)]);
}
