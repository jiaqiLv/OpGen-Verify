
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

