
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = acoshf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __expf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])));
  }
}

