
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + __cosf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))])) + ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = acosf((ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

