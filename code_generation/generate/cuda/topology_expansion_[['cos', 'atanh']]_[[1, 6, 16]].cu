
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]);
}

