
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_6) {
  float T_batch_matmul_NN[1];
  __shared__ float ph_6_shared[1];
  T_batch_matmul_NN[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    ph_6_shared[0] = ph_6[((k_outer_outer * 5) + (((int)blockIdx.x) % 5))];
    __syncthreads();
    T_batch_matmul_NN[0] = (T_batch_matmul_NN[0] + (ph_0[(((((int)blockIdx.x) / 5) * 8) + k_outer_outer)] * ph_6_shared[0]));
  }
  compute[((int)blockIdx.x)] = asinhf(T_batch_matmul_NN[0]);
}

