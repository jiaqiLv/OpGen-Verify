
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}
