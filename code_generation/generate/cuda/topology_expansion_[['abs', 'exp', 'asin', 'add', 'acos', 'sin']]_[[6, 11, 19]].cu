
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + asinf(__expf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __sinf(acosf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

