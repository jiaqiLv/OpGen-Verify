
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = ceilf(fabsf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], (atanhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

