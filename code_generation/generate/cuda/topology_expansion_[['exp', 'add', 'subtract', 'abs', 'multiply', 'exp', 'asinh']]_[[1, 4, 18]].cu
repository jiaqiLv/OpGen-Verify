
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __expf((ph_0[((int)blockIdx.x)] * fabsf(ph_0[((int)blockIdx.x)])));
}

extern "C" __global__ void default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((int)blockIdx.x)] = __expf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

