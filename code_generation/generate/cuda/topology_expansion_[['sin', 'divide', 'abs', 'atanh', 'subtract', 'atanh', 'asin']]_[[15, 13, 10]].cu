
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = asinf((ph_0[((int)blockIdx.x)] - atanhf(ph_0[((int)blockIdx.x)])));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = fabsf((ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] / __sinf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))])));
}
