
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __sinf(__expf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_4(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

