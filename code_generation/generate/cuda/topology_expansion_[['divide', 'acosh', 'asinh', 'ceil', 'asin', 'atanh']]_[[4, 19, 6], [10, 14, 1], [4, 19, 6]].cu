
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = ceilf(asinhf(ph_0[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanhf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

