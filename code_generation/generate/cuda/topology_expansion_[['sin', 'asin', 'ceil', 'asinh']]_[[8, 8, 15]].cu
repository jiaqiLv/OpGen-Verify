
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = ceilf(asinf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]));
}
