
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __expf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = acosf(atanhf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0) {
  float T_batch_matmul_NN_local[2];
  __shared__ float compute_shared[48];
  for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
    T_batch_matmul_NN_local[j_c_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    compute_shared[((int)threadIdx.x)] = ceilf(ph_0[(((((((int)blockIdx.x) >> 1) * 192) + ((((int)threadIdx.x) >> 1) * 8)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))]);
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
        T_batch_matmul_NN_local[j_c_inner] = (T_batch_matmul_NN_local[j_c_inner] + (compute_shared[(((((int)threadIdx.x) >> 1) * 2) + k_outer_inner)] * ph_0[((((((((((int)blockIdx.x) >> 1) * 192) + ((((int)threadIdx.x) >> 4) * 64)) + (k_outer_outer * 16)) + (k_outer_inner * 8)) + ((((int)blockIdx.x) & 1) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + j_c_inner)]));
      }
    }
  }
  for (int j_inner = 0; j_inner < 2; ++j_inner) {
    T_batch_matmul_NN[((((((((int)blockIdx.x) >> 1) * 192) + ((((int)threadIdx.x) >> 1) * 8)) + ((((int)blockIdx.x) & 1) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + j_inner)] = T_batch_matmul_NN_local[j_inner];
  }
}

