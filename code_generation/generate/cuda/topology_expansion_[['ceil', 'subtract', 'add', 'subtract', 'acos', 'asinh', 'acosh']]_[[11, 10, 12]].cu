
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(acosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (0.000000e+00f - (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = asinhf(acosf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}
