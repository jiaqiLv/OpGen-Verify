
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[4];
  __shared__ float compute_shared[160];
  __shared__ float ph_4_shared[40];
  for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
    T_batch_matmul_NN_local[b_c_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_inner_init + 2)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 20; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 8) + ((int)threadIdx.x))] = __expf(ph_0[((((((ax0_ax1_fused_ax2_fused_outer_outer * 2) + (((int)threadIdx.x) >> 2)) / 5) * 40) + (((int)blockIdx.x) * 20)) + (((ax0_ax1_fused_ax2_fused_outer_outer * 8) + ((int)threadIdx.x)) % 20))]);
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 5; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_4_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 8) + ((int)threadIdx.x))] = ph_4[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 8) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
      T_batch_matmul_NN_local[b_c_inner] = (T_batch_matmul_NN_local[b_c_inner] + (compute_shared[(((((((int)threadIdx.x) >> 1) * 40) + (b_c_inner * 20)) + ((((int)threadIdx.x) & 1) * 5)) + k_outer_inner)] * ph_4_shared[((((((int)threadIdx.x) >> 1) * 10) + (b_c_inner * 5)) + k_outer_inner)]));
      T_batch_matmul_NN_local[(b_c_inner + 2)] = (T_batch_matmul_NN_local[(b_c_inner + 2)] + (compute_shared[((((((((int)threadIdx.x) >> 1) * 40) + (b_c_inner * 20)) + ((((int)threadIdx.x) & 1) * 5)) + k_outer_inner) + 10)] * ph_4_shared[((((((int)threadIdx.x) >> 1) * 10) + (b_c_inner * 5)) + k_outer_inner)]));
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[(((((((int)threadIdx.x) >> 1) * 16) + (b_inner * 8)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) & 1))] = T_batch_matmul_NN_local[b_inner];
    T_batch_matmul_NN[((((((((int)threadIdx.x) >> 1) * 16) + (b_inner * 8)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) & 1)) + 2)] = T_batch_matmul_NN_local[(b_inner + 2)];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

