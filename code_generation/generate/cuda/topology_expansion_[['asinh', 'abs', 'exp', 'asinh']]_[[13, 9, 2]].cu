
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = __expf(fabsf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

