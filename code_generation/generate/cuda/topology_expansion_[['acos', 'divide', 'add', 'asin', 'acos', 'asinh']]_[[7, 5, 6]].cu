
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = asinhf(acosf(ph_0[((int)blockIdx.x)]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] / acosf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])) + ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

