
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((int)blockIdx.x)] = (ph_0[((int)blockIdx.x)] / ph_3[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

