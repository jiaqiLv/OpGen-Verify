
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

