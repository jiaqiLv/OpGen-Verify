
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(20) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  float T_batch_matmul_NN_local[2];
  __shared__ float ph_5_shared[32];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  T_batch_matmul_NN_local[1] = 0.000000e+00f;
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_outer_outer * 5) + (((int)threadIdx.x) >> 2)) < 8) {
      ph_5_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 20) + ((int)threadIdx.x))] = ph_5[(((ax0_ax1_fused_ax2_fused_outer_outer * 100) + (((int)threadIdx.x) * 5)) + ((int)blockIdx.x))];
    }
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 4; ++k_inner) {
    T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (ph_0[((((int)threadIdx.x) * 4) + k_inner)] * ph_5_shared[(((((int)threadIdx.x) / 5) * 4) + k_inner)]));
    T_batch_matmul_NN_local[1] = (T_batch_matmul_NN_local[1] + (ph_0[(((((int)threadIdx.x) * 4) + k_inner) + 80)] * ph_5_shared[((((((int)threadIdx.x) / 5) * 4) + k_inner) + 16)]));
  }
  T_batch_matmul_NN[((((int)threadIdx.x) * 5) + ((int)blockIdx.x))] = T_batch_matmul_NN_local[0];
  T_batch_matmul_NN[(((((int)threadIdx.x) * 5) + ((int)blockIdx.x)) + 100)] = T_batch_matmul_NN_local[1];
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

