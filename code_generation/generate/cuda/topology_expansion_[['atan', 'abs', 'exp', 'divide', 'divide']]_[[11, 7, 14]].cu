
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((int)blockIdx.x)] = (ph_0[((int)blockIdx.x)] / (ph_0[((int)blockIdx.x)] / __expf(fabsf(ph_0[((int)blockIdx.x)]))));
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))]);
}

