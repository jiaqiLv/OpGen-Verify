
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __expf(__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __sinf(fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

