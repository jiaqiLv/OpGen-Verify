
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = asinhf(fmodf(ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fabsf(acoshf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

