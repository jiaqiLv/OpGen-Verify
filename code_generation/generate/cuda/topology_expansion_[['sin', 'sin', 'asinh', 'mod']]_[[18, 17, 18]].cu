
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], asinhf(__sinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
}

extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __sinf(ph_0[((int)blockIdx.x)]);
}

