
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(120) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[3];
  __shared__ float ph_3_shared[360];
  for (int b_inner_init = 0; b_inner_init < 3; ++b_inner_init) {
    T_batch_matmul_NN[b_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 120) + ((int)threadIdx.x))] = ph_3[((ax0_ax1_fused_ax2_fused_outer_outer * 120) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 8; ++k_inner) {
    for (int b_inner = 0; b_inner < 3; ++b_inner) {
      T_batch_matmul_NN[b_inner] = (T_batch_matmul_NN[b_inner] + (ph_0[(((((((int)threadIdx.x) / 40) * 192) + (b_inner * 64)) + (((((int)threadIdx.x) % 40) / 5) * 8)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) / 40) * 120) + (b_inner * 40)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
    }
  }
  for (int i0_inner = 0; i0_inner < 3; ++i0_inner) {
    compute[((((((int)threadIdx.x) / 40) * 120) + (i0_inner * 40)) + (((int)threadIdx.x) % 40))] = __expf(T_batch_matmul_NN[i0_inner]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (asinhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

