
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_4(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = acosf(__cosf(ph_0[((int)blockIdx.x)]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}
