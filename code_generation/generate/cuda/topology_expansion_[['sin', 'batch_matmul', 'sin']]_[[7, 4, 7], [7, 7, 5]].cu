
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(72) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  float T_batch_matmul_NN[8];
  __shared__ float compute_shared[576];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 2; ++i_outer_inner_init) {
    for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
      for (int j_inner_init = 0; j_inner_init < 2; ++j_inner_init) {
        T_batch_matmul_NN[(((i_outer_inner_init * 4) + (i_inner_init * 2)) + j_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 72) + ((int)threadIdx.x))] = __sinf(ph_0[((ax0_ax1_fused_ax2_fused_outer_outer * 72) + ((int)threadIdx.x))]);
  }
  __syncthreads();
  for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      for (int i_inner = 0; i_inner < 2; ++i_inner) {
        for (int j_inner = 0; j_inner < 2; ++j_inner) {
          T_batch_matmul_NN[(((i_outer_inner * 4) + (i_inner * 2)) + j_inner)] = (T_batch_matmul_NN[(((i_outer_inner * 4) + (i_inner * 2)) + j_inner)] + (ph_0[(((((((int)threadIdx.x) >> 2) * 32) + (i_outer_inner * 16)) + (i_inner * 8)) + k_inner)] * compute_shared[(((((((int)threadIdx.x) >> 3) * 64) + (k_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + j_inner)]));
        }
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    for (int i2_inner = 0; i2_inner < 2; ++i2_inner) {
      compute[(((((((int)threadIdx.x) >> 2) * 32) + (i1_inner * 8)) + ((((int)threadIdx.x) & 3) * 2)) + i2_inner)] = __sinf(T_batch_matmul_NN[((i1_inner * 2) + i2_inner)]);
    }
  }
}

