
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanf(fmodf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}
