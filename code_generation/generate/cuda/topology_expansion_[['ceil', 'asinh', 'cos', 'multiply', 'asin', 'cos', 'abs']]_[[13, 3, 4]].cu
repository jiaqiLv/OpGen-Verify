
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] * __cosf(asinhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __cosf(asinf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = fabsf(asinf(ph_0[((int)blockIdx.x)]));
}
