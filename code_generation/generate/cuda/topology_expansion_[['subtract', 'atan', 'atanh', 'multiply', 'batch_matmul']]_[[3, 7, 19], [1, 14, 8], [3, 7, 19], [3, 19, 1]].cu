
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[8];
  __shared__ float ph_8_shared[40];
  for (int b_c_inner_init = 0; b_c_inner_init < 4; ++b_c_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_inner_init * 2) + i_c_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 10; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_8_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))] = ph_8[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 5; ++k_inner) {
    for (int b_c_inner = 0; b_c_inner < 4; ++b_c_inner) {
      for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
        T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_inner)] = (T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_inner)] + (atanhf(ph_0[(((((((((int)threadIdx.x) >> 1) * 160) + (b_c_inner * 40)) + (((int)blockIdx.x) * 20)) + ((((int)threadIdx.x) & 1) * 10)) + (i_c_inner * 5)) + k_inner)]) * ph_8_shared[((((((int)threadIdx.x) >> 1) * 20) + (b_c_inner * 5)) + k_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      T_batch_matmul_NN[((((((((int)threadIdx.x) >> 1) * 32) + (b_inner * 8)) + (((int)blockIdx.x) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 2) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

