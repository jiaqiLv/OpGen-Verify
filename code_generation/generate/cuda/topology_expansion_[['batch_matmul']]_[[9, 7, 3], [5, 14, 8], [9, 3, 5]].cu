
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[2];
  __shared__ float ph_0_shared[40];
  __shared__ float ph_3_shared[8];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  T_batch_matmul_NN_local[1] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      ph_0_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 10) + ((int)threadIdx.x))] = ph_0[((((((((int)blockIdx.x) / 5) * 80) + (ax0_ax1_fused_ax2_fused_outer_outer * 20)) + ((((int)threadIdx.x) >> 1) * 4)) + (k_outer_outer * 2)) + (((int)threadIdx.x) & 1))];
    }
    if (((int)threadIdx.x) < 8) {
      ph_3_shared[((int)threadIdx.x)] = ph_3[((((((((int)blockIdx.x) / 5) * 80) + ((((int)threadIdx.x) >> 1) * 20)) + (k_outer_outer * 10)) + ((((int)threadIdx.x) & 1) * 5)) + (((int)blockIdx.x) % 5))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 2; ++k_inner) {
      T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (ph_0_shared[((((int)threadIdx.x) * 2) + k_inner)] * ph_3_shared[(((((int)threadIdx.x) / 5) * 2) + k_inner)]));
      T_batch_matmul_NN_local[1] = (T_batch_matmul_NN_local[1] + (ph_0_shared[(((((int)threadIdx.x) * 2) + k_inner) + 20)] * ph_3_shared[((((((int)threadIdx.x) / 5) * 2) + k_inner) + 4)]));
    }
  }
  T_batch_matmul_NN[((((((int)blockIdx.x) / 5) * 100) + (((int)threadIdx.x) * 5)) + (((int)blockIdx.x) % 5))] = T_batch_matmul_NN_local[0];
  T_batch_matmul_NN[(((((((int)blockIdx.x) / 5) * 100) + (((int)threadIdx.x) * 5)) + (((int)blockIdx.x) % 5)) + 50)] = T_batch_matmul_NN_local[1];
}

