
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

