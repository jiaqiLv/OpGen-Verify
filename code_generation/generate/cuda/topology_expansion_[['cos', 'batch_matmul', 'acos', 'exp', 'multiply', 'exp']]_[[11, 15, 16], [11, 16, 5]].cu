
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[9];
  __shared__ float ph_3_shared[4];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 3; ++i_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 3; ++i_c_inner_init) {
      T_batch_matmul_NN_local[((i_c_outer_inner_init * 3) + i_c_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 4; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      ph_3_shared[ax0_ax1_fused_ax2_fused_outer_outer] = ph_3[(((k_outer_outer * 20) + (ax0_ax1_fused_ax2_fused_outer_outer * 5)) + ((int)blockIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 3; ++i_c_outer_inner) {
        for (int i_c_inner = 0; i_c_inner < 3; ++i_c_inner) {
          T_batch_matmul_NN_local[((i_c_outer_inner * 3) + i_c_inner)] = (T_batch_matmul_NN_local[((i_c_outer_inner * 3) + i_c_inner)] + (ph_0[((((i_c_outer_inner * 24) + (i_c_inner * 8)) + (k_outer_outer * 4)) + k_outer_inner)] * ph_3_shared[k_outer_inner]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 9; ++i_inner) {
    T_batch_matmul_NN[((i_inner * 5) + ((int)blockIdx.x))] = T_batch_matmul_NN_local[i_inner];
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __expf(__expf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

