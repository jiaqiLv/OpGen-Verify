
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[2];
  __shared__ float ph_8_shared[20];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    T_batch_matmul_NN_local[b_c_outer_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_8_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))] = ph_8[((((((int)blockIdx.x) >> 2) * 20) + (ax0_ax1_fused_ax2_fused_outer_outer * 4)) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      T_batch_matmul_NN_local[b_c_outer_inner] = (T_batch_matmul_NN_local[b_c_outer_inner] + (fmodf(ph_0[(((((((((int)blockIdx.x) >> 2) * 160) + ((((int)threadIdx.x) >> 1) * 80)) + (b_c_outer_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + ((((int)threadIdx.x) & 1) * 5)) + k_outer_inner)], asinf(ph_0[(((((((((int)blockIdx.x) >> 2) * 160) + ((((int)threadIdx.x) >> 1) * 80)) + (b_c_outer_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + ((((int)threadIdx.x) & 1) * 5)) + k_outer_inner)])) * ph_8_shared[((((((int)threadIdx.x) >> 1) * 10) + (b_c_outer_inner * 5)) + k_outer_inner)]));
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[((((((((int)blockIdx.x) >> 2) * 32) + ((((int)threadIdx.x) >> 1) * 16)) + (b_inner * 8)) + ((((int)blockIdx.x) & 3) * 2)) + (((int)threadIdx.x) & 1))] = T_batch_matmul_NN_local[b_inner];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

