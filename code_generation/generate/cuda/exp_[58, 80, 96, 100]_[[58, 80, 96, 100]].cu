
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __expf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

