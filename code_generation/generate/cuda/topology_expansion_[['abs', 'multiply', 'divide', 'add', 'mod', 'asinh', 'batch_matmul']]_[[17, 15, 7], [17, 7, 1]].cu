
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[8];
  __shared__ float ph_8_shared[20];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 4; ++b_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_inner_init * 2) + i_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 20; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_8_shared[ax0_ax1_fused_ax2_fused_outer_outer] = ph_8[(((((int)blockIdx.x) >> 2) * 20) + ax0_ax1_fused_ax2_fused_outer_outer)];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int b_c_inner = 0; b_c_inner < 4; ++b_c_inner) {
        T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_outer_inner)] = (T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_outer_inner)] + (fmodf(ph_0[((((((((int)blockIdx.x) >> 2) * 160) + (b_c_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + (i_c_outer_inner * 5)) + k_outer_inner)], (ph_0[((((((((int)blockIdx.x) >> 2) * 160) + (b_c_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + (i_c_outer_inner * 5)) + k_outer_inner)] + ((ph_0[((((((((int)blockIdx.x) >> 2) * 160) + (b_c_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + (i_c_outer_inner * 5)) + k_outer_inner)] * fabsf(ph_0[((((((((int)blockIdx.x) >> 2) * 160) + (b_c_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + (i_c_outer_inner * 5)) + k_outer_inner)])) / ph_0[((((((((int)blockIdx.x) >> 2) * 160) + (b_c_inner * 40)) + ((((int)blockIdx.x) & 3) * 10)) + (i_c_outer_inner * 5)) + k_outer_inner)]))) * ph_8_shared[((b_c_inner * 5) + k_outer_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      T_batch_matmul_NN[(((((((int)blockIdx.x) >> 2) * 32) + (b_inner * 8)) + ((((int)blockIdx.x) & 3) * 2)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 2) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = asinhf(fmodf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))], (ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] + ((ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] * fabsf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))])) / ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]))));
}

