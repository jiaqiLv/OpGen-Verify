
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[8];
  __shared__ float ph_3_shared[40];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
      T_batch_matmul_NN_local[((b_c_outer_inner_init * 4) + i_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    ph_3_shared[((int)threadIdx.x)] = ph_3[((((((int)threadIdx.x) / 5) * 25) + (k_outer_outer * 5)) + (((int)threadIdx.x) % 5))];
    __syncthreads();
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
        T_batch_matmul_NN_local[((b_c_outer_inner * 4) + i_c_outer_inner)] = (T_batch_matmul_NN_local[((b_c_outer_inner * 4) + i_c_outer_inner)] + (ph_0[((((((((int)threadIdx.x) / 10) * 80) + (b_c_outer_inner * 40)) + (((((int)threadIdx.x) % 10) / 5) * 20)) + (i_c_outer_inner * 5)) + k_outer_outer)] * ph_3_shared[((((((int)threadIdx.x) / 10) * 10) + (b_c_outer_inner * 5)) + (((int)threadIdx.x) % 5))]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 4; ++i_inner) {
      T_batch_matmul_NN[((((((((int)threadIdx.x) / 10) * 80) + (b_inner * 40)) + (((((int)threadIdx.x) % 10) / 5) * 20)) + (i_inner * 5)) + (((int)threadIdx.x) % 5))] = T_batch_matmul_NN_local[((b_inner * 4) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_5(float* __restrict__ T_batch_matmul_NN, float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(T_batch_matmul_NN[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ T_batch_matmul_NN, float* __restrict__ compute) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(T_batch_matmul_NN[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

