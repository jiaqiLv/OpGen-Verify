
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((int)blockIdx.x)] = fmodf((ph_0[((int)blockIdx.x)] + atanf(ph_0[((int)blockIdx.x)])), ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

