
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

