
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[9];
  __shared__ float compute_shared[18];
  __shared__ float ph_4_shared[2];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 9; ++i_c_outer_inner_init) {
    T_batch_matmul_NN_local[i_c_outer_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 18; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      compute_shared[ax0_ax1_fused_ax2_fused_outer_outer] = __cosf(ph_0[((((ax0_ax1_fused_ax2_fused_outer_outer >> 1) * 8) + (k_outer_outer * 2)) + (ax0_ax1_fused_ax2_fused_outer_outer & 1))]);
    }
    *(float2*)(ph_4_shared + 0) = *(float2*)(ph_4 + (k_outer_outer * 2));
    __syncthreads();
    for (int i_c_outer_inner = 0; i_c_outer_inner < 9; ++i_c_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_batch_matmul_NN_local[i_c_outer_inner] = (T_batch_matmul_NN_local[i_c_outer_inner] + (compute_shared[((i_c_outer_inner * 2) + k_inner)] * ph_4_shared[k_inner]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 9; ++i_inner) {
    T_batch_matmul_NN[i_inner] = T_batch_matmul_NN_local[i_inner];
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(__sinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

