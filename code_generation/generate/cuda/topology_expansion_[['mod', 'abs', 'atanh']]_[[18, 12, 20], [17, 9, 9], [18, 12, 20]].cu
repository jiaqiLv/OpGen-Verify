
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

