
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel_3(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[16];
  __shared__ float ph_3_shared[200];
  for (int b_outer_inner_init = 0; b_outer_inner_init < 2; ++b_outer_inner_init) {
    for (int i_outer_inner_init = 0; i_outer_inner_init < 8; ++i_outer_inner_init) {
      T_batch_matmul_NN[((b_outer_inner_init * 8) + i_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 10; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 20) + ((int)threadIdx.x))] = ph_3[((ax0_ax1_fused_ax2_fused_outer_outer * 20) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int b_outer_inner = 0; b_outer_inner < 2; ++b_outer_inner) {
    for (int i_outer_inner = 0; i_outer_inner < 8; ++i_outer_inner) {
      for (int k_inner = 0; k_inner < 5; ++k_inner) {
        T_batch_matmul_NN[((b_outer_inner * 8) + i_outer_inner)] = (T_batch_matmul_NN[((b_outer_inner * 8) + i_outer_inner)] + (ph_0[(((((((int)threadIdx.x) / 5) * 80) + (b_outer_inner * 40)) + (i_outer_inner * 5)) + k_inner)] * ph_3_shared[(((((((int)threadIdx.x) / 5) * 50) + (b_outer_inner * 25)) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      T_mod[(((((((int)threadIdx.x) / 5) * 80) + (ax0_inner * 40)) + (ax1_inner * 5)) + (((int)threadIdx.x) % 5))] = fmodf(T_batch_matmul_NN[((ax0_inner * 8) + ax1_inner)], ph_0[(((((((int)threadIdx.x) / 5) * 80) + (ax0_inner * 40)) + (ax1_inner * 5)) + (((int)threadIdx.x) % 5))]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

