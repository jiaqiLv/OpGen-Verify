
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __sinf(atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = ceilf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

