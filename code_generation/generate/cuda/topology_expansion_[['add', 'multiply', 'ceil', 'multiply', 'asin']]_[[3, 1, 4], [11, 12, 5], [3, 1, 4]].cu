
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinf(ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

