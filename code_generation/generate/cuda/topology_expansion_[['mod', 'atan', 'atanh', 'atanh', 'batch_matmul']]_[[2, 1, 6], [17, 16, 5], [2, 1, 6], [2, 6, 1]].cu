
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanhf(atanhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[4];
  __shared__ float ph_8_shared[8];
  for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
    T_batch_matmul_NN_local[i_c_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(i_c_inner_init + 2)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    ph_8_shared[((int)threadIdx.x)] = ph_8[((((int)threadIdx.x) * 5) + k_outer_outer)];
    __syncthreads();
    for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
      T_batch_matmul_NN_local[i_c_inner] = (T_batch_matmul_NN_local[i_c_inner] + (atanhf(ph_0[((((((int)threadIdx.x) * 40) + (((int)blockIdx.x) * 20)) + (i_c_inner * 5)) + k_outer_outer)]) * ph_8_shared[((int)threadIdx.x)]));
      T_batch_matmul_NN_local[(i_c_inner + 2)] = (T_batch_matmul_NN_local[(i_c_inner + 2)] + (atanhf(ph_0[(((((((int)threadIdx.x) * 40) + (((int)blockIdx.x) * 20)) + (i_c_inner * 5)) + k_outer_outer) + 10)]) * ph_8_shared[((int)threadIdx.x)]));
    }
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    T_batch_matmul_NN[(((((int)threadIdx.x) * 8) + (((int)blockIdx.x) * 4)) + i_inner)] = T_batch_matmul_NN_local[i_inner];
    T_batch_matmul_NN[((((((int)threadIdx.x) * 8) + (((int)blockIdx.x) * 4)) + i_inner) + 2)] = T_batch_matmul_NN_local[(i_inner + 2)];
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

