
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = asinf(__cosf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

