
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_3(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = fmodf(atanhf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}
