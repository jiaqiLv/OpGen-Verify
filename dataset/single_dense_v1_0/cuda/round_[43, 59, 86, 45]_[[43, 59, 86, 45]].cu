
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 4909095) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = roundf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

