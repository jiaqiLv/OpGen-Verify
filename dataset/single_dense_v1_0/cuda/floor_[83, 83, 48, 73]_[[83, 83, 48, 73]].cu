
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = floorf(data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
}

