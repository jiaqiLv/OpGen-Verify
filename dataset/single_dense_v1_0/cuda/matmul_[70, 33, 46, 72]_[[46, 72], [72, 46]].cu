
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(23) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[2];
  __shared__ float left_matrix_shared[552];
  __shared__ float right_matrix_shared[48];
  for (int ax1_c_inner_init = 0; ax1_c_inner_init < 2; ++ax1_c_inner_init) {
    T_matmul_local[ax1_c_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 3; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 24; ++ax0_ax1_fused_outer_outer) {
      left_matrix_shared[((ax0_ax1_fused_outer_outer * 23) + ((int)threadIdx.x))] = left_matrix[(((((((int)blockIdx.x) / 23) * 1656) + ((((ax0_ax1_fused_outer_outer * 23) + ((int)threadIdx.x)) / 24) * 72)) + (k_outer_outer * 24)) + (((ax0_ax1_fused_outer_outer * 23) + ((int)threadIdx.x)) % 24))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 3; ++ax0_ax1_fused_outer_outer_1) {
      if (((ax0_ax1_fused_outer_outer_1 * 23) + ((int)threadIdx.x)) < 48) {
        right_matrix_shared[((ax0_ax1_fused_outer_outer_1 * 23) + ((int)threadIdx.x))] = right_matrix[((((k_outer_outer * 1104) + ((((ax0_ax1_fused_outer_outer_1 * 23) + ((int)threadIdx.x)) >> 1) * 46)) + ((((int)blockIdx.x) % 23) * 2)) + ((ax0_ax1_fused_outer_outer_1 + ((int)threadIdx.x)) & 1))];
      }
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 3; ++k_inner) {
        for (int ax1_c_inner = 0; ax1_c_inner < 2; ++ax1_c_inner) {
          T_matmul_local[ax1_c_inner] = (T_matmul_local[ax1_c_inner] + (left_matrix_shared[(((((int)threadIdx.x) * 24) + (k_outer_inner * 3)) + k_inner)] * right_matrix_shared[(((k_outer_inner * 6) + (k_inner * 2)) + ax1_c_inner)]));
        }
      }
    }
  }
  for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
    T_matmul[(((((((int)blockIdx.x) / 23) * 1058) + (((int)threadIdx.x) * 46)) + ((((int)blockIdx.x) % 23) * 2)) + ax1_inner)] = T_matmul_local[ax1_inner];
  }
}

