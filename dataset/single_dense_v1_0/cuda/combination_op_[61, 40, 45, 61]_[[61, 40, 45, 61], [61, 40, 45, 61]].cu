
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_add, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 837225) {
    T_add[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (sqrtf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) + __cosf(data_1[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

