
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 1544265) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __expf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

