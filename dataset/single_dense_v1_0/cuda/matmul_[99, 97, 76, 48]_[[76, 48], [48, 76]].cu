
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(361) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[1];
  __shared__ float left_matrix_shared[152];
  __shared__ float right_matrix_shared[152];
  T_matmul_local[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 6; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 76) {
      *(float2*)(left_matrix_shared + (((int)threadIdx.x) * 2)) = *(float2*)(left_matrix + (((((((int)blockIdx.x) >> 2) * 912) + ((((int)threadIdx.x) >> 2) * 48)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)));
    }
    if (((int)threadIdx.x) < 76) {
      int2 __1;
        int2 __2;
          int2 __3;
            int2 v_ = make_int2((k_outer_outer * 608), (k_outer_outer * 608));
            int2 __4;
              int2 __5;
                int2 v__1 = make_int2(((((int)threadIdx.x) * 2))+(1*0), ((((int)threadIdx.x) * 2))+(1*1));
                int2 v__2 = make_int2(19, 19);
                __5.x = (v__1.x/v__2.x);
                __5.y = (v__1.y/v__2.y);
              int2 v__3 = make_int2(76, 76);
              __4.x = (__5.x*v__3.x);
              __4.y = (__5.y*v__3.y);
            __3.x = (v_.x+__4.x);
            __3.y = (v_.y+__4.y);
          int2 v__4 = make_int2(((((int)blockIdx.x) & 3) * 19), ((((int)blockIdx.x) & 3) * 19));
          __2.x = (__3.x+v__4.x);
          __2.y = (__3.y+v__4.y);
        int2 __6;
          int2 v__5 = make_int2(((((int)threadIdx.x) * 2))+(1*0), ((((int)threadIdx.x) * 2))+(1*1));
          int2 v__6 = make_int2(19, 19);
          __6.x = (v__5.x%v__6.x);
          __6.y = (v__5.y%v__6.y);
        __1.x = (__2.x+__6.x);
        __1.y = (__2.y+__6.y);
      *(float2*)(right_matrix_shared + (((int)threadIdx.x) * 2)) = make_float2(right_matrix[__1.x],right_matrix[__1.y]);
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_matmul_local[0] = (T_matmul_local[0] + (left_matrix_shared[((((((int)threadIdx.x) / 19) * 8) + (k_outer_inner * 2)) + k_inner)] * right_matrix_shared[(((k_outer_inner * 38) + (k_inner * 19)) + (((int)threadIdx.x) % 19))]));
      }
    }
  }
  T_matmul[(((((((int)blockIdx.x) >> 2) * 1444) + ((((int)threadIdx.x) / 19) * 76)) + ((((int)blockIdx.x) & 3) * 19)) + (((int)threadIdx.x) % 19))] = T_matmul_local[0];
}

