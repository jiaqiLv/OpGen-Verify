
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(42) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 21) + (((int)threadIdx.x) >> 1)) < 3339820) {
    compute[((((int)blockIdx.x) * 42) + ((int)threadIdx.x))] = tanf(data[((((int)blockIdx.x) * 42) + ((int)threadIdx.x))]);
  }
}

