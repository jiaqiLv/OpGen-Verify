
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_7) {
  float T_batch_matmul_NN_local[32];
  __shared__ float compute_shared[160];
  __shared__ float ph_7_shared[40];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_outer_inner_init * 4) + i_c_inner_init)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_outer_inner_init * 4) + i_c_inner_init) + 8)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_outer_inner_init * 4) + i_c_inner_init) + 16)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_outer_inner_init * 4) + i_c_inner_init) + 24)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 160; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[ax0_ax1_fused_ax2_fused_outer_outer] = atanhf(ph_0[((((ax0_ax1_fused_ax2_fused_outer_outer / 20) * 40) + (((int)blockIdx.x) * 20)) + (ax0_ax1_fused_ax2_fused_outer_outer % 20))]);
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 40; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_7_shared[ax0_ax1_fused_ax2_fused_outer_outer_1] = ph_7[ax0_ax1_fused_ax2_fused_outer_outer_1];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        T_batch_matmul_NN_local[((b_c_outer_inner * 4) + i_c_inner)] = (T_batch_matmul_NN_local[((b_c_outer_inner * 4) + i_c_inner)] + (compute_shared[(((b_c_outer_inner * 20) + (i_c_inner * 5)) + k_outer_inner)] * ph_7_shared[((b_c_outer_inner * 5) + k_outer_inner)]));
        T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + i_c_inner) + 8)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + i_c_inner) + 8)] + (compute_shared[((((b_c_outer_inner * 20) + (i_c_inner * 5)) + k_outer_inner) + 40)] * ph_7_shared[(((b_c_outer_inner * 5) + k_outer_inner) + 10)]));
        T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + i_c_inner) + 16)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + i_c_inner) + 16)] + (compute_shared[((((b_c_outer_inner * 20) + (i_c_inner * 5)) + k_outer_inner) + 80)] * ph_7_shared[(((b_c_outer_inner * 5) + k_outer_inner) + 20)]));
        T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + i_c_inner) + 24)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + i_c_inner) + 24)] + (compute_shared[((((b_c_outer_inner * 20) + (i_c_inner * 5)) + k_outer_inner) + 120)] * ph_7_shared[(((b_c_outer_inner * 5) + k_outer_inner) + 30)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 4; ++i_inner) {
      T_batch_matmul_NN[(((b_inner * 8) + (((int)blockIdx.x) * 4)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 4) + i_inner)];
      T_batch_matmul_NN[((((b_inner * 8) + (((int)blockIdx.x) * 4)) + i_inner) + 16)] = T_batch_matmul_NN_local[(((b_inner * 4) + i_inner) + 8)];
      T_batch_matmul_NN[((((b_inner * 8) + (((int)blockIdx.x) * 4)) + i_inner) + 32)] = T_batch_matmul_NN_local[(((b_inner * 4) + i_inner) + 16)];
      T_batch_matmul_NN[((((b_inner * 8) + (((int)blockIdx.x) * 4)) + i_inner) + 48)] = T_batch_matmul_NN_local[(((b_inner * 4) + i_inner) + 24)];
    }
  }
}

