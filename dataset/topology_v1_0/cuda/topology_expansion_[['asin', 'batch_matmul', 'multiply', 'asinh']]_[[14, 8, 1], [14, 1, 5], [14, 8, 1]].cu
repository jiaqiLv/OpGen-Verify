
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(15) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[1];
  __shared__ float ph_3_shared[20];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 10) {
      *(float2*)(ph_3_shared + (((int)threadIdx.x) * 2)) = *(float2*)(ph_3 + ((k_outer_outer * 20) + (((int)threadIdx.x) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 2; ++k_inner) {
        T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (ph_0[(((((((int)blockIdx.x) * 24) + ((((int)threadIdx.x) / 5) * 8)) + (k_outer_outer * 4)) + (k_outer_inner * 2)) + k_inner)] * ph_3_shared[(((k_outer_inner * 10) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
      }
    }
  }
  T_batch_matmul_NN[((((int)blockIdx.x) * 15) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[0];
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_multiply[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_5[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_5) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf(ph_5[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

