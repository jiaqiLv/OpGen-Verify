
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((int)blockIdx.x)] = fmodf(atanhf(ph_0[((int)blockIdx.x)]), ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

