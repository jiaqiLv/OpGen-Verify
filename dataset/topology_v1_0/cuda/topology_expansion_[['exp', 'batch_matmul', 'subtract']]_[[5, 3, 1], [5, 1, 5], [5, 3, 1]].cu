
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_5[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(40) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[2];
  __shared__ float ph_3_shared[200];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    T_batch_matmul_NN_local[b_c_outer_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 3; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    for (int ax0_ax1_fused_ax2_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_inner_s < 2; ++ax0_ax1_fused_ax2_fused_inner_s) {
      if (((ax0_ax1_fused_ax2_fused_outer_outer * 2) + (((int)threadIdx.x) / 20)) < 5) {
        ph_3_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_inner_s)] = ph_3[(((ax0_ax1_fused_ax2_fused_outer_outer * 80) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_inner_s)];
      }
    }
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
      T_batch_matmul_NN_local[b_c_outer_inner] = (T_batch_matmul_NN_local[b_c_outer_inner] + (ph_0[((((((((int)threadIdx.x) / 10) * 80) + (b_c_outer_inner * 40)) + (((int)blockIdx.x) * 10)) + (((((int)threadIdx.x) % 10) / 5) * 5)) + k_outer_inner)] * ph_3_shared[(((((((int)threadIdx.x) / 10) * 50) + (b_c_outer_inner * 25)) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5))]));
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[(((((((int)threadIdx.x) / 10) * 80) + (b_inner * 40)) + (((int)blockIdx.x) * 10)) + (((int)threadIdx.x) % 10))] = T_batch_matmul_NN_local[b_inner];
  }
}

