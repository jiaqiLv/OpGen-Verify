
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = ceilf(fmodf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))], __cosf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __sinf(atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}
