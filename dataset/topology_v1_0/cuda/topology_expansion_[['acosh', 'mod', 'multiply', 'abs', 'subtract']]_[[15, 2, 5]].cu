
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] - fabsf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))], acoshf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]);
}

