
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((int)blockIdx.x)] = (ph_0[((int)blockIdx.x)] * acoshf(atanf(ph_0[((int)blockIdx.x)])));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]);
}

