
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_8) {
  float T_batch_matmul_NN_local[8];
  __shared__ float ph_8_shared[10];
  for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_inner_init * 2) + i_c_inner_init)] = 0.000000e+00f;
      T_batch_matmul_NN_local[(((b_c_inner_init * 2) + i_c_inner_init) + 4)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_8_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x))] = ph_8[(((((int)blockIdx.x) * 10) + (ax0_ax1_fused_ax2_fused_outer_outer * 2)) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
      for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
        T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_inner)] = (T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_inner)] + (ceilf(ph_0[(((((((int)blockIdx.x) * 80) + (b_c_inner * 40)) + (((int)threadIdx.x) * 10)) + (i_c_inner * 5)) + k_outer_inner)]) * ph_8_shared[((b_c_inner * 5) + k_outer_inner)]));
        T_batch_matmul_NN_local[(((b_c_inner * 2) + i_c_inner) + 4)] = (T_batch_matmul_NN_local[(((b_c_inner * 2) + i_c_inner) + 4)] + (ceilf(ph_0[((((((((int)blockIdx.x) * 80) + (b_c_inner * 40)) + (((int)threadIdx.x) * 10)) + (i_c_inner * 5)) + k_outer_inner) + 20)]) * ph_8_shared[((b_c_inner * 5) + k_outer_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      T_batch_matmul_NN[((((((int)blockIdx.x) * 16) + (b_inner * 8)) + (((int)threadIdx.x) * 2)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 2) + i_inner)];
      T_batch_matmul_NN[(((((((int)blockIdx.x) * 16) + (b_inner * 8)) + (((int)threadIdx.x) * 2)) + i_inner) + 4)] = T_batch_matmul_NN_local[(((b_inner * 2) + i_inner) + 4)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / fabsf(ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fmodf(ceilf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

