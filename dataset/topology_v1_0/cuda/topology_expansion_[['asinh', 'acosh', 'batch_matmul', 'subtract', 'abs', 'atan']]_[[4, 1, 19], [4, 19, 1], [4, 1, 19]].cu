
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(72) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  float T_batch_matmul_NN[1];
  __shared__ float compute_shared[576];
  T_batch_matmul_NN[0] = 0.000000e+00f;
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 72) + ((int)threadIdx.x))] = acoshf(ph_0[((ax0_ax1_fused_ax2_fused_outer_outer * 72) + ((int)threadIdx.x))]);
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 8; ++k_inner) {
    T_batch_matmul_NN[0] = (T_batch_matmul_NN[0] + (compute_shared[((((int)threadIdx.x) * 8) + k_inner)] * ph_0[((((((int)threadIdx.x) >> 3) * 64) + (k_inner * 8)) + ((int)blockIdx.x))]));
  }
  T_subtract[((((int)threadIdx.x) * 8) + ((int)blockIdx.x))] = (ph_0[((((int)threadIdx.x) * 8) + ((int)blockIdx.x))] - T_batch_matmul_NN[0]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

