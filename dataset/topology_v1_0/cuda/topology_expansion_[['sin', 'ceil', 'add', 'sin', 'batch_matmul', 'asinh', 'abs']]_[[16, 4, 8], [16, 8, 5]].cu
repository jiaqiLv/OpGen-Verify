
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0) {
  float T_batch_matmul_NN_local[32];
  __shared__ float compute_shared[576];
  for (int j_c_outer_inner_init = 0; j_c_outer_inner_init < 4; ++j_c_outer_inner_init) {
    for (int i_c_inner_init = 0; i_c_inner_init < 4; ++i_c_inner_init) {
      for (int j_c_inner_init = 0; j_c_inner_init < 2; ++j_c_inner_init) {
        T_batch_matmul_NN_local[(((i_c_inner_init * 8) + (j_c_outer_inner_init * 2)) + j_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 32; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 18) + ((int)threadIdx.x))] = __sinf(ph_0[((ax0_ax1_fused_ax2_fused_outer_outer * 18) + ((int)threadIdx.x))]);
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
    for (int j_c_outer_inner = 0; j_c_outer_inner < 4; ++j_c_outer_inner) {
      for (int i_c_inner = 0; i_c_inner < 4; ++i_c_inner) {
        for (int j_c_inner = 0; j_c_inner < 2; ++j_c_inner) {
          T_batch_matmul_NN_local[(((i_c_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner)] = (T_batch_matmul_NN_local[(((i_c_inner * 8) + (j_c_outer_inner * 2)) + j_c_inner)] + (ph_0[(((((int)threadIdx.x) * 32) + (i_c_inner * 8)) + k_outer_inner)] * compute_shared[(((((((int)threadIdx.x) >> 1) * 64) + (k_outer_inner * 8)) + (j_c_outer_inner * 2)) + j_c_inner)]));
        }
      }
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    for (int j_inner = 0; j_inner < 8; ++j_inner) {
      T_batch_matmul_NN[(((((int)threadIdx.x) * 32) + (i_inner * 8)) + j_inner)] = T_batch_matmul_NN_local[((i_inner * 8) + j_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ compute) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(T_batch_matmul_NN[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_4(float* __restrict__ T_batch_matmul_NN, float* __restrict__ compute) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = fabsf(T_batch_matmul_NN[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

