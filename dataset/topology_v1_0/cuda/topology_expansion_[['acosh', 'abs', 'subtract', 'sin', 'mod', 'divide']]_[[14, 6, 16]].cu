
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], __sinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])) / ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

