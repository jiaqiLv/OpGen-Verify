
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

