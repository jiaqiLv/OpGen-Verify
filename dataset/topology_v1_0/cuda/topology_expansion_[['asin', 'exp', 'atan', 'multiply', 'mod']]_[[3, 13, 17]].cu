
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((int)blockIdx.x)] = fmodf(ph_0[((int)blockIdx.x)], (ph_0[((int)blockIdx.x)] * atanf(__expf(ph_0[((int)blockIdx.x)]))));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]);
}

