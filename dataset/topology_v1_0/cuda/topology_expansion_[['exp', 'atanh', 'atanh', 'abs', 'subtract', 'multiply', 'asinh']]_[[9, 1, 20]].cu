
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] - fabsf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - fabsf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
