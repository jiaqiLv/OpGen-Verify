
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_5) {
  float T_batch_matmul_NN_local[2];
  __shared__ float ph_5_shared[80];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  T_batch_matmul_NN_local[1] = 0.000000e+00f;
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_5_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 10) + ((int)threadIdx.x))] = ph_5[((((((int)blockIdx.x) / 5) * 80) + (ax0_ax1_fused_ax2_fused_outer_outer * 10)) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
    T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (ph_0[(((((((int)blockIdx.x) / 5) * 80) + ((((int)threadIdx.x) / 5) * 20)) + ((((int)blockIdx.x) % 5) * 4)) + k_outer_inner)] * ph_5_shared[((((((int)threadIdx.x) / 5) * 20) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5))]));
    T_batch_matmul_NN_local[1] = (T_batch_matmul_NN_local[1] + (ph_0[((((((((int)blockIdx.x) / 5) * 80) + ((((int)threadIdx.x) / 5) * 20)) + ((((int)blockIdx.x) % 5) * 4)) + k_outer_inner) + 40)] * ph_5_shared[(((((((int)threadIdx.x) / 5) * 20) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5)) + 40)]));
  }
  T_batch_matmul_NN[(((((((int)blockIdx.x) / 5) * 100) + ((((int)threadIdx.x) / 5) * 25)) + ((((int)blockIdx.x) % 5) * 5)) + (((int)threadIdx.x) % 5))] = T_batch_matmul_NN_local[0];
  T_batch_matmul_NN[((((((((int)blockIdx.x) / 5) * 100) + ((((int)threadIdx.x) / 5) * 25)) + ((((int)blockIdx.x) % 5) * 5)) + (((int)threadIdx.x) % 5)) + 50)] = T_batch_matmul_NN_local[1];
}

