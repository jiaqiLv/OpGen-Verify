
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = atanhf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf(fabsf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

