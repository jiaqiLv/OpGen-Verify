
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_6) {
  float T_batch_matmul_NN[9];
  __shared__ float ph_6_shared[1];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 3; ++i_outer_inner_init) {
    T_batch_matmul_NN[i_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN[(i_outer_inner_init + 3)] = 0.000000e+00f;
    T_batch_matmul_NN[(i_outer_inner_init + 6)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    ph_6_shared[0] = ph_6[((k_outer_outer * 5) + ((int)blockIdx.x))];
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 3; ++i_outer_inner) {
      T_batch_matmul_NN[i_outer_inner] = (T_batch_matmul_NN[i_outer_inner] + (ph_0[((i_outer_inner * 8) + k_outer_outer)] * ph_6_shared[0]));
      T_batch_matmul_NN[(i_outer_inner + 3)] = (T_batch_matmul_NN[(i_outer_inner + 3)] + (ph_0[(((i_outer_inner * 8) + k_outer_outer) + 24)] * ph_6_shared[0]));
      T_batch_matmul_NN[(i_outer_inner + 6)] = (T_batch_matmul_NN[(i_outer_inner + 6)] + (ph_0[(((i_outer_inner * 8) + k_outer_outer) + 48)] * ph_6_shared[0]));
    }
  }
  for (int i1_inner = 0; i1_inner < 3; ++i1_inner) {
    compute[((i1_inner * 5) + ((int)blockIdx.x))] = atanhf(T_batch_matmul_NN[i1_inner]);
    compute[(((i1_inner * 5) + ((int)blockIdx.x)) + 15)] = atanhf(T_batch_matmul_NN[(i1_inner + 3)]);
    compute[(((i1_inner * 5) + ((int)blockIdx.x)) + 30)] = atanhf(T_batch_matmul_NN[(i1_inner + 6)]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

