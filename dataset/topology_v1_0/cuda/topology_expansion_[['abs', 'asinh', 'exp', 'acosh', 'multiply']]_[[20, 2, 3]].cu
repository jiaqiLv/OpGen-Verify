
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] * acoshf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]));
}
