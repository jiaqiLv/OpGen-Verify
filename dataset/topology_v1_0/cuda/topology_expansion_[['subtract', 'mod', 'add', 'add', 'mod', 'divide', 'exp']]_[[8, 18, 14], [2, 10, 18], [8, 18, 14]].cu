
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + fmodf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])) + ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

