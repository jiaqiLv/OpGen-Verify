
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = ceilf(__expf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    T_divide[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / __sinf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * asinhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]))));
  }
}

