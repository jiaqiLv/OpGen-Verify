
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __cosf((ph_0[((int)blockIdx.x)] - acoshf(ph_0[((int)blockIdx.x)])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

