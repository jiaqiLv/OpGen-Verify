
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acosf(__sinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fmodf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

