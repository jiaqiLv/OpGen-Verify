
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(36) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}
