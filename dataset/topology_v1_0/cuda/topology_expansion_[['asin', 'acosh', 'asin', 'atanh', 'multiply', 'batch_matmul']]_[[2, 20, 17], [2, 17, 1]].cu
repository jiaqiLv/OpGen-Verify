
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_2(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_7) {
  float T_batch_matmul_NN_local[8];
  __shared__ float T_multiply_shared[80];
  __shared__ float ph_7_shared[10];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 2; ++b_c_outer_inner_init) {
    for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        T_batch_matmul_NN_local[(((b_c_outer_inner_init * 4) + (i_c_outer_inner_init * 2)) + i_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 10; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    for (int ax0_ax1_fused_ax2_fused_inner_s = 0; ax0_ax1_fused_ax2_fused_inner_s < 4; ++ax0_ax1_fused_ax2_fused_inner_s) {
      T_multiply_shared[(((ax0_ax1_fused_ax2_fused_outer_outer * 8) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_inner_s)] = (ph_0[((((((int)blockIdx.x) * 80) + (ax0_ax1_fused_ax2_fused_outer_outer * 8)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_inner_s)] * atanhf(ph_0[((((((int)blockIdx.x) * 80) + (ax0_ax1_fused_ax2_fused_outer_outer * 8)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_inner_s)]));
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 5; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_7_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 2) + ((int)threadIdx.x))] = ph_7[(((((int)blockIdx.x) * 10) + (ax0_ax1_fused_ax2_fused_outer_outer_1 * 2)) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int b_c_outer_inner = 0; b_c_outer_inner < 2; ++b_c_outer_inner) {
    for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
      for (int k_inner = 0; k_inner < 5; ++k_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + (i_c_outer_inner * 2)) + i_c_inner)] = (T_batch_matmul_NN_local[(((b_c_outer_inner * 4) + (i_c_outer_inner * 2)) + i_c_inner)] + (T_multiply_shared[(((((b_c_outer_inner * 40) + (((int)threadIdx.x) * 20)) + (i_c_outer_inner * 10)) + (i_c_inner * 5)) + k_inner)] * ph_7_shared[((b_c_outer_inner * 5) + k_inner)]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 4; ++i_inner) {
      T_batch_matmul_NN[((((((int)blockIdx.x) * 16) + (b_inner * 8)) + (((int)threadIdx.x) * 4)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 4) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

