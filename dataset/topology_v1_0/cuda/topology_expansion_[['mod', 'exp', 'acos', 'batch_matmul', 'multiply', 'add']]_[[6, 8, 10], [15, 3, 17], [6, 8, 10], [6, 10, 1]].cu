
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void default_function_kernel_2(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((int)blockIdx.x)] = (fmodf(ph_0[((int)blockIdx.x)], ph_3[((int)blockIdx.x)]) * ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_7) {
  float T_batch_matmul_NN_local[2];
  __shared__ float compute_shared[40];
  __shared__ float ph_7_shared[40];
  for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
    T_batch_matmul_NN_local[b_c_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 10; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x))] = acosf(ph_0[((((((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x)) / 5) * 40) + (((int)blockIdx.x) * 5)) + (((ax0_ax1_fused_ax2_fused_outer_outer * 4) + ((int)threadIdx.x)) % 5))]);
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 10; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_7_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 4) + ((int)threadIdx.x))] = ph_7[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 4) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_inner = 0; k_inner < 5; ++k_inner) {
    for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
      T_batch_matmul_NN_local[b_c_inner] = (T_batch_matmul_NN_local[b_c_inner] + (compute_shared[(((((int)threadIdx.x) * 10) + (b_c_inner * 5)) + k_inner)] * ph_7_shared[(((((int)threadIdx.x) * 10) + (b_c_inner * 5)) + k_inner)]));
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    T_batch_matmul_NN[(((((int)threadIdx.x) * 16) + (b_inner * 8)) + ((int)blockIdx.x))] = T_batch_matmul_NN_local[b_inner];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

