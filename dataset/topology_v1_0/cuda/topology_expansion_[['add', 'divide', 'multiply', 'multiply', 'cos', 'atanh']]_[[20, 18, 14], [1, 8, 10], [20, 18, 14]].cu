
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] * (ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))])) * ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = atanhf((ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}
