
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fmodf(atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = asinhf(atanf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(45) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[8];
  __shared__ float ph_3_shared[90];
  for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
    T_batch_matmul_NN[i_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN[(i_inner_init + 2)] = 0.000000e+00f;
    T_batch_matmul_NN[(i_inner_init + 4)] = 0.000000e+00f;
    T_batch_matmul_NN[(i_inner_init + 6)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
      ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 45) + ((int)threadIdx.x))] = ph_3[(((((((ax0_ax1_fused_ax2_fused_outer_outer * 9) + (((int)threadIdx.x) / 5)) >> 1) * 40) + (k_outer_outer * 10)) + ((((((int)threadIdx.x) / 5) + ax0_ax1_fused_ax2_fused_outer_outer) & 1) * 5)) + (((int)threadIdx.x) % 5))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 2; ++k_inner) {
      for (int i_inner = 0; i_inner < 2; ++i_inner) {
        T_batch_matmul_NN[i_inner] = (T_batch_matmul_NN[i_inner] + (ph_0[(((((((int)threadIdx.x) / 5) * 64) + (i_inner * 8)) + (k_outer_outer * 2)) + k_inner)] * ph_3_shared[((((((int)threadIdx.x) / 5) * 10) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN[(i_inner + 2)] = (T_batch_matmul_NN[(i_inner + 2)] + (ph_0[((((((((int)threadIdx.x) / 5) * 64) + (i_inner * 8)) + (k_outer_outer * 2)) + k_inner) + 16)] * ph_3_shared[((((((int)threadIdx.x) / 5) * 10) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN[(i_inner + 4)] = (T_batch_matmul_NN[(i_inner + 4)] + (ph_0[((((((((int)threadIdx.x) / 5) * 64) + (i_inner * 8)) + (k_outer_outer * 2)) + k_inner) + 32)] * ph_3_shared[((((((int)threadIdx.x) / 5) * 10) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
        T_batch_matmul_NN[(i_inner + 6)] = (T_batch_matmul_NN[(i_inner + 6)] + (ph_0[((((((((int)threadIdx.x) / 5) * 64) + (i_inner * 8)) + (k_outer_outer * 2)) + k_inner) + 48)] * ph_3_shared[((((((int)threadIdx.x) / 5) * 10) + (k_inner * 5)) + (((int)threadIdx.x) % 5))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 2; ++i1_inner) {
    compute[((((((int)threadIdx.x) / 5) * 40) + (i1_inner * 5)) + (((int)threadIdx.x) % 5))] = atanhf(T_batch_matmul_NN[i1_inner]);
    compute[(((((((int)threadIdx.x) / 5) * 40) + (i1_inner * 5)) + (((int)threadIdx.x) % 5)) + 10)] = atanhf(T_batch_matmul_NN[(i1_inner + 2)]);
    compute[(((((((int)threadIdx.x) / 5) * 40) + (i1_inner * 5)) + (((int)threadIdx.x) % 5)) + 20)] = atanhf(T_batch_matmul_NN[(i1_inner + 4)]);
    compute[(((((((int)threadIdx.x) / 5) * 40) + (i1_inner * 5)) + (((int)threadIdx.x) % 5)) + 30)] = atanhf(T_batch_matmul_NN[(i1_inner + 6)]);
  }
}

