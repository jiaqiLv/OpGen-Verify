
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_6) {
  float T_batch_matmul_NN[4];
  __shared__ float ph_6_shared[4];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 4; ++i_outer_inner_init) {
    T_batch_matmul_NN[i_outer_inner_init] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 5; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 4) {
      ph_6_shared[((int)threadIdx.x)] = ph_6[(((((((int)blockIdx.x) / 5) * 100) + (((int)threadIdx.x) * 25)) + (k_outer_outer * 5)) + (((int)blockIdx.x) % 5))];
    }
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 4; ++i_outer_inner) {
      T_batch_matmul_NN[i_outer_inner] = (T_batch_matmul_NN[i_outer_inner] + (ph_0[(((((((int)blockIdx.x) / 5) * 160) + (((int)threadIdx.x) * 20)) + (i_outer_inner * 5)) + k_outer_outer)] * ph_6_shared[(((int)threadIdx.x) >> 1)]));
    }
  }
  for (int i1_inner = 0; i1_inner < 4; ++i1_inner) {
    compute[(((((((int)blockIdx.x) / 5) * 160) + (((int)threadIdx.x) * 20)) + (i1_inner * 5)) + (((int)blockIdx.x) % 5))] = acoshf(T_batch_matmul_NN[i1_inner]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

