
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}
