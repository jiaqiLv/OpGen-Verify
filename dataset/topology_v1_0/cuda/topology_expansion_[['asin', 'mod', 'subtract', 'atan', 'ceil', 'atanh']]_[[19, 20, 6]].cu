
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = atanhf(ceilf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(36) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
