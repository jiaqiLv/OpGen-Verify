
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(36) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = fmodf(fabsf(ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]);
}

