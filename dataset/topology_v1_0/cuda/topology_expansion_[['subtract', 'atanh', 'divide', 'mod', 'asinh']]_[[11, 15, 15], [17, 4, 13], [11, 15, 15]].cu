
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / atanhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((int)blockIdx.x)] = fmodf((ph_0[((int)blockIdx.x)] / atanhf(ph_0[((int)blockIdx.x)])), ph_0[((int)blockIdx.x)]);
}

