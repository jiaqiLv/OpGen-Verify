
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = asinf(__cosf(ph_0[((int)blockIdx.x)]));
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = asinhf(ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

