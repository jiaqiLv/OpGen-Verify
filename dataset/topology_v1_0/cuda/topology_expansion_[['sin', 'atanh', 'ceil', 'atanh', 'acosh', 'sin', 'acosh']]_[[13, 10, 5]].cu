
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(atanhf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
