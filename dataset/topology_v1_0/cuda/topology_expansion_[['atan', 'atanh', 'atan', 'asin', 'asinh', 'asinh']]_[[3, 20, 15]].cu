
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = asinhf(asinhf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]));
}

