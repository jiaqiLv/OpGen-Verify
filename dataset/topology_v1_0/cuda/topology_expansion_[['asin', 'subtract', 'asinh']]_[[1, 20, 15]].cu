
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] - asinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))])));
  }
}

