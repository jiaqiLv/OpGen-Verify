
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(180) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN[1];
  __shared__ float ph_3_shared[180];
  T_batch_matmul_NN[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    ph_3_shared[((int)threadIdx.x)] = ph_3[((((((int)threadIdx.x) / 20) * 40) + (k_outer_outer * 20)) + (((int)threadIdx.x) % 20))];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
      T_batch_matmul_NN[0] = (T_batch_matmul_NN[0] + (ph_0[((((((((int)threadIdx.x) / 20) * 64) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.x) % 20) / 5) * 8)) + (k_outer_outer * 4)) + k_outer_inner)] * ph_3_shared[((((((int)threadIdx.x) / 20) * 20) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5))]));
    }
  }
  compute[((((((int)threadIdx.x) / 20) * 40) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) % 20))] = acosf(T_batch_matmul_NN[0]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

