
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((0.000000e+00f - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = __sinf((ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf((0.000000e+00f - ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}
