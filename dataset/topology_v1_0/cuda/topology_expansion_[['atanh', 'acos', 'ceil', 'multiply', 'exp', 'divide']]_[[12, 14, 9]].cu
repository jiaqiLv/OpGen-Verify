
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] * ceilf(acosf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])));
}

