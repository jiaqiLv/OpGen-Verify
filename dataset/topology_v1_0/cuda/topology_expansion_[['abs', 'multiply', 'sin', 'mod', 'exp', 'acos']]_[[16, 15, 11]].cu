
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = acosf(__expf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], __sinf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * fabsf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]))));
  }
}

