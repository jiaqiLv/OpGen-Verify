
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = acosf(asinf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]));
}
