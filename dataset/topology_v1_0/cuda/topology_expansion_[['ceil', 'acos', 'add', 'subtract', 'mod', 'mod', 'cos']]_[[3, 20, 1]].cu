
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    T_mod[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fmodf(fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], (0.000000e+00f - acosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]))), ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __cosf(fmodf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], (0.000000e+00f - acosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]))));
  }
}

