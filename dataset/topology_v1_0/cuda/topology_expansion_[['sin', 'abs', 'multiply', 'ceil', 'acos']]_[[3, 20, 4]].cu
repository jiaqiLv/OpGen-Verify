
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (fabsf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

