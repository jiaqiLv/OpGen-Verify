
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(fmodf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(50) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_10, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[1];
  __shared__ float ph_10_shared[10];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 10) {
      ph_10_shared[((int)threadIdx.x)] = ph_10[((((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) / 5) * 20)) + (k_outer_outer * 5)) + (((int)threadIdx.x) % 5))];
    }
    __syncthreads();
    T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (fmodf(ph_0[(((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) / 5) * 4)) + k_outer_outer)], ph_3[(((((int)blockIdx.x) * 40) + ((((int)threadIdx.x) / 5) * 4)) + k_outer_outer)]) * ph_10_shared[(((((int)threadIdx.x) / 25) * 5) + (((int)threadIdx.x) % 5))]));
  }
  T_batch_matmul_NN[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[0];
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])) + ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}

