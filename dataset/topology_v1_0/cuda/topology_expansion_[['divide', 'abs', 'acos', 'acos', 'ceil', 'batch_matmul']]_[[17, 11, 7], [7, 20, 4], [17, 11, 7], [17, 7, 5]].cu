
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(25) default_function_kernel_3(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3, float* __restrict__ ph_9) {
  float T_batch_matmul_NN_local[8];
  __shared__ float T_divide_shared[160];
  __shared__ float ph_9_shared[160];
  for (int b_c_outer_inner_init = 0; b_c_outer_inner_init < 4; ++b_c_outer_inner_init) {
    T_batch_matmul_NN_local[b_c_outer_inner_init] = 0.000000e+00f;
    T_batch_matmul_NN_local[(b_c_outer_inner_init + 4)] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 7; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_outer_outer * 5) + (((int)threadIdx.x) / 5)) < 32) {
      T_divide_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 25) + ((int)threadIdx.x))] = (ph_0[((ax0_ax1_fused_ax2_fused_outer_outer * 25) + ((int)threadIdx.x))] / ph_3[((ax0_ax1_fused_ax2_fused_outer_outer * 25) + ((int)threadIdx.x))]);
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 7; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    if (((ax0_ax1_fused_ax2_fused_outer_outer_1 * 5) + (((int)threadIdx.x) / 5)) < 32) {
      ph_9_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 25) + ((int)threadIdx.x))] = ph_9[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 25) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 4; ++k_outer_inner) {
    for (int b_c_outer_inner = 0; b_c_outer_inner < 4; ++b_c_outer_inner) {
      T_batch_matmul_NN_local[b_c_outer_inner] = (T_batch_matmul_NN_local[b_c_outer_inner] + (T_divide_shared[(((b_c_outer_inner * 20) + ((((int)threadIdx.x) / 5) * 4)) + k_outer_inner)] * ph_9_shared[(((b_c_outer_inner * 20) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5))]));
      T_batch_matmul_NN_local[(b_c_outer_inner + 4)] = (T_batch_matmul_NN_local[(b_c_outer_inner + 4)] + (T_divide_shared[((((b_c_outer_inner * 20) + ((((int)threadIdx.x) / 5) * 4)) + k_outer_inner) + 80)] * ph_9_shared[((((b_c_outer_inner * 20) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5)) + 80)]));
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    T_batch_matmul_NN[((b_inner * 25) + ((int)threadIdx.x))] = T_batch_matmul_NN_local[b_inner];
    T_batch_matmul_NN[(((b_inner * 25) + ((int)threadIdx.x)) + 100)] = T_batch_matmul_NN_local[(b_inner + 4)];
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

