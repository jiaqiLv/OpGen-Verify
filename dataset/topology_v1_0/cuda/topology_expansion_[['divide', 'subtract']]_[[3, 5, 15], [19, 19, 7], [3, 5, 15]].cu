
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(5) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))] - ph_3[((((int)blockIdx.x) * 5) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))]);
}
