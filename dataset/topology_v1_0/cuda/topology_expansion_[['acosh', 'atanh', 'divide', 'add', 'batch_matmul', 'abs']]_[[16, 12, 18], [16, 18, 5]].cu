
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_add, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    T_add[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + (atanhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(5) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_6) {
  float T_batch_matmul_NN[9];
  __shared__ float ph_6_shared[40];
  for (int i_inner_init = 0; i_inner_init < 9; ++i_inner_init) {
    T_batch_matmul_NN[i_inner_init] = 0.000000e+00f;
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 8; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    ph_6_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 5) + ((int)threadIdx.x))] = ph_6[((ax0_ax1_fused_ax2_fused_outer_outer * 5) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
    for (int k_inner = 0; k_inner < 4; ++k_inner) {
      for (int i_inner = 0; i_inner < 9; ++i_inner) {
        T_batch_matmul_NN[i_inner] = (T_batch_matmul_NN[i_inner] + (ph_0[(((i_inner * 8) + (k_outer_inner * 4)) + k_inner)] * ph_6_shared[(((k_outer_inner * 20) + (k_inner * 5)) + ((int)threadIdx.x))]));
      }
    }
  }
  for (int i1_inner = 0; i1_inner < 9; ++i1_inner) {
    compute[((i1_inner * 5) + ((int)threadIdx.x))] = fabsf(T_batch_matmul_NN[i1_inner]);
  }
}

