
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fabsf(atanhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))], fabsf(__sinf(ph_0[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}
