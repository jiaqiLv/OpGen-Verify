
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[16];
  __shared__ float compute_shared[80];
  __shared__ float ph_4_shared[10];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 4; ++i_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 2; ++b_c_inner_init) {
      for (int i_c_inner_init = 0; i_c_inner_init < 2; ++i_c_inner_init) {
        T_batch_matmul_NN_local[(((b_c_inner_init * 8) + (i_c_outer_inner_init * 2)) + i_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 80; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[ax0_ax1_fused_ax2_fused_outer_outer] = fabsf(ph_0[((((int)blockIdx.x) * 80) + ax0_ax1_fused_ax2_fused_outer_outer)]);
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 10; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_4_shared[ax0_ax1_fused_ax2_fused_outer_outer_1] = ph_4[((((int)blockIdx.x) * 10) + ax0_ax1_fused_ax2_fused_outer_outer_1)];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    for (int i_c_outer_inner = 0; i_c_outer_inner < 4; ++i_c_outer_inner) {
      for (int b_c_inner = 0; b_c_inner < 2; ++b_c_inner) {
        for (int i_c_inner = 0; i_c_inner < 2; ++i_c_inner) {
          T_batch_matmul_NN_local[(((b_c_inner * 8) + (i_c_outer_inner * 2)) + i_c_inner)] = (T_batch_matmul_NN_local[(((b_c_inner * 8) + (i_c_outer_inner * 2)) + i_c_inner)] + (compute_shared[((((b_c_inner * 40) + (i_c_outer_inner * 10)) + (i_c_inner * 5)) + k_outer_inner)] * ph_4_shared[((b_c_inner * 5) + k_outer_inner)]));
        }
      }
    }
  }
  for (int b_inner = 0; b_inner < 2; ++b_inner) {
    for (int i_inner = 0; i_inner < 8; ++i_inner) {
      T_batch_matmul_NN[(((((int)blockIdx.x) * 16) + (b_inner * 8)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 8) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

