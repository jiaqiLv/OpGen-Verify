
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ ph_0) {
  T_divide[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) / ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}
