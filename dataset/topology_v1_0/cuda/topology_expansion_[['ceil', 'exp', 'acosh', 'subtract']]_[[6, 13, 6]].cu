
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] - acoshf(__expf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

