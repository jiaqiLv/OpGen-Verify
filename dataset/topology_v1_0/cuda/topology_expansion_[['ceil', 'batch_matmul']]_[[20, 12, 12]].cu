
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  float T_batch_matmul_NN_local[1];
  __shared__ float ph_3_shared[50];
  T_batch_matmul_NN_local[0] = 0.000000e+00f;
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 2; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    if (((ax0_ax1_fused_ax2_fused_outer_outer * 4) + (((int)threadIdx.x) / 10)) < 5) {
      ph_3_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 40) + ((int)threadIdx.x))] = ph_3[((((((int)blockIdx.x) >> 1) * 50) + (ax0_ax1_fused_ax2_fused_outer_outer * 40)) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 5; ++k_outer_inner) {
    T_batch_matmul_NN_local[0] = (T_batch_matmul_NN_local[0] + (ph_0[((((((((int)blockIdx.x) >> 1) * 80) + ((((int)threadIdx.x) / 20) * 40)) + ((((int)blockIdx.x) & 1) * 20)) + (((((int)threadIdx.x) % 20) / 5) * 5)) + k_outer_inner)] * ph_3_shared[((((((int)threadIdx.x) / 20) * 25) + (k_outer_inner * 5)) + (((int)threadIdx.x) % 5))]));
  }
  T_batch_matmul_NN[(((((((int)blockIdx.x) >> 1) * 80) + ((((int)threadIdx.x) / 20) * 40)) + ((((int)blockIdx.x) & 1) * 20)) + (((int)threadIdx.x) % 20))] = T_batch_matmul_NN_local[0];
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

