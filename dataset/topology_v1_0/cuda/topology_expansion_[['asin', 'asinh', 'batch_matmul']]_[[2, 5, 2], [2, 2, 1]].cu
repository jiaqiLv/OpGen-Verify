
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_batch_matmul_NN, float* __restrict__ ph_0, float* __restrict__ ph_4) {
  float T_batch_matmul_NN_local[8];
  __shared__ float compute_shared[80];
  __shared__ float ph_4_shared[40];
  for (int i_c_outer_inner_init = 0; i_c_outer_inner_init < 2; ++i_c_outer_inner_init) {
    for (int b_c_inner_init = 0; b_c_inner_init < 4; ++b_c_inner_init) {
      T_batch_matmul_NN_local[((b_c_inner_init * 2) + i_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer = 0; ax0_ax1_fused_ax2_fused_outer_outer < 40; ++ax0_ax1_fused_ax2_fused_outer_outer) {
    compute_shared[((ax0_ax1_fused_ax2_fused_outer_outer * 2) + ((int)threadIdx.x))] = asinhf(ph_0[(((((ax0_ax1_fused_ax2_fused_outer_outer / 5) * 40) + (((int)blockIdx.x) * 10)) + ((ax0_ax1_fused_ax2_fused_outer_outer % 5) * 2)) + ((int)threadIdx.x))]);
  }
  for (int ax0_ax1_fused_ax2_fused_outer_outer_1 = 0; ax0_ax1_fused_ax2_fused_outer_outer_1 < 20; ++ax0_ax1_fused_ax2_fused_outer_outer_1) {
    ph_4_shared[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 2) + ((int)threadIdx.x))] = ph_4[((ax0_ax1_fused_ax2_fused_outer_outer_1 * 2) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int i_c_outer_inner = 0; i_c_outer_inner < 2; ++i_c_outer_inner) {
    for (int k_inner = 0; k_inner < 5; ++k_inner) {
      for (int b_c_inner = 0; b_c_inner < 4; ++b_c_inner) {
        T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_outer_inner)] = (T_batch_matmul_NN_local[((b_c_inner * 2) + i_c_outer_inner)] + (compute_shared[((((((int)threadIdx.x) * 40) + (b_c_inner * 10)) + (i_c_outer_inner * 5)) + k_inner)] * ph_4_shared[(((((int)threadIdx.x) * 20) + (b_c_inner * 5)) + k_inner)]));
      }
    }
  }
  for (int b_inner = 0; b_inner < 4; ++b_inner) {
    for (int i_inner = 0; i_inner < 2; ++i_inner) {
      T_batch_matmul_NN[((((((int)threadIdx.x) * 32) + (b_inner * 8)) + (((int)blockIdx.x) * 2)) + i_inner)] = T_batch_matmul_NN_local[((b_inner * 2) + i_inner)];
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

