
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void default_function_kernel(float* __restrict__ T_subtract, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_subtract[((int)blockIdx.x)] = (ph_0[((int)blockIdx.x)] - ph_3[((int)blockIdx.x)]);
}

