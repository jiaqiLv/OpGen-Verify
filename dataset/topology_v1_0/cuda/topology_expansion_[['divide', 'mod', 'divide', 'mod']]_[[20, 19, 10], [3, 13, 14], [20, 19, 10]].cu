
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_mod[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = fmodf((ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] / fmodf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], ph_3[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])), ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}
