
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
}
