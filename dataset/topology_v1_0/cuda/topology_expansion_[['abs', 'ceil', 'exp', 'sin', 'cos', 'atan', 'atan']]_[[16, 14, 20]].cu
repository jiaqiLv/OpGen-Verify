
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = fabsf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __expf(ceilf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_4(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}
