
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(14) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) < 323) {
    adaptive_pool_max[((((int)blockIdx.x) * 14) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 12; ++rv0) {
    for (int rv1 = 0; rv1 < 2; ++rv1) {
      if (((((int)blockIdx.x) * 14) + ((int)threadIdx.x)) < 323) {
        adaptive_pool_max[((((int)blockIdx.x) * 14) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 14) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 336) + (((int)threadIdx.x) * 24)) + (rv0 * 2)) + rv1)]);
      }
    }
  }
}

