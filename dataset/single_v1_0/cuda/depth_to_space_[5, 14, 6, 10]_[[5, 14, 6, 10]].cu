
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ depth_to_space) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 225) {
    depth_to_space[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = data[(((((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) / 45) * 840) + ((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) % 60) / 5) % 2) * 360)) + (((((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 20) % 2) * 180)) + (((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) % 45) / 15) * 60)) + ((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) % 60) / 5) / 2) * 10)) + ((((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 20) / 2))];
  }
}

