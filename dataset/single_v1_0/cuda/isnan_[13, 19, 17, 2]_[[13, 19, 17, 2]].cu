
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(signed char* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 4199) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((signed char)(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] != data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}
