
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(7) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) < 225) {
    adaptive_pool_max[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 2; ++rv0) {
    for (int rv1 = 0; rv1 < 16; ++rv1) {
      if (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) < 225) {
        adaptive_pool_max[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))], data[((((((int)blockIdx.x) * 224) + (((int)threadIdx.x) * 32)) + (rv0 * 16)) + rv1)]);
      }
    }
  }
}

