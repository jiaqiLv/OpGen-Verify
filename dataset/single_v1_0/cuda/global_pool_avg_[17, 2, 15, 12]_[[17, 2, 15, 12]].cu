
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 17) {
    adaptive_pool_avg[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * 5.555556e-03f);
  }
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 15; ++rv0) {
    for (int rv1 = 0; rv1 < 12; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 360) + (((int)threadIdx.x) * 180)) + (rv0 * 12)) + rv1)]);
    }
  }
}

