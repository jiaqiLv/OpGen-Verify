
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      for (int rv2 = 0; rv2 < 3; ++rv2) {
        pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], (((((1 <= ((((((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) >> 3)) / 7) * 2) + rv0)) && ((((((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) >> 3)) / 7) + (rv0 >> 1)) < 2)) && (1 <= (((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) % 7) * 2) + rv1))) && (1 <= (((((int)threadIdx.x) & 7) * 2) + rv2))) ? data[(((((((((((int)blockIdx.x) / 7) * 672) + (((((((int)blockIdx.x) % 7) * 2) + (((int)threadIdx.x) >> 3)) / 7) * 448)) + (rv0 * 224)) + ((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) % 7) * 32)) + (rv1 * 16)) + ((((int)threadIdx.x) & 7) * 2)) + rv2) - 241)] : -3.402823e+38f));
      }
    }
  }
}

