
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(15) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((int)threadIdx.x)] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 17; ++rv0) {
    for (int rv1 = 0; rv1 < 4; ++rv1) {
      adaptive_pool_max[((int)threadIdx.x)] = max(adaptive_pool_max[((int)threadIdx.x)], data[(((((int)threadIdx.x) * 68) + (rv0 * 4)) + rv1)]);
    }
  }
}

