
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((int)threadIdx.x)] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 6; ++rv0) {
    for (int rv1 = 0; rv1 < 18; ++rv1) {
      adaptive_pool_max[((int)threadIdx.x)] = max(adaptive_pool_max[((int)threadIdx.x)], data[(((((int)threadIdx.x) * 108) + (rv0 * 18)) + rv1)]);
    }
  }
}

