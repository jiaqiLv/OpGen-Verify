
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(11) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 11) + ((int)threadIdx.x))] = sinhf(data[((((int)blockIdx.x) * 11) + ((int)threadIdx.x))]);
}

