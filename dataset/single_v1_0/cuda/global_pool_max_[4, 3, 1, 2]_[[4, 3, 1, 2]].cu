
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((int)threadIdx.x)] = -3.402823e+38f;
  for (int rv1 = 0; rv1 < 2; ++rv1) {
    adaptive_pool_max[((int)threadIdx.x)] = max(adaptive_pool_max[((int)threadIdx.x)], data[((((int)threadIdx.x) * 2) + rv1)]);
  }
}

