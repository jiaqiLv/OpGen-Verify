
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 105) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 9; ++k) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 105) {
      compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 288) + (((int)threadIdx.x) * 9)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(14) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((((int)blockIdx.x) * 14) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int k = 0; k < 9; ++k) {
    T_softmax_maxelem[((((int)blockIdx.x) * 14) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 14) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 126) + (((int)threadIdx.x) * 9)) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 945) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 9)]) - __logf(compute_1[(((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) / 9)]));
  }
}
