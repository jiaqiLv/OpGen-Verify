
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 1)) < 187) {
    compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = atanhf(data[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
  }
}

