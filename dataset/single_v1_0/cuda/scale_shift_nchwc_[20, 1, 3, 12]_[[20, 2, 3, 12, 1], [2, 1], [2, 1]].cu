
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 9) * 2) + (((int)threadIdx.x) >> 2)) / 9)]) + Shift[((((((int)blockIdx.x) % 9) * 2) + (((int)threadIdx.x) >> 2)) / 9)]);
}

