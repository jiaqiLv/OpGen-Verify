
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(28) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((int)threadIdx.x)] = ((data[((int)threadIdx.x)] * Scale[((((int)threadIdx.x) & 3) >> 1)]) + Shift[((((int)threadIdx.x) & 3) >> 1)]);
}

