
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[11];
  __shared__ float left_matrix_shared[22];
  __shared__ float right_matrix_shared[2];
  T_matmul_local[0] = 0.000000e+00f;
  T_matmul_local[1] = 0.000000e+00f;
  T_matmul_local[2] = 0.000000e+00f;
  T_matmul_local[3] = 0.000000e+00f;
  T_matmul_local[4] = 0.000000e+00f;
  T_matmul_local[5] = 0.000000e+00f;
  T_matmul_local[6] = 0.000000e+00f;
  T_matmul_local[7] = 0.000000e+00f;
  T_matmul_local[8] = 0.000000e+00f;
  T_matmul_local[9] = 0.000000e+00f;
  T_matmul_local[10] = 0.000000e+00f;
  for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 22; ++ax0_ax1_fused_outer_outer) {
    left_matrix_shared[ax0_ax1_fused_outer_outer] = left_matrix[ax0_ax1_fused_outer_outer];
  }
  for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 2; ++ax0_ax1_fused_outer_outer_1) {
    right_matrix_shared[ax0_ax1_fused_outer_outer_1] = right_matrix[((ax0_ax1_fused_outer_outer_1 * 11) + ((int)blockIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
    T_matmul_local[0] = (T_matmul_local[0] + (left_matrix_shared[k_outer_inner] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[1] = (T_matmul_local[1] + (left_matrix_shared[(k_outer_inner + 2)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[2] = (T_matmul_local[2] + (left_matrix_shared[(k_outer_inner + 4)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[3] = (T_matmul_local[3] + (left_matrix_shared[(k_outer_inner + 6)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[4] = (T_matmul_local[4] + (left_matrix_shared[(k_outer_inner + 8)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[5] = (T_matmul_local[5] + (left_matrix_shared[(k_outer_inner + 10)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[6] = (T_matmul_local[6] + (left_matrix_shared[(k_outer_inner + 12)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[7] = (T_matmul_local[7] + (left_matrix_shared[(k_outer_inner + 14)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[8] = (T_matmul_local[8] + (left_matrix_shared[(k_outer_inner + 16)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[9] = (T_matmul_local[9] + (left_matrix_shared[(k_outer_inner + 18)] * right_matrix_shared[k_outer_inner]));
    T_matmul_local[10] = (T_matmul_local[10] + (left_matrix_shared[(k_outer_inner + 20)] * right_matrix_shared[k_outer_inner]));
  }
  T_matmul[((int)blockIdx.x)] = T_matmul_local[0];
  T_matmul[(((int)blockIdx.x) + 11)] = T_matmul_local[1];
  T_matmul[(((int)blockIdx.x) + 22)] = T_matmul_local[2];
  T_matmul[(((int)blockIdx.x) + 33)] = T_matmul_local[3];
  T_matmul[(((int)blockIdx.x) + 44)] = T_matmul_local[4];
  T_matmul[(((int)blockIdx.x) + 55)] = T_matmul_local[5];
  T_matmul[(((int)blockIdx.x) + 66)] = T_matmul_local[6];
  T_matmul[(((int)blockIdx.x) + 77)] = T_matmul_local[7];
  T_matmul[(((int)blockIdx.x) + 88)] = T_matmul_local[8];
  T_matmul[(((int)blockIdx.x) + 99)] = T_matmul_local[9];
  T_matmul[(((int)blockIdx.x) + 110)] = T_matmul_local[10];
}

