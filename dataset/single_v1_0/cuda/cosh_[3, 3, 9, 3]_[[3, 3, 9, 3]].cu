
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 243) {
    compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = coshf(data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
  }
}

