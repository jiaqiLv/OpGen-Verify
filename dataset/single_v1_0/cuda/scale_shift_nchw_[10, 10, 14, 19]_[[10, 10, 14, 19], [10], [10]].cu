
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) < 3325) {
    ScaleShift[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) * 24) + (((int)threadIdx.x) >> 1)) % 1330) / 133)]) + Shift[((((((int)blockIdx.x) * 24) + (((int)threadIdx.x) >> 1)) % 1330) / 133)]);
  }
}

