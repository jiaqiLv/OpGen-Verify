
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(45) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 45) + ((int)threadIdx.x))] = asinf(data[((((int)blockIdx.x) * 45) + ((int)threadIdx.x))]);
}
