
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(20) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 20) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 50) / 25) * 5) + (((int)threadIdx.x) % 5))]) + Shift[((((((int)blockIdx.x) % 50) / 25) * 5) + (((int)threadIdx.x) % 5))]);
}

