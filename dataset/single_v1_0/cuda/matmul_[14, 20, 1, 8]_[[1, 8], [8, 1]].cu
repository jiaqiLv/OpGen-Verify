
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  T_matmul[0] = 0.000000e+00f;
  for (int k = 0; k < 8; ++k) {
    T_matmul[0] = (T_matmul[0] + (left_matrix[k] * right_matrix[k]));
  }
}

