
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] / (((float)(((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 3) + 3) % 8) == 0) ? ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) + 19) >> 3) : (((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) + 19) >> 3) + 1)) - (((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) >> 3))) * ((float)(((((((((int)threadIdx.x) & 7) * 5) + 5) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) : (((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 5) >> 3)))));
}

extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < (((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 3) + 3) % 8) == 0) ? ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) + 19) >> 3) : (((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) + 19) >> 3) + 1)) - (((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 5) + 5) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) : (((((((int)threadIdx.x) & 7) * 5) + 5) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 5) >> 3)); ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] + data[((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) >> 3) * 95) + ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 19) >> 3) * 5)) + (rv0 * 5)) + (((((int)threadIdx.x) & 7) * 5) >> 3)) + rv1)]);
    }
  }
}

