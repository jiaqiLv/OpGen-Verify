
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 6) + 6) % 8) == 0) ? ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 3) + 3) >> 2) : (((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 3) + 3) >> 2) + 1)) - (((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 6) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 4) + 4) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 5) + 5) >> 1) : (((((((int)threadIdx.x) & 7) * 5) + 5) >> 1) + 1)) - (((((int)threadIdx.x) & 7) * 20) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) >> 3) * 120) + ((((((((int)blockIdx.x) * 5) + (((int)threadIdx.x) >> 3)) & 7) * 6) >> 3) * 20)) + (rv0 * 20)) + (((((int)threadIdx.x) & 7) * 20) >> 3)) + rv1)]);
    }
  }
}

