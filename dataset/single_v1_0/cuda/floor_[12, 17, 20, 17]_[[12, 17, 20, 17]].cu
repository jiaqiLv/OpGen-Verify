
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = floorf(data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))]);
}
