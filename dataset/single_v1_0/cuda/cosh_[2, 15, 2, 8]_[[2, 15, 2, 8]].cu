
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(6) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))] = coshf(data[((((int)blockIdx.x) * 6) + ((int)threadIdx.x))]);
}

