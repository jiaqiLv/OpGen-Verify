
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[8];
  __shared__ float left_matrix_shared[68];
  __shared__ float right_matrix_shared[136];
  for (int ax1_c_outer_inner_init = 0; ax1_c_outer_inner_init < 2; ++ax1_c_outer_inner_init) {
    for (int ax0_c_inner_init = 0; ax0_c_inner_init < 2; ++ax0_c_inner_init) {
      for (int ax1_c_inner_init = 0; ax1_c_inner_init < 2; ++ax1_c_inner_init) {
        T_matmul_local[(((ax0_c_inner_init * 4) + (ax1_c_outer_inner_init * 2)) + ax1_c_inner_init)] = 0.000000e+00f;
      }
    }
  }
  for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 17; ++ax0_ax1_fused_outer_outer) {
    left_matrix_shared[((ax0_ax1_fused_outer_outer * 4) + ((int)threadIdx.x))] = left_matrix[(((((int)blockIdx.x) * 68) + (ax0_ax1_fused_outer_outer * 4)) + ((int)threadIdx.x))];
  }
  for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 17; ++ax0_ax1_fused_outer_outer_1) {
    *(float2*)(right_matrix_shared + ((ax0_ax1_fused_outer_outer_1 * 8) + (((int)threadIdx.x) * 2))) = *(float2*)(right_matrix + ((ax0_ax1_fused_outer_outer_1 * 8) + (((int)threadIdx.x) * 2)));
  }
  __syncthreads();
  for (int ax1_c_outer_inner = 0; ax1_c_outer_inner < 2; ++ax1_c_outer_inner) {
    for (int k_inner = 0; k_inner < 17; ++k_inner) {
      for (int ax0_c_inner = 0; ax0_c_inner < 2; ++ax0_c_inner) {
        for (int ax1_c_inner = 0; ax1_c_inner < 2; ++ax1_c_inner) {
          T_matmul_local[(((ax0_c_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner)] = (T_matmul_local[(((ax0_c_inner * 4) + (ax1_c_outer_inner * 2)) + ax1_c_inner)] + (left_matrix_shared[((((((int)threadIdx.x) >> 1) * 34) + (ax0_c_inner * 17)) + k_inner)] * right_matrix_shared[((((k_inner * 8) + ((((int)threadIdx.x) & 1) * 4)) + (ax1_c_outer_inner * 2)) + ax1_c_inner)]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      T_matmul[(((((((int)blockIdx.x) * 32) + ((((int)threadIdx.x) >> 1) * 16)) + (ax0_inner * 8)) + ((((int)threadIdx.x) & 1) * 4)) + ax1_inner)] = T_matmul_local[((ax0_inner * 4) + ax1_inner)];
    }
  }
}

