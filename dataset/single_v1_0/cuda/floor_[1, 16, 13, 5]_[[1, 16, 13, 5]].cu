
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 65) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = floorf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

