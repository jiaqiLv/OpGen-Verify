
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < ((((((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) + 1) % 8) == 0) ? ((((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) * 17) + 17) >> 3) : (((((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) * 17) + 17) >> 3) + 1)) - ((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) * 2)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 5) + 5) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 13) + 13) >> 3) : (((((((int)threadIdx.x) & 7) * 13) + 13) >> 3) + 1)) - (((((int)threadIdx.x) & 7) * 13) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))], data[((((((((((int)blockIdx.x) * 3) + (((int)threadIdx.x) >> 4)) >> 2) * 221) + ((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) * 26)) + (rv0 * 13)) + (((((int)threadIdx.x) & 7) * 13) >> 3)) + rv1)]);
    }
  }
}

