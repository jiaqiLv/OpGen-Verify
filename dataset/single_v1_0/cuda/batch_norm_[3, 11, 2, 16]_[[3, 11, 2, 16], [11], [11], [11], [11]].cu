
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(44) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ moving_mean, float* __restrict__ moving_var) {
  T_divide[((((int)blockIdx.x) * 44) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 44) + ((int)threadIdx.x))] - moving_mean[((((((int)blockIdx.x) & 7) * 11) + (((int)threadIdx.x) >> 2)) >> 3)]) / sqrtf((moving_var[((((((int)blockIdx.x) & 7) * 11) + (((int)threadIdx.x) >> 2)) >> 3)] + 1.000000e-05f)));
}

