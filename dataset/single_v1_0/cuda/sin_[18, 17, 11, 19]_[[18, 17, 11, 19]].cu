
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 31977) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = __sinf(data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

