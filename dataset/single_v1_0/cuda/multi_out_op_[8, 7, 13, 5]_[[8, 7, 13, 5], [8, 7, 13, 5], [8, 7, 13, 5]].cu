
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(52) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 455) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

