
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[4];
  __shared__ float left_matrix_shared[36];
  __shared__ float right_matrix_shared[36];
  T_matmul_local[0] = 0.000000e+00f;
  T_matmul_local[1] = 0.000000e+00f;
  T_matmul_local[2] = 0.000000e+00f;
  T_matmul_local[3] = 0.000000e+00f;
  for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 3; ++ax0_ax1_fused_outer_outer) {
    *(float3*)(left_matrix_shared + ((ax0_ax1_fused_outer_outer * 12) + (((int)threadIdx.x) * 3))) = *(float3*)(left_matrix + ((ax0_ax1_fused_outer_outer * 12) + (((int)threadIdx.x) * 3)));
  }
  for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 9; ++ax0_ax1_fused_outer_outer_1) {
    right_matrix_shared[((ax0_ax1_fused_outer_outer_1 * 4) + ((int)threadIdx.x))] = right_matrix[((ax0_ax1_fused_outer_outer_1 * 4) + ((int)threadIdx.x))];
  }
  __syncthreads();
  for (int k_outer_inner = 0; k_outer_inner < 9; ++k_outer_inner) {
    T_matmul_local[0] = (T_matmul_local[0] + (left_matrix_shared[k_outer_inner] * right_matrix_shared[((k_outer_inner * 4) + ((int)threadIdx.x))]));
    T_matmul_local[1] = (T_matmul_local[1] + (left_matrix_shared[(k_outer_inner + 9)] * right_matrix_shared[((k_outer_inner * 4) + ((int)threadIdx.x))]));
    T_matmul_local[2] = (T_matmul_local[2] + (left_matrix_shared[(k_outer_inner + 18)] * right_matrix_shared[((k_outer_inner * 4) + ((int)threadIdx.x))]));
    T_matmul_local[3] = (T_matmul_local[3] + (left_matrix_shared[(k_outer_inner + 27)] * right_matrix_shared[((k_outer_inner * 4) + ((int)threadIdx.x))]));
  }
  T_matmul[((int)threadIdx.x)] = T_matmul_local[0];
  T_matmul[(((int)threadIdx.x) + 4)] = T_matmul_local[1];
  T_matmul[(((int)threadIdx.x) + 8)] = T_matmul_local[2];
  T_matmul[(((int)threadIdx.x) + 12)] = T_matmul_local[3];
}

