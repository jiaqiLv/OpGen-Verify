
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * weight[((((int)blockIdx.x) % 25) / 5)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((int)blockIdx.x) / 25) * 320) + ((((int)blockIdx.x) % 5) * 64)) + ((int)threadIdx.x))] * 2.000000e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 5; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + (data[(((((((int)blockIdx.x) / 5) * 1600) + (k1 * 320)) + ((((int)blockIdx.x) % 5) * 64)) + ((int)threadIdx.x))] * data[(((((((int)blockIdx.x) / 5) * 1600) + (k1 * 320)) + ((((int)blockIdx.x) % 5) * 64)) + ((int)threadIdx.x))]));
  }
}

