
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(50) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 25) + (((int)threadIdx.x) >> 1)) < 5202) {
    compute[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))] = (1.000000e+00f / sqrtf(data[((((int)blockIdx.x) * 50) + ((int)threadIdx.x))]));
  }
}

