
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      for (int rv2 = 0; rv2 < 3; ++rv2) {
        pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], (((((1 <= ((((((int)threadIdx.x) & 15) >> 3) * 2) + rv0)) && ((((((int)threadIdx.x) & 15) >> 3) + (rv0 >> 1)) < 2)) && (1 <= rv1)) && (1 <= (((((int)threadIdx.x) & 7) * 2) + rv2))) ? data[((((((((((int)blockIdx.x) * 192) + ((((int)threadIdx.x) >> 4) * 96)) + (((((int)threadIdx.x) & 15) >> 3) * 64)) + (rv0 * 32)) + (rv1 * 16)) + ((((int)threadIdx.x) & 7) * 2)) + rv2) - 49)] : -3.402823e+38f));
      }
    }
  }
}

