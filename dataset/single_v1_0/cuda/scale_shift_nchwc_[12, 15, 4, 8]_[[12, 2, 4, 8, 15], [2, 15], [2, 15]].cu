
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 30) / 15) * 15) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 15))]) + Shift[((((((int)blockIdx.x) % 30) / 15) * 15) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 15))]);
}

