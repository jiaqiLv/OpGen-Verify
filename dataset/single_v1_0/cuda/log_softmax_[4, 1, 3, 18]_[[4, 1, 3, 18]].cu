
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 18; ++k) {
    compute[((int)threadIdx.x)] = (compute[((int)threadIdx.x)] + __expf((data[((((int)threadIdx.x) * 18) + k)] - T_softmax_maxelem[((int)threadIdx.x)])));
  }
}

extern "C" __global__ void __launch_bounds__(18) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] - T_softmax_maxelem[((int)blockIdx.x)]) - __logf(compute_1[((int)blockIdx.x)]));
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 18; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], data[((((int)threadIdx.x) * 18) + k)]);
  }
}

