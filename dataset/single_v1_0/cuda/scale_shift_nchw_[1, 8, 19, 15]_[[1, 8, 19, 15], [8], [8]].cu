
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] * Scale[(((((int)blockIdx.x) * 8) + (((int)threadIdx.x) / 5)) / 57)]) + Shift[(((((int)blockIdx.x) * 8) + (((int)threadIdx.x) / 5)) / 57)]);
}

