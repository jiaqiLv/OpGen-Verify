
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 486) / 243) * 12) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 12))]) + Shift[((((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 486) / 243) * 12) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 12))]);
}
