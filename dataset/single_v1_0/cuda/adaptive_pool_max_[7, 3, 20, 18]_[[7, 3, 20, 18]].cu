
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ adaptive_pool_max, float* __restrict__ data) {
  adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < (((((((((int)blockIdx.x) & 7) * 4) + 4) % 8) == 0) ? ((((((int)blockIdx.x) & 7) * 5) + 5) >> 1) : (((((((int)blockIdx.x) & 7) * 5) + 5) >> 1) + 1)) - (((((int)blockIdx.x) & 7) * 20) >> 3)); ++rv0) {
    for (int rv1 = 0; rv1 < ((((((((int)threadIdx.x) * 2) + 2) % 8) == 0) ? (((((int)threadIdx.x) * 9) + 9) >> 2) : ((((((int)threadIdx.x) * 9) + 9) >> 2) + 1)) - ((((int)threadIdx.x) * 18) >> 3)); ++rv1) {
      adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(adaptive_pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], data[((((((((int)blockIdx.x) >> 3) * 360) + ((((((int)blockIdx.x) & 7) * 20) >> 3) * 18)) + (rv0 * 18)) + ((((int)threadIdx.x) * 18) >> 3)) + rv1)]);
    }
  }
}

