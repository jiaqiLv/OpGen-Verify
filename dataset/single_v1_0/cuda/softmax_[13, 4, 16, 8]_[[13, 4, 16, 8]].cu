
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  T_softmax_norm[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (__expf((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3))])) / T_softmax_expsum[((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_expsum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 8; ++k) {
    T_softmax_expsum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 512) + (((int)threadIdx.x) * 8)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])));
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int k = 0; k < 8; ++k) {
    T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 128) + (((int)threadIdx.x) * 8)) + k)]);
  }
}

