
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 1053) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 15; ++k) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 1053) {
      T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 480) + (((int)threadIdx.x) * 15)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) < 15795) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 15)]) - __logf(compute_1[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 15)]));
  }
}

extern "C" __global__ void __launch_bounds__(52) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 15; ++k) {
    compute[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 780) + (((int)threadIdx.x) * 15)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 52) + ((int)threadIdx.x))])));
  }
}

