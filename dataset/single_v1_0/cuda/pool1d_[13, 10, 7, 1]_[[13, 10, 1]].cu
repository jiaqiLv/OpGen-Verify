
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) < 65) {
    pool_max[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) < 65) {
      pool_max[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))], (((1 <= rv0) && (rv0 < 2)) ? data[((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) + rv0) - 1)] : -3.402823e+38f));
    }
  }
}

