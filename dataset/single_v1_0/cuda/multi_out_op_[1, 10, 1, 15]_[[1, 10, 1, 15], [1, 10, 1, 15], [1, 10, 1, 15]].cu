
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = __cosf((data[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(10) default_function_kernel(float* __restrict__ compute, float* __restrict__ data, float* __restrict__ data_1) {
  compute[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] = sqrtf((data[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))] + data_1[((((int)blockIdx.x) * 10) + ((int)threadIdx.x))]));
}

