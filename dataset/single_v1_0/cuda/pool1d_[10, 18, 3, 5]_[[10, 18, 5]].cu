
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 135) {
    pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 2)) < 135) {
      pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], (((1 <= (((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 3) * 2) + rv0)) && (((rv0 >> 1) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 3)) < 3)) ? data[(((((((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) / 3) * 5) + ((((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 3) * 2)) + rv0) - 1)] : -3.402823e+38f));
    }
  }
}

