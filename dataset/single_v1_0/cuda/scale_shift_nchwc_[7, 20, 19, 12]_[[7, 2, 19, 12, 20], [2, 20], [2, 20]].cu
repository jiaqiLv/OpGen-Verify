
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(60) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 60) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 152) / 76) * 20) + (((int)threadIdx.x) % 20))]) + Shift[((((((int)blockIdx.x) % 152) / 76) * 20) + (((int)threadIdx.x) % 20))]);
}

