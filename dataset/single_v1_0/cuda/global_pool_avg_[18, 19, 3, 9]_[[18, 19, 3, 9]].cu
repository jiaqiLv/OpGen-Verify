
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) < 171) {
    adaptive_pool_avg[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] * 3.703704e-02f);
  }
}

extern "C" __global__ void __launch_bounds__(2) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 9; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 54) + (((int)threadIdx.x) * 27)) + (rv0 * 9)) + rv1)]);
    }
  }
}

