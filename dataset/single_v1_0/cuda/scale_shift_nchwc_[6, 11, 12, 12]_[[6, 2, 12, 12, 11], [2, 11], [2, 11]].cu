
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * Scale[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) % 198) / 99) * 11) + (((((int)blockIdx.x) * 9) + ((int)threadIdx.x)) % 11))]) + Shift[((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) % 198) / 99) * 11) + (((((int)blockIdx.x) * 9) + ((int)threadIdx.x)) % 11))]);
}

