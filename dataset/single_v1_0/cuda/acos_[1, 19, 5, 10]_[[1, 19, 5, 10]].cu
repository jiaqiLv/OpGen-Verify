
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 1)) < 475) {
    compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = acosf(data[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
  }
}

