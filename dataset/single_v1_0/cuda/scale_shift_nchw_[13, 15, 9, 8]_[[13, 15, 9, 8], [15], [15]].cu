
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(40) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 40) + ((int)threadIdx.x))] * Scale[((((((int)blockIdx.x) % 27) * 5) + (((int)threadIdx.x) >> 3)) / 9)]) + Shift[((((((int)blockIdx.x) % 27) * 5) + (((int)threadIdx.x) >> 3)) / 9)]);
}

