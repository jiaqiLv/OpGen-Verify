
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(24) default_function_kernel(float* __restrict__ Scale, float* __restrict__ ScaleShift, float* __restrict__ Shift, float* __restrict__ data) {
  ScaleShift[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] * Scale[((((int)blockIdx.x) % 150) / 10)]) + Shift[((((int)blockIdx.x) % 150) / 10)]);
}

