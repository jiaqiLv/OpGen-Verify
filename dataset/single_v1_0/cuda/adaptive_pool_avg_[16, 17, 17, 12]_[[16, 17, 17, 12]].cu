
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / (((float)(((((((((((int)blockIdx.x) & 1) * 4) + (((int)threadIdx.x) >> 3)) + 1) % 8) == 0) ? ((((((int)blockIdx.x) & 1) * 17) + ((((((int)threadIdx.x) >> 3) * 17) + 17) >> 2)) >> 1) : (((((((int)blockIdx.x) & 1) * 17) + ((((((int)threadIdx.x) >> 3) * 17) + 17) >> 2)) >> 1) + 1)) - ((((int)threadIdx.x) >> 3) * 2)) - ((((int)blockIdx.x) & 1) * 8))) * ((float)(((((((((int)threadIdx.x) & 7) * 4) + 4) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 3) + 3) >> 1) : (((((((int)threadIdx.x) & 7) * 3) + 3) >> 1) + 1)) - (((((int)threadIdx.x) & 7) * 12) >> 3)))));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < ((((((((int)threadIdx.x) >> 3) + 1) % 8) == 0) ? ((((((int)threadIdx.x) >> 3) * 17) + 17) >> 3) : (((((((int)threadIdx.x) >> 3) * 17) + 17) >> 3) + 1)) - ((((int)threadIdx.x) >> 3) * 2)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 4) + 4) % 8) == 0) ? ((((((int)threadIdx.x) & 7) * 3) + 3) >> 1) : (((((((int)threadIdx.x) & 7) * 3) + 3) >> 1) + 1)) - (((((int)threadIdx.x) & 7) * 12) >> 3)); ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + data[(((((((int)blockIdx.x) * 204) + ((((int)threadIdx.x) >> 3) * 24)) + (rv0 * 12)) + (((((int)threadIdx.x) & 7) * 12) >> 3)) + rv1)]);
    }
  }
}

