
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(9) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ data, float* __restrict__ weight) {
  T_cast[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] * weight[((((int)blockIdx.x) & 3) >> 1)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((((int)blockIdx.x) >> 2) * 18) + ((((int)blockIdx.x) & 1) * 9)) + ((int)threadIdx.x))] * 5.000000e-01f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ T_multiply_red, float* __restrict__ data) {
  T_multiply_red[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k1 = 0; k1 < 2; ++k1) {
    T_multiply_red[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (T_multiply_red[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + (data[((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) / 9) * 36) + (k1 * 18)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 18))] * data[((((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) / 9) * 36) + (k1 * 18)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 18))]));
  }
}

