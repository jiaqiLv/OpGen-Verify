
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(7) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))] = sqrtf(data[((((int)blockIdx.x) * 7) + ((int)threadIdx.x))]);
}
