
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(signed char* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 1617) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((signed char)(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] != data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
  }
}

