
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(26) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))] = __log10f(data[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))]);
}

