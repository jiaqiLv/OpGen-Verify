
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] * 9.090909e-03f);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 10; ++rv0) {
    for (int rv1 = 0; rv1 < 11; ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] + data[((((((int)blockIdx.x) * 440) + (((int)threadIdx.x) * 110)) + (rv0 * 11)) + rv1)]);
    }
  }
}

