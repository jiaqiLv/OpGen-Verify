
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(7) default_function_kernel(float* __restrict__ T_matmul, float* __restrict__ left_matrix, float* __restrict__ right_matrix) {
  float T_matmul_local[4];
  __shared__ float left_matrix_shared[16];
  __shared__ float right_matrix_shared[112];
  for (int ax1_c_outer_inner_init = 0; ax1_c_outer_inner_init < 2; ++ax1_c_outer_inner_init) {
    for (int ax0_c_inner_init = 0; ax0_c_inner_init < 2; ++ax0_c_inner_init) {
      T_matmul_local[((ax0_c_inner_init * 2) + ax1_c_outer_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 3; ++ax0_ax1_fused_outer_outer) {
      if (((ax0_ax1_fused_outer_outer * 7) + ((int)threadIdx.x)) < 16) {
        left_matrix_shared[((ax0_ax1_fused_outer_outer * 7) + ((int)threadIdx.x))] = left_matrix[((((((int)blockIdx.x) * 32) + ((((ax0_ax1_fused_outer_outer * 7) + ((int)threadIdx.x)) >> 3) * 16)) + (k_outer_outer * 8)) + (((ax0_ax1_fused_outer_outer * 7) + ((int)threadIdx.x)) & 7))];
      }
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      right_matrix_shared[((ax0_ax1_fused_outer_outer_1 * 7) + ((int)threadIdx.x))] = right_matrix[(((k_outer_outer * 112) + (ax0_ax1_fused_outer_outer_1 * 7)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 8; ++k_outer_inner) {
      for (int ax1_c_outer_inner = 0; ax1_c_outer_inner < 2; ++ax1_c_outer_inner) {
        for (int ax0_c_inner = 0; ax0_c_inner < 2; ++ax0_c_inner) {
          T_matmul_local[((ax0_c_inner * 2) + ax1_c_outer_inner)] = (T_matmul_local[((ax0_c_inner * 2) + ax1_c_outer_inner)] + (left_matrix_shared[((ax0_c_inner * 8) + k_outer_inner)] * right_matrix_shared[(((k_outer_inner * 14) + (((int)threadIdx.x) * 2)) + ax1_c_outer_inner)]));
        }
      }
    }
  }
  for (int ax0_inner = 0; ax0_inner < 2; ++ax0_inner) {
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      T_matmul[((((((int)blockIdx.x) * 28) + (ax0_inner * 14)) + (((int)threadIdx.x) * 2)) + ax1_inner)] = T_matmul_local[((ax0_inner * 2) + ax1_inner)];
    }
  }
}

