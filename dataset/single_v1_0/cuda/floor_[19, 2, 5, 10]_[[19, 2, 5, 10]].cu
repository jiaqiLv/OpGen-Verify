
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 475) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = floorf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

