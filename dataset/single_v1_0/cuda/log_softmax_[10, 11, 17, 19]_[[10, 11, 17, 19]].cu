
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 1)) < 17765) {
    compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 19)]) - __logf(compute_1[(((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) / 19)]));
  }
}

extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) < 935) {
    T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 19; ++k) {
    if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 1)) < 935) {
      T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 304) + (((int)threadIdx.x) * 19)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 935) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 19; ++k) {
    if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 1)) < 935) {
      compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 608) + (((int)threadIdx.x) * 19)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
    }
  }
}
