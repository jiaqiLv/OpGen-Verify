
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(30) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 17; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], data[((((int)threadIdx.x) * 17) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(24) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) / 6)) < 85) {
    compute[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 24) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) / 17)]) - __logf(compute_1[(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) / 17)]));
  }
}

extern "C" __global__ void __launch_bounds__(30) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  compute[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 17; ++k) {
    compute[((int)threadIdx.x)] = (compute[((int)threadIdx.x)] + __expf((data[((((int)threadIdx.x) * 17) + k)] - T_softmax_maxelem[((int)threadIdx.x)])));
  }
}

